#include "hip/hip_runtime.h"
#pragma once
#include"Graph_D.cuh"
#include "kernel_functions.cuh"

namespace Graph_D_H
{
	void Graph_D_H::Graph::startGPU()
	{
		tests();
	}

	void Graph_D_H::Graph::translateBeforeCH()
	{
		ID_hash_D = ID_hash;
		//ranks_D = ranks;
		//partition_Tree_D = partition_Tree;
		nonAdjcentNode_D = nonAdjcentNode;
		NANHash_D = NANHash;
		//father_D = father;
		CHTreeHash_D = CHTreeHash;
		CHTree_D = CHTree;
		CHInfo_D = CHInfo_H;
		visited_D = visited;
	}

	void Graph_D_H::Graph::partition_TD_Parallel_D(int height)
	{

		int heightTarget = height;
		makeCHPerHeight(TreeHeight, cudaThreadNum, heightTarget, nonAdjcentNode_D, ID_hash_D, CHTreeHash_D,
			CHTree_D, NANHash_D,visited_D);

	}

	void Graph_D_H::Graph::partition_TD_Parallel_D_2(int height)
	{

		int heightTarget = height;
		makeCHPerHeight_2(TreeHeight, cudaThreadNum, heightTarget, nonAdjcentNode_D, ID_hash_D, CHTreeHash_D,
			CHTree_D, NANHash_D, visited_D, CHInfo_D);

	}
	void Graph_D_H::Graph::translateInTD()
	{
		CHTreeHash = CHTreeHash_D;
		CHTree = CHTree_D;
		nonAdjcentNode = nonAdjcentNode_D;
		visited = visited_D;
		CHInfo_H = CHInfo_D;
	}
	void Graph::translateCHTree()
	{
		//father = father_D;
		CHTreeHash = CHTreeHash_D;
		CHTree = CHTree_D;
		nonAdjcentNode = nonAdjcentNode_D;
	}


	void Graph::translateOULARMQ()
	{
		firstAppeare_D = firstAppeare;
		RMQ_OneLine_D = RMQ_OneLine;
		//OULA_DFS_D = OULA_DFS_;
	}

	void Graph_D_H::Graph::translateRMQ_noHub()
	{
		firstAppeare_D = firstAppeare;
		H2H_TreeHeight_Hash_D = H2H_TreeHeight_Hash;
		RMQHash_D = RMQHash;
		//RMQ_Height_D = RMQ_Height;
		RMQ_ID_D = RMQ_ID;
	}

	void Graph::translateH2HLabel() {

		H2H_startIndex_D = H2H_startIndex;
		H2H_label_D = H2H_label;
		TreeBFS_D = TreeBFS;
	}


	void Graph::translateH2HLabelBack() {

		H2H_startIndex = H2H_startIndex_D;
		H2H_label = H2H_label_D;
	}

	void Graph_D_H::Graph::translateQuery()
	{
		x_D = x;
		y_D = y;
		result_D = result;
		//translateOULARMQ();
	}

	void Graph_D_H::Graph::translateQueryResultBack()
	{
		result = result_D;
	}

	void Graph_D_H::Graph::makeH2Hlabel_D() {
		thrust::host_vector<int> frontier = {};
		frontier.push_back(head);
		makeH2HLabel(1, cudaThreadNum, H2H_startIndex_D, H2H_label_D, Childs, ChildHash, frontier);
	}


	void Graph_D_H::Graph::makeH2Hlabel_mix() {
		queue<int> frontier;
		for (int i = ChildHash[head].second; i < ChildHash[head].second + ChildHash[head].first; i++)
		{
			frontier.push(Childs[i]);
		}
		int height = 1;
		while (!frontier.empty())
		{
			int size = frontier.size();
			if (size > 128)
				break;
			height++;
			//cout << "height: " << height++ << " size : " << size <<" total: "<<total << endl;
			for (int i = 0; i < size; i++)
			{
				int ID = frontier.front();
				frontier.pop();
				int H2HSize = H2H_startIndex[ID].second - 1;
				int IDNowH2Hindex = H2H_startIndex[ID].first;
				//displaySingalLabel(ID);
				//construct others
				for (int k = IDNowH2Hindex + H2HSize - 1; k >= IDNowH2Hindex; k--)
				{
					if (!H2H_label[k].isTreeNode) {
						continue;
					}
					//int adjH2HStartIndex = H2H_startIndex[H2H_label[k].Node].first;
					int q = k - 1;
					for (; q >= IDNowH2Hindex; q--)
					{
						if (H2H_label[q].isTreeNode) {
							//k = q + 1;
							continue;
						}
						int templength = H2H_label[k].dis + H2H_label[q - IDNowH2Hindex + H2H_startIndex[H2H_label[k].Node].first].dis;
						if (H2H_label[q].dis > templength) {
							H2H_label[q].dis = templength;
							H2H_label[q].Hub = H2H_label[k].Node;
						}
						//H2H_label[q].dis = min(H2H_label[q].dis, H2H_label[k].dis + H2H_label[q - IDNowH2Hindex + H2H_startIndex[H2H_label[k].Node].first].dis);
					}
				}

				for (int j = ChildHash[ID].second; j < ChildHash[ID].second + ChildHash[ID].first; j++)
				{
					frontier.push(Childs[j]);
				}
			}
		}

		thrust::host_vector<int> frontier1 = {};

		while (!frontier.empty()) {
			frontier1.push_back(frontier.front());
			frontier.pop();
		}
		H2H_startIndex_D = H2H_startIndex;
		H2H_label_D = H2H_label;
		makeH2HLabel(height, cudaThreadNum, H2H_startIndex_D, H2H_label_D, Childs, ChildHash, frontier1);
	}

	void Graph_D_H::Graph::makeH2Hlabel_noComm()
	{
		Graph_D_H::time_Mine time;

		//cout << "construct H2H on CPU start " << endl;
		//time.updateStart();

		int tempHeight = 0;
		//for (; tempHeight < TreeHash.size() - 1; tempHeight++)
		//{
		//	int startIndex = TreeHash[tempHeight];
		//	int endIndex = TreeHash[tempHeight + 1];
		//	if (endIndex - startIndex > 256) {
		//		break;
		//	}
		//	int tempThreadNum = min(threadNumber, endIndex - startIndex);

		//	int tempSize = (endIndex - startIndex) / tempThreadNum + 1;
		//	vector<vector<int>> candidateHeap(tempThreadNum, vector<int>());
		//	int it = 0, temp = 0;
		//	for (int i = startIndex; i < endIndex; i++) {
		//		int ID = TreeBFS[i];
		//		candidateHeap[it].emplace_back(ID);
		//		temp++;
		//		if (temp == tempSize) {
		//			it++;
		//			temp = 0;
		//		}
		//	}
		//	std::vector<std::thread> threads;

		//	for (int i = 0; i < tempThreadNum; i++) {

		//		threads.emplace_back(
		//			[this, &candidateHeap, i]() {
		//				for (auto& ID : candidateHeap[i]) {
		//					int H2HSize = H2H_startIndex[ID].second - 1;
		//					int IDNowH2Hindex = H2H_startIndex[ID].first;

		//					for (int k = IDNowH2Hindex + H2HSize - 1; k >= IDNowH2Hindex; k--)
		//					{
		//						if (!H2H_label[k].isTreeNode) {
		//							continue;
		//						}

		//						for (int q = k - 1; q >= IDNowH2Hindex; q--)
		//						{
		//							//if (H2H_label[q].isTreeNode) {
		//							//	continue;
		//							//}
		//							int templength = H2H_label[k].dis + H2H_label[q - IDNowH2Hindex + H2H_startIndex[H2H_label[k].Node].first].dis;
		//							if (H2H_label[q].dis > templength) {
		//								H2H_label[q].dis = templength;
		//								H2H_label[q].Hub = H2H_label[k].Node;
		//							}
		//						}
		//						for (int q = k + 1; q <= IDNowH2Hindex + H2HSize - 1; q++)
		//						{
		//							int templength = H2H_label[k].dis + H2H_label[k - IDNowH2Hindex + H2H_startIndex[H2H_label[q].Node].first].dis;
		//							if (H2H_label[q].dis > templength) {
		//								H2H_label[q].dis = templength;
		//								H2H_label[q].Hub = H2H_label[k].Node;
		//							}
		//						}
		//					}
		//				}
		//			}
		//		);

		//	}

		//	for (auto& its : threads) {
		//		its.join();
		//	}

		//}
		//time.updateEnd();
		//H2HUsingTime_CPU = time.get_microsecond_duration();
		//cout << "\t CPU construct end, using time: " << time.get_microsecond_duration() << endl;


		//cout << "translate H2H start " << endl;
		//time.updateStart();
		//H2H_startIndex_D = H2H_startIndex;
		//H2H_label_D = H2H_label;
		//time.updateEnd();
		//H2HTranslateTime = time.get_microsecond_duration();
		//cout << "\t translate H2H end, using time: " << time.get_microsecond_duration() << endl;


		cout << "H2H construct in GPU start " << endl;
		time.updateStart();
		makeH2HLabel_noCommunication(tempHeight, cudaThreadNum, H2H_startIndex_D, H2H_label_D, TreeBFS_D, TreeHash);
		time.updateEnd();
		H2HUsingTime_GPU = time.get_microsecond_duration();
		cout << "\t translate H2H end, using time: " << time.get_microsecond_duration() << endl;
	}



	void Graph_D_H::Graph::translateBeforeQuery()
	{

		H2H_startIndex_D = H2H_startIndex;
		H2H_label_D = H2H_label;
	}

	void Graph_D_H::Graph::translateH2H_noHub()
	{
		H2H_pos_hash_D = H2H_pos_hash;
		H2H_dis_hash_D = H2H_dis_hash;
		//H2H_pos_ID_D = H2H_pos_ID;
		H2H_pos_POS_D = H2H_pos_POS;
		H2H_dis_D = H2H_dis;
	}

	void Graph_D_H::Graph::translateH2HBack_noHub()
	{
		H2H_dis = H2H_dis_D;
	}



	long long int Graph::H2Hquery_bunch_UsingLCA_D(int size)
	{
		result_D = result;
		return H2HQuery_UsingLCA(x_D, y_D, result_D, firstAppeare_D, RMQ_OneLine_D, H2H_startIndex_D,
			H2H_label_D, RMQ_Size, RMQ_Line_Size, cudaThreadNum, size);
	}

	long long int Graph::H2Hquery_bunch_noLCA_D(int size)
	{
		result_D = result;
		return H2HQuery_NoLCA(x_D,y_D,result_D,H2H_startIndex_D,H2H_label_D,cudaThreadNum,size);
	}



	long long int Graph_D_H::Graph::H2Hquery_bunch_UsingLCA_D_noHub(int size)
	{
		//cout << "\t using before result size: " << result.size() << " result_D size: " << result_D.size() << endl;
		//result_D = result;
		//cout << "\t using after result size: " << result.size() << " result_D size: " << result_D.size() << endl;
		return H2HQuery_UsingLCA_noHub(x_D, y_D, result_D, H2H_pos_hash_D, H2H_dis_hash_D, 
				H2H_pos_POS_D, H2H_dis_D, firstAppeare_D, RMQHash_D, RMQ_ID_D, H2H_TreeHeight_Hash_D, cudaThreadNum, size);

	}


	long long int Graph_D_H::Graph::H2Hquery_bunch_noLCA_D_noHub(int size)
	{
		//cout << "\t no before result size: " << result.size() << " result_D size: " << result_D.size() << endl;
		//result_D = result; 
		//cout << "no after result size: " << result.size() << " result_D size: " << result_D.size() << endl;
		return H2HQuery_noLCA_noHub(x_D, y_D, result_D, H2H_dis_hash_D,
			H2H_dis_D, firstAppeare_D, RMQHash_D, RMQ_ID_D, H2H_TreeHeight_Hash_D, cudaThreadNum, size);

	}




	void Graph_D_H::Graph::makeH2HLabel_noHub_noComm()
	{
		Graph_D_H::time_Mine time;


		std::cout << "construct H2H on CPU multi-thread start " << std::endl;
		time.updateStart();

		for (int64_t tempHeight = 0; tempHeight < changeToGPUHeight; tempHeight++)
		{
			int64_t startIndex = TreeHash[tempHeight];
			int64_t endIndex = TreeHash[tempHeight + 1];

			int64_t tempThreadNum = min((int64_t)threadNumber, endIndex - startIndex);

			int64_t tempSize = (endIndex - startIndex) / tempThreadNum + 1;
			vector<vector<int64_t>> candidateHeap(tempThreadNum, vector<int64_t>());
			int64_t it = 0, temp = 0;
			for (int64_t i = startIndex; i < endIndex; i++) {
				int64_t ID = TreeBFS[i];
				candidateHeap[it].emplace_back(ID);
				temp++;
				if (temp == tempSize) {
					it++;
					temp = 0;
				}
			}
			std::vector<std::thread> threads;

			for (int i = 0; i < tempThreadNum; i++) {

				threads.emplace_back(
					[this, &candidateHeap, i]() {
						for (auto& nodeID : candidateHeap[i]) {
							for (int64_t temp = H2H_pos_hash[(int64_t)nodeID + 1] - 2; temp >= H2H_pos_hash[(int64_t)nodeID]; temp--) {
								int adjID = H2H_pos_ID[temp];
								int pos = H2H_pos_POS[temp];
								int64_t indexNode = H2H_dis_hash[nodeID];
								int64_t index1_adj = H2H_dis_hash[adjID];
								int tempLength_ = H2H_dis[indexNode + (int64_t)pos];
								for (int j = pos - 1; j >= 0; j--) {
									H2H_dis[indexNode + (int64_t)j] = std::min(H2H_dis[indexNode + (int64_t)j],
										tempLength_ + H2H_dis[index1_adj + (int64_t)j]);
								}
								// L4 label
								int64_t fatherID = nodeID;
								for (int j = (int)(H2H_dis_hash[nodeID + 1] - H2H_dis_hash[nodeID]) - 2; j > pos; j--) {
									fatherID = (int64_t)father[fatherID];

									H2H_dis[indexNode + (int64_t)j] = std::min(H2H_dis[indexNode + (int64_t)j],
										tempLength_ + H2H_dis[H2H_dis_hash[fatherID] + (int64_t)pos]);
								}
							}
						}
					}
				);

			}

			for (auto& its : threads) {
				its.join();
			}

		}
		time.updateEnd();
		H2HUsingTime_CPU = time.get_microsecond_duration();
		std::cout << "\t CPU construct end, using time: " << time.get_microsecond_duration() << std::endl;

		cout << "translate H2H start " << endl;
		time.updateStart();
		H2H_pos_hash_D = H2H_pos_hash;
		H2H_dis_hash_D = H2H_dis_hash;
		H2H_pos_ID_D = H2H_pos_ID;
		H2H_pos_POS_D = H2H_pos_POS;
		H2H_dis_D = H2H_dis;
		time.updateEnd();
		H2HTranslateTime = time.get_microsecond_duration();
		cout << "\t translate H2H end, using time: " << time.get_microsecond_duration() << endl;

		cout << "H2H construct in GPU start " << endl;
		time.updateStart();


		makeH2HLabel_noCommunication_noHub_2(cudaThreadNum, H2H_pos_hash_D, H2H_dis_hash_D, H2H_pos_ID_D, H2H_pos_POS_D,
			H2H_dis_D, father_D, frontier_Hash_D, frontier_ID_D, frontier_Hash.size() - 1);
		time.updateEnd();
		H2HUsingTime_GPU = time.get_microsecond_duration();
		cout << "\t construct H2H end, using time: " << time.get_microsecond_duration() << endl;
	}


	void Graph_D_H::Graph::makeH2HLabel_noHub_noComm_2()
	{
		Graph_D_H::time_Mine time;

		int tempHeight11111 = 2;
		std::cout << "construct H2H on CPU multi-thread start " << std::endl;
		time.updateStart();

		for (int64_t tempHeight = 0; tempHeight < tempHeight11111; tempHeight++)
		{
			int64_t startIndex = TreeHash[tempHeight];
			int64_t endIndex = TreeHash[tempHeight + 1];

			int64_t tempThreadNum = min((int64_t)threadNumber, endIndex - startIndex);

			int64_t tempSize = (endIndex - startIndex) / tempThreadNum + 1;
			vector<vector<int64_t>> candidateHeap(tempThreadNum, vector<int64_t>());
			int64_t it = 0, temp = 0;
			for (int64_t i = startIndex; i < endIndex; i++) {
				int64_t ID = TreeBFS[i];
				candidateHeap[it].emplace_back(ID);
				temp++;
				if (temp == tempSize) {
					it++;
					temp = 0;
				}
			}
			std::vector<std::thread> threads;

			for (int i = 0; i < tempThreadNum; i++) {

				threads.emplace_back(
					[this, &candidateHeap, i]() {
						for (auto& nodeID : candidateHeap[i]) {
							for (int64_t temp = H2H_pos_hash[(int64_t)nodeID + 1] - 2; temp >= H2H_pos_hash[(int64_t)nodeID]; temp--) {
								int adjID = H2H_pos_ID[temp];
								int pos = H2H_pos_POS[temp];
								int64_t indexNode = H2H_dis_hash[nodeID];
								int64_t index1_adj = H2H_dis_hash[adjID];
								int tempLength_ = H2H_dis[indexNode + (int64_t)pos];
								for (int j = pos - 1; j >= 0; j--) {
									H2H_dis[indexNode + (int64_t)j] = std::min(H2H_dis[indexNode + (int64_t)j],
										tempLength_ + H2H_dis[index1_adj + (int64_t)j]);
								}
								// L4 label
								//int64_t fatherID = nodeID;
								//for (int j = (int)(H2H_dis_hash[nodeID + 1] - H2H_dis_hash[nodeID]) - 2; j > pos; j--) {
								//	fatherID = (int64_t)father[fatherID];

								//	H2H_dis[indexNode + (int64_t)j] = std::min(H2H_dis[indexNode + (int64_t)j],
								//		tempLength_ + H2H_dis[H2H_dis_hash[fatherID] + (int64_t)pos]);
								//}
							}
						}
					}
				);

			}

			for (auto& its : threads) {
				its.join();
			}

		}
		time.updateEnd();
		H2HUsingTime_CPU = time.get_microsecond_duration();
		std::cout << "\t CPU construct end, using time: " << time.get_microsecond_duration() << std::endl;

		cout << "translate H2H start " << endl;
		time.updateStart();
		H2H_pos_hash_D = H2H_pos_hash;
		H2H_dis_hash_D = H2H_dis_hash;
		H2H_pos_ID_D = H2H_pos_ID;
		H2H_pos_POS_D = H2H_pos_POS;
		H2H_dis_D = H2H_dis;
		time.updateEnd();
		H2HTranslateTime = time.get_microsecond_duration();
		cout << "\t translate H2H end, using time: " << time.get_microsecond_duration() << endl;

		cout << "H2H construct in GPU start " << endl;
		H2HUsingTime_GPU = makeH2HLabel_noCommunication_noHub(tempHeight11111, cudaThreadNum, H2H_pos_hash_D, H2H_dis_hash_D, H2H_pos_ID_D, H2H_pos_POS_D,
			H2H_dis_D, father_D, TreeBFS_D, TreeHash);
		//H2HUsingTime_GPU = time.get_microsecond_duration();
		cout << "\t construct H2H end, using time: " << H2HUsingTime_GPU << endl;
	}

	void Graph_D_H::Graph::makeH2HLabel_noHub_noComm_3()
	{
		Graph_D_H::time_Mine time;

		int tempHeight11111 = 2;
		std::cout << "construct H2H on CPU multi-thread start " << std::endl;
		time.updateStart();

		for (int64_t tempHeight = 0; tempHeight < tempHeight11111; tempHeight++)
		{
			int64_t startIndex = TreeHash[tempHeight];
			int64_t endIndex = TreeHash[tempHeight + 1];

			int64_t tempThreadNum = min((int64_t)threadNumber, endIndex - startIndex);

			int64_t tempSize = (endIndex - startIndex) / tempThreadNum + 1;
			vector<vector<int64_t>> candidateHeap(tempThreadNum, vector<int64_t>());
			int64_t it = 0, temp = 0;
			for (int64_t i = startIndex; i < endIndex; i++) {
				int64_t ID = TreeBFS[i];
				candidateHeap[it].emplace_back(ID);
				temp++;
				if (temp == tempSize) {
					it++;
					temp = 0;
				}
			}
			std::vector<std::thread> threads;

			for (int i = 0; i < tempThreadNum; i++) {

				threads.emplace_back(
					[this, &candidateHeap, i]() {
						for (auto& nodeID : candidateHeap[i]) {
							for (int64_t temp = H2H_pos_hash[(int64_t)nodeID + 1] - 2; temp >= H2H_pos_hash[(int64_t)nodeID]; temp--) {
								int adjID = H2H_pos_ID[temp];
								int pos = H2H_pos_POS[temp];
								int64_t indexNode = H2H_dis_hash[nodeID];
								int64_t index1_adj = H2H_dis_hash[adjID];
								int tempLength_ = H2H_dis[indexNode + (int64_t)pos];
								for (int j = pos - 1; j >= 0; j--) {
									H2H_dis[indexNode + (int64_t)j] = std::min(H2H_dis[indexNode + (int64_t)j],
										tempLength_ + H2H_dis[index1_adj + (int64_t)j]);
								}
								// L4 label
								//int64_t fatherID = nodeID;
								//for (int j = (int)(H2H_dis_hash[nodeID + 1] - H2H_dis_hash[nodeID]) - 2; j > pos; j--) {
								//	fatherID = (int64_t)father[fatherID];

								//	H2H_dis[indexNode + (int64_t)j] = std::min(H2H_dis[indexNode + (int64_t)j],
								//		tempLength_ + H2H_dis[H2H_dis_hash[fatherID] + (int64_t)pos]);
								//}
							}
						}
					}
				);

			}

			for (auto& its : threads) {
				its.join();
			}

		}
		time.updateEnd();
		H2HUsingTime_CPU = time.get_microsecond_duration();
		std::cout << "\t CPU construct end, using time: " << time.get_microsecond_duration() << std::endl;

		cout << "translate H2H start " << endl;
		time.updateStart();

		H2H_dis_hash_D = H2H_dis_hash;
		H2H_dis_D = H2H_dis;

		TreeBFS_ID_D = TreeBFS_ID;
		TreeBFS_adj_D = TreeBFS_adj;
		TreeBFS_pos_D = TreeBFS_pos;
		//TreeBFS_changeTime_D = TreeBFS_changeTime;
		time.updateEnd();
		H2HTranslateTime = time.get_microsecond_duration();
		cout << "\t translate H2H end, using time: " << time.get_microsecond_duration() << endl;

		cout << "H2H construct in GPU start " << endl;
		H2HUsingTime_GPU = makeH2HLabel_noCommunication_noHub_3(tempHeight11111, cudaThreadNum, H2H_dis_hash_D, H2H_dis_D, 
			TreeBFS_ID_D, TreeBFS_adj_D, TreeBFS_pos_D, //TreeBFS_changeTime_D,
			father_D, TreeBFS_Hash);
		//H2HUsingTime_GPU = time.get_microsecond_duration();
		cout << "\t construct H2H end, using time: " << H2HUsingTime_GPU << endl;

		TreeBFS_ID.clear();
		TreeBFS_ID.shrink_to_fit();
		TreeBFS_ID_D.clear();
		TreeBFS_ID_D.shrink_to_fit();
		TreeBFS_adj.clear();
		TreeBFS_adj.shrink_to_fit();
		TreeBFS_adj_D.clear();
		TreeBFS_adj_D.shrink_to_fit();
		TreeBFS_pos.clear();
		TreeBFS_pos.shrink_to_fit();
		TreeBFS_pos_D.clear();
		TreeBFS_pos_D.shrink_to_fit();


		H2H_pos_ID_D = H2H_pos_ID;
		H2H_pos_POS_D = H2H_pos_POS;
		H2H_pos_hash_D = H2H_pos_hash;
	}

}