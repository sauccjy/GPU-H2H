#include "hip/hip_runtime.h"
#pragma once
#include"Graph_D.cuh"

Graph_D_H::Graph::Graph(const string graphName , int TreeHeight, const string Nodefile, const string EdgeFile, int threadPoolSize, int changeHeight, string PartitionCutFile, string PartitionIDFile)
{
	this->graphName = graphName;
	this->NodeFile = Nodefile;
	this->EdgeFile = EdgeFile;
	this->changeHeight = changeHeight;
	mainTreeHeight = TreeHeight;
	//this->TreeHeight = TreeHeight;
	this->partitionCut = PartitionCutFile;
	this->partitionID = PartitionIDFile;

	threadNumber = threadPoolSize;
	time_Mine time;

	cout << "graph node start loading" << endl;
	time.updateStart();
	//load Node
	fstream partitionNode(PartitionIDFile, ios::in | ios::out);
	partitionNode >> NodeNumber;
	NE_P.assign(NodeNumber, myPair<double>());
	ID_hash.assign(NodeNumber, -1);
	int Hash = -1, ID = -1;
	double la = 0, gr = 0;
	for (int i = 0; i < NodeNumber; i++) {
		partitionNode >> Hash >> la >> gr >> ID;
		NE_P[Hash].setPairs(ID, la, gr);
		ID_hash[ID] = Hash;
	}
	partitionNode.close();
	time.updateEnd();
	cout << "\t graph node end and using time:" << time.get_microsecond_duration() << endl;

	cout << "graph edge start loading" << endl;
	time.updateStart();
	fstream Edge(EdgeFile, ios::in | ios::out);
	Edge >>NodeNumber>> EdgeNumber;
	int NodeID1 = 0, NodeID2 = 0, EdgeWeight = 0;
	char s;
	//int tempNodeID = 0;
	int actualEdgeNumber = EdgeNumber;
	adjList.assign( NodeNumber , thrust::host_vector<pairs>() );

	for (int i = 0; i < EdgeNumber; i++)
	{
		Edge >>s >> NodeID1 >> NodeID2 >> EdgeWeight;
		if (NodeID1 == NodeID2)
		{
			actualEdgeNumber--;
			continue;
		}
		adjList[NodeID1 - 1].push_back(pairs(NodeID2 - 1, EdgeWeight));
	}
	EdgeNumber = actualEdgeNumber;
	Edge.close();
	time.updateEnd();
	cout << "\t graph edge finished and using time:" << time.get_microsecond_duration() << endl;

	cout << "Partition Cut loading" << endl;
	time.updateStart();

	fstream Partition1(PartitionCutFile, ios::in | ios::out);
	int maxHeight;
	Partition1 >> maxHeight;
	maxHeight = min(maxHeight, mainTreeHeight);
	mainTreeHeight = maxHeight;
	this->TreeHeight = mainTreeHeight;
	long long int fullPar = std::pow(2, maxHeight) - 1;

	partition_Tree.assign(fullPar, pairs());
	int h = 0, left = 0, right = 0;
	for (long long int i = 0; i < fullPar; i++) {
		Partition1 >> h >> left >> right;
		partition_Tree[i].pairsReset(left, right);
	}
	Partition1.close();
	time.updateEnd();
	cout<<"\t Partition Cut loading end and using time: "<< time.get_microsecond_duration() << endl;

	//displayPartition();

	
	CHInfo_H.assign(NodeNumber, pairs(-1,0));
	//Graph_D_H::CHInfo.assign(NodeNumber, pairs(0, 0));
	
	for (int i = 0; i < NodeNumber; i++)
	{
		CHInfo_H[i].first = adjList[i].size();
	}
	//CHInfo_D = CHInfo_H;
	visited.assign(NodeNumber, false);
	CHAdjlist.assign(NodeNumber, unordered_map<int, pairs>());
}


Graph_D_H::Graph::~Graph()
{
}

void Graph_D_H::Graph::checklink()
{
	vector<bool> visit(NodeNumber,false);
	priority_queue<int> que;
	int size = 0;
	que.push(0);
	visit[0] = true;
	while (!que.empty()) {
		int ID = que.top();
		que.pop();
		size++;
		for (int i = 0; i < adjList[ID].size(); i++)
		{
			if (visit[adjList[ID][i].first])
				continue;
			que.push(adjList[ID][i].first);
			visit[adjList[ID][i].first] = true;
		}

	}
	if (size == NodeNumber) {
		cout << "connect!"<<endl;
	}
	else {
		cout << "disconnect!"<<endl;
	}
}



void Graph_D_H::Graph::printConstructInfo()
{
	string constructData = "./ConstructInfo/"+graphName+"/";
	if(ContractM == 1 && ConstructMethod == 1){
		constructData += "CPU-CPU.csv";
	}
	if(ContractM == 2 && ConstructMethod == 1){
		constructData += "GPU-CPU.csv";
	}
	if(ContractM == 1 && ConstructMethod == 2){
		constructData += "CPU-GPU.csv";
	}
	if(ContractM == 2 && ConstructMethod == 2){
		constructData += "GPU-GPU.csv";
	}
	// if (ConstructMethod == 1) {
	// 	constructData += "MultiThread.csv";
	// }
	// else {
	// 	constructData += "GPUHybrid.csv";
	// }

	//string constructData = "constructInfo.csv";
	std::ofstream outputFile(constructData, std::ios::app);
	if (outputFile.is_open()) {

		outputFile << "graphName,PartitionTreeHeight,changeHeight,ParititionTime,makePartitionRankTreeTime,AlllocateLUBTime,LUBSize,translateLUBToGPUTime,TDInGPUTime,translateBackAndMallocToCPUTime,vertexesConstructedInGPU,DecompositionTreeSize,TDInCPUTime,TDTREEHeight,,RMQsize,translateRMQTime,H2HLabelSize,,changeConstructDeviceSize,H2HmallocTime,H2HConstructBFSTreeTime,H2HTranslateBFSTreeTime,H2HUsingTime_CPU,H2HTranslateTime,H2HUsingTime_GPU,PartitionMethod,PartitionMethod\n";

		outputFile << graphName << "," << TreeHeight << "," << changeHeight << "," << partitionTime 
			<< "," << makePartitionRankTreeTime << "," << AllocateLUBTime << ","
			<< LUBSize << "," << translateLUBToGPUTime << "," << TDInGPUTime << "," 
			<< translateBackAndMallocToCPUTime << "," 
			<< vertexesConstructedInGPU << "," << DecompositionTreeSize 
			<< "," << TDInCPUTime << "," << TDTREEHeight << ",," << RMQsize << "," 
			<< translateRMQTime << "," << H2HLabelSize << ",," << changeConstructDeviceSize << "," 
			<< H2HmallocTime << "," << H2HConstructBFSTreeTime << "," << H2HTranslateBFSTreeTime 
			<< "," << H2HUsingTime_CPU << "," << H2HTranslateTime 
			<< "," << H2HUsingTime_GPU <<","<< PartitionMethod << "\n";


		outputFile.close();
		//std::cout << "Data has been written to output.csv\n";
	}
	else {
		std::cerr << "Unable to open file for writing.\n";
	}
}


