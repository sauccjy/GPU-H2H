#include "hip/hip_runtime.h"
#pragma once
#include"Graph_D.cuh"

namespace Graph_D_H
{
	void Graph_D_H::Graph::displayAdjList()
	{
		for (int i = 0; i < adjList.size(); i++){
			cout << "Node ID : " << i << " ";
			for (int j = 0; j < adjList[i].size(); j++)
				cout << "<" << adjList[i][j].first << "," << adjList[i][j].second << "> ";
			cout << endl;
		}
	}

	void Graph_D_H::Graph::displayGraph(std::map<int, std::map<int, int>>& standard_graph_in_partition)
	{
		cout << "\tsubGraph" << endl;
		for (auto& i : standard_graph_in_partition) {
			cout << "\tvertex id: " << i.first<<" and adjscent: ";
			for (auto& j : standard_graph_in_partition.at(i.first)) {
				cout << j.first << ",";
			}
			cout << endl;
		}

	}

	void Graph_D_H::Graph::displayPartition()
	{
		cout << "______________________PARTITION________________________" << endl;
		int tempHeight = 1;
		while (tempHeight <= mainTreeHeight)
		{
			cout << "height:" << tempHeight << ";  ";
			int lowestIndexStart = std::pow(2, tempHeight - 1) - 1;
			int lowestIndexEnd = std::pow(2, tempHeight) - 2;
			tempHeight++;
			for (int i = lowestIndexStart; i <= lowestIndexEnd; i++)
			{
				int left = partition_Tree[i].first;
				int right = partition_Tree[i].second;
				cout << "(";
				for (int j = left; j < right - 1; j++)
				{
					cout << NE_P[j].NodeID << ",";
				}
				cout << NE_P[right - 1].NodeID << ")";
			}
			cout << endl;
		}
		cout << "______________________ID_HASH________________________" << endl;
		for (int i = 0; i < NodeNumber; i++)
		{
			cout << "ID:" << i << " hash:" << ID_hash[i] << "; ";
		}
		cout << endl;
	}

	void Graph_D_H::Graph::displayOrderAndCHTree()
	{
		cout << "______________________ORDER_CHTREE________________________" << endl;
		int tempHeight = TreeHeight;
		while (tempHeight > 0)
		{
			cout << "height:" << tempHeight << ";  "<<endl;
			int lowestIndexStart = std::pow(2, tempHeight - 1) - 1;
			int lowestIndexEnd = std::pow(2, tempHeight) - 2;
			tempHeight--;
			for (int i = lowestIndexStart; i <= lowestIndexEnd; i++)
			{
				int nonAdjcentNode_index = NANHash[i].first;
				int nonAdjcentNode_index_end = nonAdjcentNode_index + NANHash[i].NodeID;
				cout << "start index: " << nonAdjcentNode_index << "; partition Node Size: " << nonAdjcentNode_index_end <<
					"; partition node max size: " << NANHash[i].second<< "; nonadjcent Node: ";
				for (int j = nonAdjcentNode_index; j < nonAdjcentNode_index_end; j++)
				{
					cout <<"<"<< nonAdjcentNode[j].first << "," << nonAdjcentNode[j].second << "> ";
				}
				cout << endl;
			}
			cout << endl;
		}
		cout << "______________________Rank________________________" << endl;
		for (int i = 0; i < nonAdjcentNode.size(); i++)
		{
			cout << nonAdjcentNode[i].second << " ";
		}
		cout << endl;

		cout << "______________________ID_HASH________________________" << endl;
		for (int i = 0; i < NodeNumber; i++)
		{
			cout<<"ID:"<<i<<" hash:" << ID_hash[i] << "; ";
		}
		cout << endl;

		cout << "______________________Tree_OLD________________________" << endl;
		for (int i = 0; i < CHTreeHash.size(); i++)
		{
			cout << "NodeID: " << NE_P[ID_hash[i]].NodeID << ": actualSize: "<<CHTreeHash[ID_hash[i]].first<<
				" maxSize: "<< CHTreeHash[ID_hash[i]].second<<endl;
			for (int j = CHTreeHash[ID_hash[i]].NodeID; j < CHTreeHash[ID_hash[i]].NodeID + CHTreeHash[ID_hash[i]].second; j++)
			{
				if (CHTree[j].first == INT_MAX)
					continue;
				cout << "(" << CHTree[j].first << "," << CHTree[j].second << "," << CHTree[j].NodeID << ") ";
			}
			cout << endl;
		}


	}

	void Graph_D_H::Graph::displayCHTree()
	{
		cout << "______________________Tree_AFTER_CH________________________" << endl;
		for (int i = 0; i < CHTreeHash.size(); i++)
		{
			cout << "NodeID: " << NE_P[ID_hash[i]].NodeID << ": actualSize: " << CHTreeHash[ID_hash[i]].first <<
				" maxSize: " << CHTreeHash[ID_hash[i]].second << endl;

			for (int j = CHTreeHash[ID_hash[i]].NodeID; j < CHTreeHash[ID_hash[i]].NodeID + CHTreeHash[ID_hash[i]].second; j++)
			{
				if (CHTree[j].first == INT_MAX)
					continue;
				cout << "(" << CHTree[j].first << "," << CHTree[j].second << "," << CHTree[j].NodeID << ") ";
			}
			cout << endl;
		}
	}

	void Graph_D_H::Graph::displayOULARMQ()
	{
		cout << "____________OULA Sequence_____________" << endl;
		for (auto i = 0; i < OULA_DFS_.size(); i++)
		{
			cout << " <" << OULA_DFS_[i].first << "," << OULA_DFS_[i].second << ">";
		}
		cout << endl;
		cout << "_____________FirstAppare______________" << endl;
		for (auto i = 0; i < firstAppeare.size(); i++)
		{
			cout << " <" << i << "," << firstAppeare[i] << ">";

		}

		cout << endl;
		cout << "______________RMQ_____________________" << endl;
		for (int j = 0; j < RMQ_Line_Size; j++)
		{
			cout << "j = " << j << " jump (1<<j)-1" << endl;
			for (int i = 0; i < RMQ_Size; i++)
			{
				if (RMQ_OneLine[i * RMQ_Line_Size + j].first != INT_MAX)
					cout << " <" << i << "," << RMQ_OneLine[i * RMQ_Line_Size + j].first << "," << RMQ_OneLine[i * RMQ_Line_Size + j].second << "> ";
			}
			cout << endl;
		}


	}

	void Graph_D_H::Graph::displayH2HLabel()
	{
		cout << "________________________H2H_CSR_LABEL__________________________" << endl;
		for (int i = 0; i < NodeNumber; i++)
		{
			cout << "NodeID: " << i << endl;
			int pos = H2H_startIndex[i].first;
			int size = H2H_startIndex[i].second;
			cout << "Node: ";
			for (int j = 0; j < size; j++)
			{
				cout << H2H_label[pos + j].Node << " ";
			}
			cout << endl;
			cout << "isTreeNode: ";
			for (int j = 0; j < size; j++)
			{
				cout << H2H_label[pos + j].isTreeNode << " ";
			}
			cout << endl;
			cout << "Hub: ";
			for (int j = 0; j < size; j++)
			{
				cout << H2H_label[pos + j].Hub << " ";
			}
			cout << endl;
			cout << "dis: ";
			for (int j = 0; j < size; j++)
			{
				cout << H2H_label[pos + j].dis << " ";
			}
			cout << endl;
			cout << "-------------------------------" << endl;
		}
	}

	void Graph_D_H::Graph::displaySingalLabel(int i)
	{
		cout << "NodeID: " << i << endl;
		int pos = H2H_startIndex[i].first;
		int size = H2H_startIndex[i].second;
		cout << "Node: ";
		for (int j = 0; j < size; j++)
		{
			cout << H2H_label[pos + j].Node << " ";
		}
		cout << endl;
		cout << "isTreeNode: ";
		for (int j = 0; j < size; j++)
		{
			cout << H2H_label[pos + j].isTreeNode << " ";
		}
		cout << endl;
		cout << "Hub: ";
		for (int j = 0; j < size; j++)
		{
			cout << H2H_label[pos + j].Hub << " ";
		}
		cout << endl;
		cout << "dis: ";
		for (int j = 0; j < size; j++)
		{
			cout << H2H_label[pos + j].dis << " ";
		}
		cout << endl;
		cout << "-------------------------------" << endl;
	}


	void Graph::displayCHT()
	{
		for(int j = 1;j < 10 ;j++)
		{
			int i = nonAdjcentNode[nonAdjcentNode.size() - j].second;
			cout << "NodeID: " << NE_P[ID_hash[i]].NodeID << ": actualSize: " << CHTreeHash[ID_hash[i]].first <<
				" maxSize: " << CHTreeHash[ID_hash[i]].second << endl;

			for (int j = CHTreeHash[ID_hash[i]].NodeID; j < CHTreeHash[ID_hash[i]].NodeID + CHTreeHash[ID_hash[i]].first; j++)
			{
				if (CHTree[j].first == INT_MAX)
					continue;
				cout << "(" << CHTree[j].first << "," << CHTree[j].second << "," << CHTree[j].NodeID << ") ";
			}
			cout << endl;
		}
	}

	void Graph::displayCHTree_mapVersion() {
		for (int j = 1; j <= 13; j++)
		{
			int i = nonAdjcentNode[nonAdjcentNode.size() - j].second;
			cout << "NodeID: " << NE_P[ID_hash[i]].NodeID << endl;

			for (auto& it : CHAdjlist[i]) {
				cout << "(" << it.first << "," << it.second.first << "," << it.second.second << ") ";
			}
			cout << endl;
		}
	}

	void Graph_D_H::Graph::displayCHAdjList()
	{
		cout << "_________________CHTree_____________________" << endl;
		for (int i = 0; i < CHAdjlist.size(); i++) {
			cout << "\tNodeID: " << i << " : ";
			for (auto& it : CHAdjlist[i]) {
				cout << "<" << it.first << "," << it.second.first << "," << it.second.second << ">";
			}
			cout << endl;
		}
	}

	void Graph_D_H::Graph::displayH2H_noHub()
	{
		cout << "________________________H2H label no Hub_____________" << endl;

		cout << "pos hash (id, index): ";
		for (int i = 0; i < H2H_pos_hash.size(); i++) {
			cout << "(" << i << "," << H2H_pos_hash[i] << ")";
		}
		cout << endl;

		cout << "pos ID and POS (index, id, pos): ";
		for (int i = 0; i < H2H_pos_ID.size(); i++) {
			cout << "(" << i << "," << H2H_pos_ID[i] << "," << H2H_pos_POS[i]<<")";
		}
		cout << endl;

		cout << "dis hash (id, index): ";
		for (int i = 0; i < H2H_dis_hash.size(); i++) {
			cout << "(" << i << "," << H2H_dis_hash[i] << ")";
		}
		cout << endl;

		cout << "dis (index, dis): ";
		for (int i = 0; i < H2H_dis.size(); i++) {
			cout << "(" << i << "," << H2H_dis[i] << ")";
		}
		cout << endl;


		cout << endl;
		cout << "____________H2H label no hub nodeID version______" << endl;
		for (int i = 0; i < NodeNumber; i++) {
			cout << "nodeID: " << i << endl;
			cout << "pos: "<<endl;
			cout << "\tID: ";
			for (int64_t j = H2H_pos_hash[i]; j < (int64_t)H2H_pos_hash[i + 1]; j++) {
				cout << H2H_pos_ID[j] << " ";
			}
			cout << endl;
			cout << "\tPOS: ";
			for (int64_t j = H2H_pos_hash[i]; j < (int64_t)H2H_pos_hash[i + 1]; j++) {
				cout << H2H_pos_POS[j] << " ";
			}
			cout << endl;
			cout << "dis: " << endl;
			cout << "\t";
			for (int64_t j = H2H_dis_hash[i]; j < (int64_t)H2H_dis_hash[i + 1]; j++) {
				cout << H2H_dis[j] << " ";
			}
			cout << endl;
			cout << endl;
		}
	}
}