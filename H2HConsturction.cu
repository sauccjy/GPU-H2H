#include "hip/hip_runtime.h"
#pragma once
#include"Graph_D.cuh"

namespace Graph_D_H
{


	void Graph_D_H::Graph::inConstructH2H_MultiThread()
	{
		Graph_D_H::time_Mine time;
		std::cout << "malloc H2H label start " << std::endl;
		time.updateStart();
		mallocH2HLabel();
		time.updateEnd();
		H2HmallocTime = time.get_microsecond_duration();
		std::cout << "\t malloc end, using time: " << time.get_microsecond_duration() << std::endl;

		std::cout << "construct bfs tree start " << endl;
		time.updateStart();
		constructBFSTree();
		time.updateEnd();
		H2HConstructBFSTreeTime = time.get_microsecond_duration();
		std::cout << "\t bfs construct end, using time: " << time.get_microsecond_duration() << std::endl;

		makeH2Hlabel_MultiThread();

	}
	void Graph_D_H::Graph::inConstructH2H_D()
	{
		mallocH2HLabel();
		constructBFSTree();
		translateH2HLabel();
		//displayH2HLabel();
		makeH2Hlabel_D();
		translateH2HLabelBack();
		//displayH2HLabel();
	}

	void Graph_D_H::Graph::inConstructH2H_mix()
	{
		mallocH2HLabel();
		makeH2Hlabel_mix();
		translateH2HLabelBack();
	}

	void Graph_D_H::Graph::inConstructH2H_noComm()
	{
		Graph_D_H::time_Mine time;
		std::cout << "malloc H2H label start " << std::endl;
		time.updateStart();
		mallocH2HLabel();
		time.updateEnd();
		H2HmallocTime = time.get_microsecond_duration();
		std::cout << "\t malloc end, using time: " << time.get_microsecond_duration() << std::endl;

		std::cout << "construct bfs tree start " << std::endl;
		time.updateStart();
		constructBFSTree();
		time.updateEnd();
		H2HConstructBFSTreeTime = time.get_microsecond_duration();
		std::cout << "\t bfs construct end, using time: " << time.get_microsecond_duration() << std::endl;

		std::cout << "translate bfs tree start " << std::endl;
		time.updateStart();
		TreeBFS_D = TreeBFS;
		time.updateEnd();
		H2HTranslateBFSTreeTime = time.get_microsecond_duration();
		std::cout << "\t bfs translate end, using time: " << time.get_microsecond_duration() << std::endl;


		makeH2Hlabel_noComm();


		translateH2HLabelBack();
	}

	void Graph_D_H::Graph::cleanAfterTreeStruct() {
	}

	void Graph_D_H::Graph::cleanBeforeQuery()
	{
		TreeBFS.clear();
		TreeBFS.shrink_to_fit();
		TreeBFS_D.clear();
		TreeBFS_D.shrink_to_fit();
		TreeHash.clear();
		TreeHash.shrink_to_fit();
		father.clear();
		father.shrink_to_fit();
		father_D.clear();
		father_D.shrink_to_fit();
	}

	void Graph_D_H::Graph::mallocH2HLabel() {

		H2H_startIndex.assign(NodeNumber, pairs(-1, 0));//first: label start index , second :label size
		H2H_label.assign(H2H_Size, H2HLabel());
		int temp = 0;
		queue<int > heap;
		heap.push(head);
		if (oulaOnly) {
			while (!heap.empty())
			{
				int nodeID = heap.front();
				heap.pop();
				H2H_startIndex[nodeID].second = OULA_Only[nodeID];
				H2H_startIndex[nodeID].first = temp;
				temp += H2H_startIndex[nodeID].second;
				H2H_label[temp - 1].Node = nodeID;
				H2H_label[temp - 1].isTreeNode = true;
				H2H_label[temp - 1].Hub = -1;
				H2H_label[temp - 1].dis = 0;
				if (nodeID != head)
				{
					for (int i = 0; i < H2H_startIndex[father[nodeID]].second; i++)
					{
						int sameID = H2H_label[i + H2H_startIndex[father[nodeID]].first].Node;
						H2H_label[i + H2H_startIndex[nodeID].first].Node = sameID;

						if (CHAdjlist[nodeID].find(sameID) != CHAdjlist[nodeID].end()) {
							H2H_label[i + H2H_startIndex[nodeID].first].isTreeNode = true;
							H2H_label[i + H2H_startIndex[nodeID].first].dis = CHAdjlist[nodeID][sameID].first;
							H2H_label[i + H2H_startIndex[nodeID].first].Hub = CHAdjlist[nodeID][sameID].second;
						}
					}
				}


				for (int i = ChildHash[nodeID].second; i < ChildHash[nodeID].second + ChildHash[nodeID].first; i++)
				{
					heap.push(Childs[i]);
				}
			}


		}
		else {
			while (!heap.empty())
			{
				int nodeID = heap.front();
				heap.pop();
				H2H_startIndex[nodeID].second = OULA_DFS_[firstAppeare[nodeID]].second;
				H2H_startIndex[nodeID].first = temp;
				temp += H2H_startIndex[nodeID].second;
				H2H_label[temp - 1].Node = nodeID;
				H2H_label[temp - 1].isTreeNode = true;
				H2H_label[temp - 1].Hub = -1;
				H2H_label[temp - 1].dis = 0;
				if (nodeID != head)
				{
					for (int i = 0; i < H2H_startIndex[father[nodeID]].second; i++)
					{
						int sameID = H2H_label[i + H2H_startIndex[father[nodeID]].first].Node;
						H2H_label[i + H2H_startIndex[nodeID].first].Node = sameID;

						if (CHAdjlist[nodeID].find(sameID) != CHAdjlist[nodeID].end()) {
							H2H_label[i + H2H_startIndex[nodeID].first].isTreeNode = true;
							H2H_label[i + H2H_startIndex[nodeID].first].dis = CHAdjlist[nodeID][sameID].first;
							H2H_label[i + H2H_startIndex[nodeID].first].Hub = CHAdjlist[nodeID][sameID].second;
						}
					}
				}
				for (int i = ChildHash[nodeID].second; i < ChildHash[nodeID].second + ChildHash[nodeID].first; i++)
				{
					heap.push(Childs[i]);
				}
			}

		}

		for (auto& it : CHAdjlist)
		{
			it.clear();
		}
		CHAdjlist.clear();

		H2H_startIndex_D = H2H_startIndex;
		H2H_label_D = H2H_label;
	}

	int Graph::findLCA(int x, int y)
	{
		if (x == y)
			return x;
		int firstA = firstAppeare[x];//position  in  OULA_DFS
		int firstB = firstAppeare[y];
		if (firstA > firstB)
			swap(firstA, firstB);
		int k = (int)(log2(firstB - firstA + 1));//k means log2(gap);
		//int k = (int)(log2(abs(firstA - firstB) + 1));
		// min(RMQ[firstA][k], RMQ[firstB - (1 << k) + 1][k]);//(1<<k - 1 ) means davicate index;
		if (RMQ_OneLine[firstA * RMQ_Line_Size + k].first <= RMQ_OneLine[(firstB - (1 << k) + 1) * RMQ_Line_Size + k].first)
			return RMQ_OneLine[firstA * RMQ_Line_Size + k].second;
		else
			return RMQ_OneLine[(firstB - (1 << k) + 1) * RMQ_Line_Size + k].second;
	}

	int Graph_D_H::Graph::findLCA_noHub(int x, int y)
	{
		if (x == y)
			return x;
		int firstA = firstAppeare[x];//position  in  OULA_DFS
		int firstB = firstAppeare[y];
		if (firstA > firstB)
			swap(firstA, firstB);
		int k = (int)(log2(firstB - firstA + 1));//k means log2(gap);

		//int64_t firstARMQPlace = RMQHash[firstA] + k;
		//int64_t firstBRMQPlace = RMQHash[firstB - (1 << k) + 1] + k;

		int ID_1 = RMQ_ID[RMQHash[firstA] + (int64_t)k];
		int ID_2 = RMQ_ID[RMQHash[firstB - (1 << k) + 1] + (int64_t)k];

		if (H2H_TreeHeight_Hash[ID_1] <= H2H_TreeHeight_Hash[ID_2]) {
			return ID_1;
		}
		else {
			return ID_2;
		}

	}

	int Graph::H2HDistancQuery_UsingLCA(int x, int y)
	{
		if (x == y)
			return 0;
		int LCA = findLCA(x, y);
		int checkPointX = H2H_startIndex[x].first;
		int checkPointY = H2H_startIndex[y].first;
		int LCApoint = H2H_startIndex[LCA].first;
		int frontier = H2H_startIndex[LCA].second;
		int result = INT_MAX;
		for (int i = 0; i < frontier; i++)
		{
			if (!H2H_label[LCApoint + i].isTreeNode)
				continue;
			result = ((H2H_label[checkPointX + i].dis + H2H_label[checkPointY + i].dis) < result) ?
				(H2H_label[checkPointX + i].dis + H2H_label[checkPointY + i].dis) : result;
		}
		return result;
	}

	int Graph::H2HDistancQuery_UsingOULA(int x, int y)
	{
		if (x == y)
			return 0;
		int checkPointX = H2H_startIndex[x].first;
		int checkPointY = H2H_startIndex[y].first;

		int frontier = min(H2H_startIndex[x].second, H2H_startIndex[y].second);
		int result = INT_MAX;
		for (int i = 0; i < frontier; i++)
		{
			if (H2H_label[checkPointX + i].Node != H2H_label[checkPointY + i].Node)
				break;
			result = ((H2H_label[checkPointX + i].dis + H2H_label[checkPointY + i].dis) < result) ?
				(H2H_label[checkPointX + i].dis + H2H_label[checkPointY + i].dis) : result;
		}
		return result;
	}

	int Graph::H2HDistancQuery_AncestorToPost(int x, int y)
	{
		//int checkPointX = H2H_startIndex[x].first;
		//int checkPointY = H2H_startIndex[y].first;
		//int minSize = H2H_startIndex[x].second;
		return H2H_label[H2H_startIndex[y].first + H2H_startIndex[x].second - 1].dis;
	}

	void Graph_D_H::Graph::makeH2Hlabel() {
		queue<int> frontier;
		for (int i = ChildHash[head].second; i < ChildHash[head].second + ChildHash[head].first; i++)
		{
			frontier.push(Childs[i]);
		}
		//int height = 1;
		while (!frontier.empty())
		{
			int size = frontier.size();
			//cout << "height: " << height++ << " size : " << size <<" total: "<<total << endl;
			for (int i = 0; i < size; i++)
			{
				int ID = frontier.front();
				frontier.pop();
				int H2HSize = H2H_startIndex[ID].second - 1;
				int IDNowH2Hindex = H2H_startIndex[ID].first;
				//displaySingalLabel(ID);
				//construct others
				for (int k = IDNowH2Hindex + H2HSize - 1; k >= IDNowH2Hindex; k--)
				{
					if (!H2H_label[k].isTreeNode) {
						continue;
					}
					//int adjH2HStartIndex = H2H_startIndex[H2H_label[k].Node].first;
					int q = k - 1;
					for (; q >= IDNowH2Hindex; q--)
					{
						int templength = H2H_label[k].dis + H2H_label[q - IDNowH2Hindex + H2H_startIndex[H2H_label[k].Node].first].dis;
						if (H2H_label[q].dis > templength) {
							H2H_label[q].dis = templength;
							H2H_label[q].Hub = H2H_label[k].Node;
						}
						//H2H_label[q].dis = min(H2H_label[q].dis, H2H_label[k].dis + H2H_label[q - IDNowH2Hindex + H2H_startIndex[H2H_label[k].Node].first].dis);
					}
					for (int q = k + 1; q <= IDNowH2Hindex + H2HSize - 1; q++)
					{
						int templength = H2H_label[k].dis + H2H_label[k - IDNowH2Hindex + H2H_startIndex[H2H_label[q].Node].first].dis;
						if (H2H_label[q].dis > templength) {
							H2H_label[q].dis = templength;
							H2H_label[q].Hub = H2H_label[k].Node;
						}
					}
				}

				for (int j = ChildHash[ID].second; j < ChildHash[ID].second + ChildHash[ID].first; j++)
				{
					frontier.push(Childs[j]);
				}
			}
		}
	}


	void Graph_D_H::Graph::constructBFSTree()
	{
		queue<int> frontier;
		int maxLabelWidth = 1;
		int tempHeight = 0;
		int height = 0;
		set<int> vertexInMaxWitdh = {};
		TreeHash.push_back(0);
		TreeBFS.push_back(head);
		TreeHash.push_back(TreeBFS.size());

		TreeBFS_Hash.push_back(0);
		TreeBFS_ID.push_back(head);
		TreeBFS_adj.push_back(head);
		TreeBFS_pos.push_back(0);
		//TreeBFS_changeTime.push_back(0);
		TreeBFS_Hash.push_back(1);

		for (int i = ChildHash[head].second; i < ChildHash[head].second + ChildHash[head].first; i++)
		{
			frontier.push(Childs[i]);
		}
		//int height = 1;
		while (!frontier.empty())
		{
			int size = frontier.size();
			//cout << "height: " << height++ << " size : " << size <<" total: "<<total << endl;
			for (int i = 0; i < size; i++)
			{
				int ID = frontier.front();
				frontier.pop();
				TreeBFS.push_back(ID);
				for (int64_t index = H2H_pos_hash[ID]; index < H2H_pos_hash[ID + 1]; index++) {
					TreeBFS_ID.push_back(ID);
					TreeBFS_adj.push_back(H2H_pos_ID[index]);
					TreeBFS_pos.push_back(H2H_pos_POS[index]);
					//TreeBFS_changeTime.push_back(H2H_pos_hash[ID + 1] - 2 - index);
				}
				for (int j = ChildHash[ID].second; j < ChildHash[ID].second + ChildHash[ID].first; j++)
				{
					frontier.push(Childs[j]);
				}
			}
			TreeHash.push_back(TreeBFS.size());
			TreeBFS_Hash.push_back(TreeBFS_ID.size());
		}


		Childs.clear();
		Childs.shrink_to_fit();
		ChildHash.clear();
		ChildHash.shrink_to_fit();
		//TreeBFS_D = TreeBFS;
	}









	void Graph_D_H::Graph::makeH2Hlabel_MultiThread()
	{
		Graph_D_H::time_Mine time;

		std::cout << "construct H2H on CPU start " << std::endl;
		time.updateStart();

		for (int tempHeight = 0; tempHeight < TreeHash.size() - 1; tempHeight++)
		{
			int startIndex = TreeHash[tempHeight];
			int endIndex = TreeHash[tempHeight + 1];

			int tempThreadNum = min(threadNumber, endIndex - startIndex);

			int tempSize = (endIndex - startIndex) / tempThreadNum + 1;
			vector<vector<int>> candidateHeap(tempThreadNum, vector<int>());
			int it = 0, temp = 0;
			for (int i = startIndex; i < endIndex; i++) {
				int ID = TreeBFS[i];
				candidateHeap[it].emplace_back(ID);
				temp++;
				if (temp == tempSize) {
					it++;
					temp = 0;
				}
			}
			std::vector<std::thread> threads;

			for (int i = 0; i < tempThreadNum; i++) {

				threads.emplace_back(
					[this, &candidateHeap, i]() {
						for (auto& ID : candidateHeap[i]) {
							int H2HSize = H2H_startIndex[ID].second - 1;
							int IDNowH2Hindex = H2H_startIndex[ID].first;

							for (int k = IDNowH2Hindex + H2HSize - 1; k >= IDNowH2Hindex; k--)
							{
								if (!H2H_label[k].isTreeNode) {
									continue;
								}

								for (int q = k - 1; q >= IDNowH2Hindex; q--)
								{
									//if (H2H_label[q].isTreeNode) {
									//	continue;
									//}
									int templength = H2H_label[k].dis + H2H_label[q - IDNowH2Hindex + H2H_startIndex[H2H_label[k].Node].first].dis;
									if (H2H_label[q].dis > templength) {
										H2H_label[q].dis = templength;
										H2H_label[q].Hub = H2H_label[k].Node;
									}
								}
								for (int q = k + 1; q <= IDNowH2Hindex + H2HSize - 1; q++)
								{
									int templength = H2H_label[k].dis + H2H_label[k - IDNowH2Hindex + H2H_startIndex[H2H_label[q].Node].first].dis;
									if (H2H_label[q].dis > templength) {
										H2H_label[q].dis = templength;
										H2H_label[q].Hub = H2H_label[k].Node;
									}
								}
							}
						}
					}
				);

			}

			for (auto& its : threads) {
				its.join();
			}

		}
		translateBeforeQuery();
		time.updateEnd();
		H2HUsingTime_CPU = time.get_microsecond_duration();
		std::cout << "\t CPU construct end, using time: " << time.get_microsecond_duration() << std::endl;
	}



	void Graph_D_H::Graph::inConstructH2H_noHub()
	{
		//malloc
		//mallocH2HLabel_noHub();

		Graph_D_H::time_Mine time;
		std::cout << "malloc H2H label start " << std::endl;
		time.updateStart();
		mallocH2HLabel_noHub();
		time.updateEnd();
		H2HmallocTime = time.get_microsecond_duration();
		std::cout << "\t malloc end, using time: " << time.get_microsecond_duration() << std::endl;

		//constructBFSTree
		std::cout << "construct bfs tree start " << endl;
		time.updateStart();
		constructBFSTree();
		time.updateEnd();
		H2HConstructBFSTreeTime = time.get_microsecond_duration();
		std::cout << "\t bfs construct end, using time: " << time.get_microsecond_duration() << std::endl;
		//make
		makeH2HLabel_noHub_serial();
	}

	void Graph_D_H::Graph::inConstructH2H_noHub_multiThread()
	{
		//malloc
		Graph_D_H::time_Mine time;
		std::cout << "malloc H2H label start " << std::endl;
		time.updateStart();
		mallocH2HLabel_noHub();
		time.updateEnd();
		H2HmallocTime = time.get_microsecond_duration();
		std::cout << "\t malloc end, using time: " << time.get_microsecond_duration() << std::endl;

		//constructBFSTree
		std::cout << "construct bfs tree start " << endl;
		time.updateStart();
		constructBFSTree();
		time.updateEnd();
		H2HConstructBFSTreeTime = time.get_microsecond_duration();
		std::cout << "\t bfs construct end, using time: " << time.get_microsecond_duration() << std::endl;
		//make
		makeH2HLabel_noHub_multiThred();
	}

	void Graph_D_H::Graph::inConstructH2H_noHub_D()
	{
		//malloc
		Graph_D_H::time_Mine time;
		std::cout << "malloc H2H label start " << std::endl;
		time.updateStart();
		mallocH2HLabel_noHub();
		time.updateEnd();
		H2HmallocTime = time.get_microsecond_duration();
		std::cout << "\t malloc end, using time: " << time.get_microsecond_duration() << std::endl;


		//constructBFSTree
		std::cout << "construct bfs tree start " << endl;
		time.updateStart();
		constructBFSTree();
		time.updateEnd();
		H2HConstructBFSTreeTime = time.get_microsecond_duration();
		std::cout << "\t bfs construct end, using time: " << time.get_microsecond_duration() << std::endl;

		std::cout << "translate bfs tree start " << std::endl;
		time.updateStart();
		frontier_Hash_D = frontier_Hash;
		frontier_ID_D = frontier_ID;
		TreeBFS_D = TreeBFS;
		//TreeBFS_changeTime_D = TreeBFS_changeTime;
		time.updateEnd();
		H2HTranslateBFSTreeTime = time.get_microsecond_duration();
		std::cout << "\t bfs translate end, using time: " << time.get_microsecond_duration() << std::endl;
		//make
		//displayH2H_noHub();

		//makeH2HLabel_noHub_noComm();
		//makeH2HLabel_noHub_noComm_2();
		makeH2HLabel_noHub_noComm_3();
		H2H_dis = H2H_dis_D;

		//H2H_pos_ID_D.clear();
		//H2H_pos_ID_D.shrink_to_fit();
		//H2H_pos_ID.clear();
		//H2H_pos_ID.shrink_to_fit();

		frontier_Hash.clear();
		frontier_Hash.shrink_to_fit();
		frontier_Hash_D.clear();
		frontier_Hash_D.shrink_to_fit();
		frontier_ID.clear();
		frontier_ID.shrink_to_fit();
		frontier_ID_D.clear();
		frontier_ID_D.shrink_to_fit();
	}

	void Graph_D_H::Graph::inConstructH2H_noHub_D_2()
	{
		//malloc
		Graph_D_H::time_Mine time;
		std::cout << "malloc H2H label start " << std::endl;
		time.updateStart();
		mallocH2HLabel_noHub();
		time.updateEnd();
		H2HmallocTime = time.get_microsecond_duration();
		std::cout << "\t malloc end, using time: " << time.get_microsecond_duration() << std::endl;


		//constructBFSTree
		std::cout << "construct bfs tree start " << endl;
		time.updateStart();
		constructBFSTree();
		time.updateEnd();
		H2HConstructBFSTreeTime = time.get_microsecond_duration();
		std::cout << "\t bfs construct end, using time: " << time.get_microsecond_duration() << std::endl;

		std::cout << "translate bfs tree start " << std::endl;
		time.updateStart();
		frontier_Hash_D = frontier_Hash;
		frontier_ID_D = frontier_ID;
		TreeBFS_D = TreeBFS;
		//TreeBFS_changeTime_D = TreeBFS_changeTime;
		time.updateEnd();
		H2HTranslateBFSTreeTime = time.get_microsecond_duration();
		std::cout << "\t bfs translate end, using time: " << time.get_microsecond_duration() << std::endl;
		//make
		//displayH2H_noHub();

		makeH2HLabel_noHub_noComm_2();
		//makeH2HLabel_noHub_noComm_3();
		H2H_dis = H2H_dis_D;

		//H2H_pos_ID_D.clear();
		//H2H_pos_ID_D.shrink_to_fit();
		//H2H_pos_ID.clear();
		//H2H_pos_ID.shrink_to_fit();

		frontier_Hash.clear();
		frontier_Hash.shrink_to_fit();
		frontier_Hash_D.clear();
		frontier_Hash_D.shrink_to_fit();
		frontier_ID.clear();
		frontier_ID.shrink_to_fit();
		frontier_ID_D.clear();
		frontier_ID_D.shrink_to_fit();
	}


	void Graph_D_H::Graph::mallocH2HLabel_noHub()
	{
		int totalSize = 0;
		for (auto& it : CHAdjlist) {
			totalSize += it.size() + 1;
		}
		cout << " \t pos size: " << totalSize << endl;
		cout << " \t dis size: " << H2H_Size << endl;
		//H2H_pos_hash.assign(NodeNumber + 1, -1);
		//H2H_dis_hash.assign(NodeNumber + 1, -1);
		//H2H_dis.assign(H2H_Size, INT_MAX);
		//H2H_pos_ID.assign(totalSize, -1);
		//H2H_pos_POS.assign(totalSize, -1);
		cout << "\t CPU allocate start! " << endl;

		try{
		H2H_pos_hash.push_back(0);
		H2H_dis_hash.push_back(0);
		for (int i = 0; i < NodeNumber; i++) {
			H2H_dis.insert(H2H_dis.end(), OULA_DFS_[firstAppeare[i]].second, INT_MAX);

			for (auto& adj : CHAdjlist[i]) {
				int adjID = adj.first;
				int adjPos = OULA_DFS_[firstAppeare[adjID]].second - 1;
				H2H_pos_ID.push_back(adjID);
				H2H_pos_POS.push_back(adjPos);
				H2H_dis[H2H_dis_hash[i] + (int64_t)adjPos] = adj.second.first;
			}
			H2H_pos_ID.push_back(i);
			H2H_pos_POS.push_back(OULA_DFS_[firstAppeare[i]].second - 1);
			H2H_dis[H2H_dis.size() - 1] = 0;
			H2H_pos_hash.push_back(H2H_pos_ID.size());
			H2H_dis_hash.push_back(H2H_dis.size());

			thrust::sort_by_key(H2H_pos_POS.begin() + H2H_pos_hash[i], H2H_pos_POS.end(),
				H2H_pos_ID.begin() + H2H_pos_hash[i]);
		}
		}
		catch(const std::bad_alloc& e){
				std::cerr << "bad allocate : " << e.what() <<" at RAM-label-allocation"<< '\n';  
				std::exit(1);
		}
		//int64_t posTemp = 0;
		//int64_t distemp = 0;
		//for (auto nodeID = 0; nodeID < NodeNumber; nodeID++) {
		//	H2H_pos_hash[nodeID] = posTemp;
		//	H2H_dis_hash[nodeID] = distemp;
		//	//H2H_dis.insert(H2H_dis.end(), OULA_DFS_[firstAppeare[nodeID]].second, INT_MAX);



		//	for (auto& adj : CHAdjlist[nodeID]) {
		//		int adjID = adj.first;
		//		int adjPos = OULA_DFS_[firstAppeare[adjID]].second - 1;
		//		H2H_pos_POS[posTemp] = adjPos;
		//		H2H_pos_ID[posTemp++] = adjID;
		//		//H2H_pos_ID.push_back(adjID);
		//		//H2H_pos_POS.push_back(adjPos);

		//		H2H_dis[H2H_dis_hash[nodeID] + adjPos] = adj.second.first;
		//	}

		//	//posTemp += CHAdjlist[nodeID].size();
		//	distemp += OULA_DFS_[firstAppeare[nodeID]].second;
		//	H2H_pos_POS[posTemp] = nodeID;
		//	H2H_pos_ID[posTemp++] = OULA_DFS_[firstAppeare[nodeID]].second - 1;
		//	H2H_dis[H2H_dis_hash[nodeID] + OULA_DFS_[firstAppeare[nodeID]].second - 1] = 0;

		//	thrust::sort_by_key(H2H_pos_POS.begin() + H2H_pos_hash[nodeID], H2H_pos_POS.begin() + posTemp,
		//		H2H_pos_ID.begin() + H2H_pos_hash[nodeID]);
		//}


		//H2H_pos_hash[NodeNumber] = H2H_pos_ID.size();
		//H2H_dis_hash[NodeNumber] = H2H_dis.size();
		cout << "\t CPU allocate end" << endl;
		//translate
		//H2H_pos_hash_D = H2H_pos_hash;
		//H2H_dis_hash_D = H2H_dis_hash;
		//H2H_pos_ID_D = H2H_pos_ID;
		//H2H_pos_POS_D = H2H_pos_POS;
		//H2H_dis_D = H2H_dis;

		double H2HLabelSize_T;
		cout << "\t CHTreeHeight: " << CHTreeHeight << endl;
		TDTREEHeight = CHTreeHeight;
		double rmq = RMQHash.size() * sizeof(int64_t) + RMQ_ID.size() * sizeof(int) + H2H_TreeHeight_Hash.size() * sizeof(int);
		RMQsize = rmq / (1024 * 1024);
		H2HLabelSize = ((double)(H2H_pos_hash.size() * sizeof(int64_t) + H2H_dis_hash.size() * sizeof(int64_t)  +
			H2H_pos_POS.size() * sizeof(int) + H2H_dis.size() * sizeof(int))) / (1024 * 1024);
		H2HLabelSize_T = (H2H_pos_POS.size() * sizeof(int) + H2H_dis.size() * sizeof(int)) / (1024 * 1024);
		cout << "\t RMQ size : " << RMQsize
			<< "MB, H2H Label size: " << H2HLabelSize_T << "MB" << endl;

		OULA_DFS_.clear();
		OULA_DFS_.shrink_to_fit();
		for (auto& it : CHAdjlist) {
			it.clear();
		}
		CHAdjlist.clear();
		CHAdjlist.shrink_to_fit();

		//displayH2H_noHub();

	}




	void Graph_D_H::Graph::makeH2HLabel_noHub_serial()
	{
		Graph_D_H::time_Mine time;

		std::cout << "construct H2H on CPU-serial start " << std::endl;
		time.updateStart();
		//for (int tempHeight = 0; tempHeight < TreeHash.size() - 1; tempHeight++)
		//{
		//	int startIndex = TreeHash[tempHeight];
		//	int endIndex = TreeHash[tempHeight + 1];
		//	for (int i = startIndex; i < endIndex; i++) {
		//		int nodeID = TreeBFS[i];

		//		for (size_t temp = H2H_pos_hash[(size_t)nodeID + 1] - 1; temp >= H2H_pos_hash[(size_t)nodeID]; temp--) {
		//			int adjID = H2H_pos_ID[temp];
		//			int pos = H2H_pos_POS[temp];
		//			size_t indexNode = H2H_dis_hash[nodeID];
		//			size_t index1_adj = H2H_dis_hash[adjID];


		//			int tempLength_ = H2H_dis[indexNode + (size_t)pos];
		//			for (size_t j = (size_t)pos - 1; j >= 0; j--) {
		//				H2H_dis[indexNode + j] = min(H2H_dis[indexNode + j],
		//					tempLength_ + H2H_dis[index1_adj + j]);
		//			}

		//			//L4 label
		//			int fatherID = nodeID;
		//			for (size_t j = H2H_dis_hash[(size_t)nodeID + 1] - H2H_dis_hash[(size_t)nodeID] - 2; j > pos; j--) {
		//				fatherID = father[fatherID];
		//				
		//				H2H_dis[indexNode + j] = min(H2H_dis[indexNode + j],
		//					tempLength_ + H2H_dis[H2H_dis_hash[fatherID] + (size_t)pos]);
		//			}
		//		}
		//	}
		//}

		for (int tempHeight = 0; tempHeight < TreeHash.size() - 1; tempHeight++) {
			int startIndex = TreeHash[tempHeight];
			int endIndex = TreeHash[tempHeight + 1];
			//cout << "at height: " << tempHeight << endl;
			for (int i = startIndex; i < endIndex; i++) {
				int nodeID = TreeBFS[i];
				//cout << "\tNodeID: " << nodeID <<"start at: "<< H2H_pos_hash[(size_t)nodeID] << "size: " << H2H_pos_hash[(size_t)nodeID + 1] - H2H_pos_hash[(size_t)nodeID] << endl;
				for (int64_t temp = H2H_pos_hash[(int64_t)nodeID + 1] - 2; temp >= H2H_pos_hash[(int64_t)nodeID]; temp--) {
					int adjID = H2H_pos_ID[temp];
					int pos = H2H_pos_POS[temp];
					int64_t indexNode = H2H_dis_hash[nodeID];
					int64_t index1_adj = H2H_dis_hash[adjID];
					int tempLength_ = H2H_dis[indexNode + (int64_t)pos];
					for (int j = pos - 1; j >= 0; j--) {
						H2H_dis[indexNode + (int64_t)j] = std::min(H2H_dis[indexNode + (int64_t)j],
							tempLength_ + H2H_dis[index1_adj + (int64_t)j]);
					}
					// L4 label
					//int64_t fatherID = nodeID;
					//for (int j = (int)(H2H_dis_hash[nodeID + 1] - H2H_dis_hash[nodeID]) - 2; j > pos; j--) {
					//	fatherID = (int64_t)father[fatherID];

					//	H2H_dis[indexNode + (int64_t)j] = std::min(H2H_dis[indexNode + (int64_t)j],
					//		tempLength_ + H2H_dis[H2H_dis_hash[fatherID] + (int64_t)pos]);
					//}
				}
			}
		}
		cout << "finish calculate" << endl;
		//translateBeforeQuery();
		translateH2H_noHub();

		time.updateEnd();
		H2HUsingTime_CPU = time.get_microsecond_duration();
		std::cout << "\t CPU construct end, using time: " << time.get_microsecond_duration() << std::endl;
	}





	void Graph::makeH2HLabel_noHub_multiThred()
	{
		Graph_D_H::time_Mine time;

		std::cout << "construct H2H on CPU multi-thread start " << std::endl;
		time.updateStart();

		for (int64_t tempHeight = 0; tempHeight < (int64_t)TreeHash.size() - 1; tempHeight++)
		{
			int64_t startIndex = TreeHash[tempHeight];
			int64_t endIndex = TreeHash[tempHeight + 1];
			Graph_D_H::time_Mine time1;
			time1.updateStart();
			int64_t tempThreadNum = min((int64_t)threadNumber, endIndex - startIndex);

			int64_t tempSize = (endIndex - startIndex) / tempThreadNum + 1;
			vector<vector<int64_t>> candidateHeap(tempThreadNum, vector<int64_t>());
			int64_t it = 0, temp = 0;
			for (int64_t i = startIndex; i < endIndex; i++) {
				int64_t ID = TreeBFS[i];
				candidateHeap[it].emplace_back(ID);
				temp++;
				if (temp == tempSize) {
					it++;
					temp = 0;
				}
			}
			std::vector<std::thread> threads;

			for (int i = 0; i < tempThreadNum; i++) {

				threads.emplace_back(
					[this, &candidateHeap, i]() {
						for (auto& nodeID : candidateHeap[i]) {
							for (int64_t temp = H2H_pos_hash[(int64_t)nodeID + 1] - 2; temp >= H2H_pos_hash[(int64_t)nodeID]; temp--) {
								int adjID = H2H_pos_ID[temp];
								int pos = H2H_pos_POS[temp];
								int64_t indexNode = H2H_dis_hash[nodeID];
								int64_t index1_adj = H2H_dis_hash[adjID];
								int tempLength_ = H2H_dis[indexNode + (int64_t)pos];
								for (int j = pos - 1; j >= 0; j--) {
									H2H_dis[indexNode + (int64_t)j] = std::min(H2H_dis[indexNode + (int64_t)j],
										tempLength_ + H2H_dis[index1_adj + (int64_t)j]);
								}
								// L4 label
								//int64_t fatherID = nodeID;
								//for (int j = (int)(H2H_dis_hash[nodeID + 1] - H2H_dis_hash[nodeID]) - 2; j > pos; j--) {
								//	fatherID = (int64_t)father[fatherID];

								//	H2H_dis[indexNode + (int64_t)j] = std::min(H2H_dis[indexNode + (int64_t)j],
								//		tempLength_ + H2H_dis[H2H_dis_hash[fatherID] + (int64_t)pos]);
								//}
							}
						}
					}
				);

			}

			for (auto& its : threads) {
				its.join();
			}
			time1.updateEnd();
			//std::cout << "At height: " << tempHeight << " Kernel execution time: " << time1.get_microsecond_duration() / 1000 << " ms" << std::endl;

		}
		//translateBeforeQuery();
		translateH2H_noHub();

		time.updateEnd();
		H2HUsingTime_CPU = time.get_microsecond_duration();
		std::cout << "\t CPU construct end, using time: " << time.get_microsecond_duration() << std::endl;
	}


	void Graph::makeH2HLabel_noHub_multiThred_2()
	{
		Graph_D_H::time_Mine time;

		std::cout << "construct H2H on CPU multi-thread start " << std::endl;
		time.updateStart();

		for (int64_t tempHeight = 0; tempHeight < (int64_t)TreeBFS_Hash.size() - 1; tempHeight++)
		{
			int64_t startIndex = TreeBFS_Hash[tempHeight];
			int64_t endIndex = TreeBFS_Hash[tempHeight + 1];
			Graph_D_H::time_Mine time1;
			time1.updateStart();
			int64_t tempThreadNum = min((int64_t)threadNumber, endIndex - startIndex);

			int64_t tempSize = (endIndex - startIndex) / tempThreadNum + 1;
			vector<vector<int64_t>> candidateHeap(tempThreadNum, vector<int64_t>());
			int64_t it = 0, temp = 0;
			for (int64_t i = startIndex; i < endIndex; i++) {
				//int64_t ID = TreeBFS[i];
				candidateHeap[it].emplace_back(i);
				temp++;
				if (temp == tempSize) {
					it++;
					temp = 0;
				}
			}
			std::vector<std::thread> threads;

			for (int i = 0; i < tempThreadNum; i++) {

				threads.emplace_back(
					[this, &candidateHeap, i]() {
						for (auto& nodeIDindex : candidateHeap[i]) {

							std::mutex mt;
							int nodeID = TreeBFS_ID[nodeIDindex];
							int adjID = TreeBFS_adj[nodeIDindex];
							int pos = TreeBFS_pos[nodeIDindex];

							int64_t indexNode = H2H_dis_hash[nodeID]; //current vertex's label position
							int64_t index1_adj = H2H_dis_hash[adjID]; //ancestor's label position

							int tempLength_ = H2H_dis[indexNode + pos];
							for (int64_t j = pos - 1; j >= 0; j--) {

								std::lock_guard<std::mutex> lock(mt);
								H2H_dis[indexNode + j] = std::min(H2H_dis[indexNode + j], (tempLength_ + H2H_dis[index1_adj + j]));
								//atomicMin(&H2H_dis[indexNode + j], (tempLength_ + H2H_dis[index1_adj + j]));
							}

							//L4_label
							//int fatherID = nodeID;
							//for (int64_t j = (int)(H2H_dis_hash[nodeID + 1] - H2H_dis_hash[nodeID]) - 2; j > pos; j--) {
							//	fatherID = father[fatherID];
							//	//atomicMin(&H2H_dis[indexNode + j], (tempLength_ + H2H_dis[H2H_dis_hash[fatherID] + pos]));
							//	std::lock_guard<std::mutex> lock(mt);
							//	H2H_dis[indexNode + j] = std::min(H2H_dis[indexNode + j], (tempLength_ + H2H_dis[H2H_dis_hash[fatherID] + pos]));
							//}

						}
					}
				);
			}

			for (auto& its : threads) {
				its.join();
			}
			time1.updateEnd();
			//std::cout << "At height: " << tempHeight << " Kernel execution time: " << time1.get_microsecond_duration() / 1000 << " ms" << std::endl;

		}
		//translateBeforeQuery();
		translateH2H_noHub();

		time.updateEnd();
		H2HUsingTime_CPU = time.get_microsecond_duration();
		std::cout << "\t CPU construct end, using time: " << time.get_microsecond_duration() << std::endl;
	}

	void Graph_D_H::Graph::inConstructH2H_noHub_multiThread_2()
	{
		//malloc
		Graph_D_H::time_Mine time;
		std::cout << "malloc H2H label start " << std::endl;
		time.updateStart();
		mallocH2HLabel_noHub();
		time.updateEnd();
		H2HmallocTime = time.get_microsecond_duration();
		std::cout << "\t malloc end, using time: " << time.get_microsecond_duration() << std::endl;

		//constructBFSTree
		std::cout << "construct bfs tree start " << endl;
		time.updateStart();
		constructBFSTree();
		time.updateEnd();
		H2HConstructBFSTreeTime = time.get_microsecond_duration();
		std::cout << "\t bfs construct end, using time: " << time.get_microsecond_duration() << std::endl;
		//make
		makeH2HLabel_noHub_multiThred_2();
	}

}
