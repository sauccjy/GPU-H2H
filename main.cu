#include "hip/hip_runtime.h"

#include"Graph_D.cuh"
using namespace std;



void TDAndH2HConstruct_Tradition_plus_G_2(Graph_D_H::Graph* Agraph, int goalHeight, int changeHeight)
{
	Agraph->ConstructMethod = 2;
	Agraph->startGPU();
	//Agraph->displayAdjList();
	Graph_D_H::time_Mine time;
	//Agraph->displayPartition();
	Agraph->makeAdjcentNode(0);
	//Agraph->displayOrderAndCHTree();

	cout << endl; cout << endl;
	cout << "________________TD start!_____________________" << endl;
	if (goalHeight != changeHeight)
	{
		cout << "translate before CH start " << endl;
		time.updateStart();
		Agraph->translateBeforeCH();
		time.updateEnd();
		Agraph->translateLUBToGPUTime = time.get_microsecond_duration();
		cout << "\t translate before CH end, using time: " << time.get_microsecond_duration() << endl;

		cout << "TD in GPU start " << endl;
		time.updateStart();
		Agraph->partition_TD_Parallel_D_2(changeHeight);
		time.updateEnd();
		Agraph->TDInGPUTime = time.get_microsecond_duration();
		cout << "\t TD in GPU using time: " << time.get_microsecond_duration() << endl;
		Agraph->translateInTD();
	}
	//Agraph->displayCHTree();
	Agraph->TDInCPU(changeHeight);
	cout << endl;
	//Agraph->displayCHAdjList();
	cout << "start OULARMQ " << endl;
	time.updateStart();
	Agraph->beforeH2H();
	time.updateEnd();
	cout << "\t OULARMQ end, using time: " << time.get_microsecond_duration() << endl;
	//Agraph->displayOULARMQ();
	cout << endl; cout << endl;
	cout << "_________________start construct H2H______________________" << endl;

	//Agraph->inConstructH2H_mix();
	Agraph->inConstructH2H_noComm();
}

void Partition_CH_Construct(Graph_D_H::Graph* Agraph, int goalHeight, int changeHeight) {
	Agraph->startGPU();
	//Agraph->displayAdjList();
	Graph_D_H::time_Mine time;

	Agraph->makeAdjcentNode(0);
	//Agraph->displayOrderAndCHTree();
	cout << endl; cout << endl;
	cout << "________________TD start!_____________________" << endl;
	Agraph->TDInCPUMultiThread(changeHeight);
	//Agraph->displayCHAdjList();

	cout << endl;
	//Agraph->displayCHAdjList();
	//time.updateEnd();
	//cout << " using time: " << time.get_microsecond_duration() << endl;
	cout << "start OULARMQ " << endl;
	time.updateStart();
	Agraph->beforeH2H();
	time.updateEnd();
	cout << "\t OULARMQ end, using time: " << time.get_microsecond_duration() << endl;
	//Agraph->displayOULARMQ();
	cout << endl; cout << endl;
	cout << "_________________start construct H2H______________________" << endl;

	Agraph->inConstructH2H_MultiThread();

	//Agraph->displayH2HLabel();
}


void TDAndH2HConstruct_Tradition_plus_G_2_noHub(Graph_D_H::Graph* Agraph, int goalHeight, int changeHeight)
{
	Agraph->ConstructMethod = 2;
	Agraph->startGPU();
	//Agraph->displayAdjList();
	Graph_D_H::time_Mine time;
	//Agraph->displayPartition();
	Agraph->makeAdjcentNode(0);
	//Agraph->displayOrderAndCHTree();

	cout << endl; cout << endl;
	cout << "________________TD start!_____________________" << endl;
	if (goalHeight != changeHeight)
	{
		cout << "translate before CH start " << endl;
		time.updateStart();
		Agraph->translateBeforeCH();
		time.updateEnd();
		Agraph->translateLUBToGPUTime = time.get_microsecond_duration();
		cout << "\t translate before CH end, using time: " << time.get_microsecond_duration() << endl;

		cout << "TD in GPU start " << endl;
		time.updateStart();
		Agraph->partition_TD_Parallel_D_2(changeHeight);
		time.updateEnd();
		Agraph->TDInGPUTime = time.get_microsecond_duration();
		cout << "\t TD in GPU using time: " << time.get_microsecond_duration() << endl;
		Agraph->translateInTD();
	}
	//Agraph->displayCHTree();
	Agraph->TDInCPU(changeHeight);
	cout << endl;
	//Agraph->displayCHAdjList();
	cout << "start OULARMQ " << endl;
	time.updateStart();
	Agraph->beforeH2H_noHub();
	time.updateEnd();
	cout << "\t OULARMQ end, using time: " << time.get_microsecond_duration() << endl;
	//Agraph->displayOULARMQ();
	cout << endl; cout << endl;
}

void Partition_CH_Construct_noHub(Graph_D_H::Graph* Agraph, int goalHeight, int changeHeight) {
	Agraph->startGPU();
	//Agraph->displayAdjList();
	Graph_D_H::time_Mine time;

	Agraph->makeAdjcentNode(0);
	//Agraph->displayOrderAndCHTree();
	cout << endl; cout << endl;
	cout << "________________TD start!_____________________" << endl;
	Agraph->TDInCPUMultiThread(changeHeight);
	//Agraph->displayCHAdjList();

	cout << endl;
	//Agraph->displayCHAdjList();
	cout << "start OULARMQ " << endl;
	time.updateStart();
	Agraph->beforeH2H_noHub();
	time.updateEnd();
	cout << "\t OULARMQ end, using time: " << time.get_microsecond_duration() << endl;
	//Agraph->displayOULARMQ();
	cout << endl; cout << endl;
}

void contraction_noHub(Graph_D_H::Graph* Agraph, const int contractDevice, const int goalHeight, const int changeHeight) {
	if (contractDevice == 0) { //CPU-only (CPU parallel contraction + global contraction)
		Partition_CH_Construct_noHub(Agraph, goalHeight, changeHeight);
		return;
	}
	if (contractDevice == 1) { //GPU
		TDAndH2HConstruct_Tradition_plus_G_2_noHub(Agraph, goalHeight, changeHeight);
		return;
	}
}

void constructLabel_noHub(Graph_D_H::Graph* Agraph, const int parallelFine, const int constructDevice) {

	cout << "_________________start construct H2H______________________" << endl;
	if (parallelFine == 0) { //serial
		Agraph->inConstructH2H_noHub();
		return;
	}
	if (parallelFine == 1) { //BFS fine parallel
		if (constructDevice == 0) { // CPU
			Agraph->inConstructH2H_noHub_multiThread();
			return;
		}
		if (constructDevice == 1) { // GPU
			Agraph->inConstructH2H_noHub_D_2();
			return;
		}
		
	}
	if (parallelFine == 2) { //TD fine parallel
		if (constructDevice == 0) { // CPU
			Agraph->inConstructH2H_noHub_multiThread_2();
			return;
		}
		if (constructDevice == 1) { // GPU
			Agraph->inConstructH2H_noHub_D();
			return;
		}
	}
}


void printInfo(Graph_D_H::Graph* Agraph) {
	Agraph->printConstructInfo();
}

void Query(Graph_D_H::Graph* Agraph) {
	Agraph->cleanBeforeQuery();
	Graph_D_H::time_Mine time;
	cout << "translate RMQ" << endl;
	time.updateStart();
	Agraph->translateRMQ_noHub();
	time.updateEnd();
	Agraph->translateRMQTime = time.get_microsecond_duration();
	cout << "\t translate RMQ using time: " << time.get_microsecond_duration() << endl;
	if (Agraph->graphName == "NY") {
		Agraph->answerRealNYQuery();
	}
	else {
		Agraph->answerRandomQuery();
	}
	cout << "translate query" << endl;
	time.updateStart();
	Agraph->translateQueryResultBack();
	time.updateEnd();
	cout << "using time: " << time.get_microsecond_duration() << "us" << endl;

}

void Query_noHub(Graph_D_H::Graph* Agraph) {
	Agraph->cleanBeforeQuery();
	Graph_D_H::time_Mine time;

	cout << "translate RMQ" << endl;
	time.updateStart();
	Agraph->translateRMQ_noHub();
	time.updateEnd();
	Agraph->translateRMQTime = time.get_microsecond_duration();
	cout << "\t translate RMQ using time: " << time.get_microsecond_duration() << endl;
	//Agraph->translateQuery();
	//if (Agraph->graphName == "NY") {
		//Agraph->answerRealNYQuery_noHub();
	//}
	//else {
		Agraph->answerRandomQuery_noHub();
	//}
	cout << "translate query" << endl;
	time.updateStart();
	Agraph->translateQueryResultBack();
	time.updateEnd();
	std::cout << "using time: " << time.get_microsecond_duration() << "us" << endl;
}



int main(int argc, char** argv)
{
	if (argc != 9)
		return 0;

	//string gname(argv[1]);
	char* endptr;

	string graphName(argv[1]);
	int TreeHeight = (int)strtol(argv[2], &endptr, 10);			//tree height 
	int changeHeight = (int)strtol(argv[3], &endptr, 10);		//change contract order base, from partition order to global order
	int threadPoolSize = (int)strtol(argv[4], &endptr, 10);		//max thread number on CPU
	int partitionMethod = (int)strtol(argv[5], &endptr, 10);	//[1,2,3,4]: 1: XYCoord partition; 2: HC2L-like partition; 3: metis partition; 4: SCOTCH partition

	int contractDevice = (int)strtol(argv[6], &endptr, 10);		//[0,1]: partition order contraction device: 0: CPU; 1: GPU
	int parallelFine = (int)strtol(argv[7], &endptr, 10);		//[0,1,2]: 0: serial; 1: BFS fine; 2: TD fine
	int constructDevice = (int)strtol(argv[8], &endptr, 10);	//[0,1]: construction device: 0: CPU; 1: GPU

	string NodeFile, EdgeFile;
	NodeFile = "./Graph/USA/";
	EdgeFile = "./Graph/USA/";
	NodeFile += graphName;
	EdgeFile += graphName;
	NodeFile += "Node.txt";
	EdgeFile += "Edge.txt";
	cout << NodeFile << endl;
	cout << EdgeFile << endl;

	string PartitionCutFile, PartitionIDFile;
	PartitionCutFile = "./Partition/" + graphName + "/";
	PartitionIDFile = PartitionCutFile;
	if (partitionMethod == 2) {
		PartitionCutFile += "MinimumCut.txt";
		PartitionIDFile += "MinimumID.txt";
	}
	else if (partitionMethod == 1) {
		PartitionCutFile += "LatitudeCut.txt";
		PartitionIDFile += "LatitudeID.txt";
	}
	else if (partitionMethod == 3) {
		PartitionCutFile += "metisCut.txt";
		PartitionIDFile += "metisID.txt";
	}
	else if (partitionMethod == 4) {
		PartitionCutFile += "scotchCut.txt";
		PartitionIDFile += "scotchID.txt";
	}
	cout << PartitionCutFile << endl;
	cout << PartitionIDFile << endl;
	//cout << "myPair size: " << sizeof(Graph_D_H::myPair<int>) << endl;

	Graph_D_H::Graph* Agraph = new Graph_D_H::Graph(graphName, TreeHeight, NodeFile, EdgeFile, threadPoolSize, changeHeight, PartitionCutFile, PartitionIDFile);
	Agraph->PartitionMethod = partitionMethod;
	Agraph->parallelFine_Construct = parallelFine;
	Agraph->contractDevice = contractDevice;
	Agraph->parallelFine = parallelFine;
	Agraph->constructDevice = constructDevice;
	Agraph->checklink();

	contraction_noHub(Agraph, contractDevice, TreeHeight, changeHeight);
	constructLabel_noHub(Agraph, parallelFine, constructDevice);
	printInfo(Agraph);
	Query_noHub(Agraph);
	
}