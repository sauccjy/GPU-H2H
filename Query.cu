#include "hip/hip_runtime.h"
#pragma once
#include"Graph_D.cuh"

namespace Graph_D_H
{
	void Graph_D_H::Graph::generateQuery()
	{
		x.assign(queryMaxSize,0);
		y.assign(queryMaxSize,0);
		result.assign(queryMaxSize,INT_MAX);

		std::default_random_engine generator(std::time(0));
		std::uniform_int_distribution<int> distribution(0, NodeNumber - 1);
		for (int i = 0; i < queryMaxSize; ++i) {
			x[i] = distribution(generator);
			y[i] = distribution(generator);
		}
	}


	void Graph_D_H::Graph::uploadRealQuery()
	{
		fstream queryFile1("./Query/NY/normal.txt", ios::in | ios::out);
		x.clear();
		y.clear();
		for (int i = 0; i < 100000; i++) {
			int start, des, other;
			queryFile1 >> start >> des >> other;
			x.push_back(start);
			y.push_back(des);
		}
		queryFile1.close();

		fstream queryFile2("./Query/NY/sparse.txt", ios::in | ios::out);
		for (int i = 0; i < 100000; i++) {
			int start, des, other;
			queryFile2 >> start >> des >> other;
			x.push_back(start);
			y.push_back(des);
		}
		queryFile2.close();
		queryMaxSize = x.size();
		result.assign(x.size(), INT_MAX);
	}


	long long int Graph::H2Hquery_bunch(int size)
	{
		Graph_D_H::time_Mine time;
		cout << "H2H_sequence start query,querySize=" <<size<< endl;
		time.updateStart();

		for (int i = 0; i < size; i++) {
			result[i] = H2HDistancQuery_UsingLCA(x[i], y[i]);
		}

		time.updateEnd();
		cout << "using time: " << time.get_microsecond_duration() << "us" << endl;

		result.assign(queryMaxSize, INT_MAX);

		return time.get_microsecond_duration();

	}

	long long int Graph::H2Hquery_bunch_MultiThread(int size) {
		const unsigned int numThreads = threadNumber;
		const size_t chunkSize = size / numThreads + 1;
		std::vector<std::thread> threads;

		Graph_D_H::time_Mine time;

		cout << "H2H_multithread start query, thread size : "<< numThreads <<" ,querySize=" << size << endl;
		time.updateStart();

		for (int i = 0; i < numThreads; ++i) {
			size_t start = i * chunkSize;
			size_t end = (i == numThreads - 1) ? size : (i + 1) * chunkSize;
			end = min((size_t)size, (i + 1) * chunkSize);
			threads.emplace_back([&, start, end] {
				for (size_t j = start; j < end; ++j) {
					result[j] = H2HDistancQuery_UsingLCA(x[j], y[j]);
				}
				});
		}
		for (auto& thread : threads) {
			thread.join();
		}

		time.updateEnd();
		cout << "using time: " << time.get_microsecond_duration() << "us" << endl;

		result.assign(queryMaxSize, INT_MAX);
		return time.get_microsecond_duration();
	}


	void Graph_D_H::Graph::cleanResult()
	{

	}





	//noHub 
	int Graph_D_H::Graph::H2HDistancQuery_UsingLCA_noHub(int x, int y)
	{
		if (x == y)
			return 0;
		int LCA = findLCA_noHub(x, y);
		int resu = INT_MAX;
		for (int i = H2H_pos_hash[LCA]; i < H2H_pos_hash[LCA + 1]; i++) {

			int templength = H2H_dis[H2H_dis_hash[x] + H2H_pos_POS[i]] + H2H_dis[H2H_dis_hash[y] + H2H_pos_POS[i]];
			if (templength < resu) {
				resu = templength;
			}
		}
		return resu;
	}

	int Graph_D_H::Graph::H2HDistancQuery_noLCA_noHub(int x, int y)
	{
		if (x == y)
			return 0;
		int FirstA = firstAppeare[x];//position  in  OULA_DFS
		int FirstB = firstAppeare[y];
		int firstA = (FirstA < FirstB) * FirstA + (FirstB <= FirstA) * FirstB;
		int firstB = (FirstA < FirstB) * FirstB + (FirstB <= FirstA) * FirstA;
		int k = (int)(log2((double)(firstB - firstA + 1)));//k means log2(gap);

		int ID_1 = RMQ_ID[RMQHash[firstA] + (int64_t)k];
		int ID_2 = RMQ_ID[RMQHash[firstB - (1 << k) + 1] + (int64_t)k];
		int LCASize = H2H_TreeHeight_Hash[ID_2];
		if (H2H_TreeHeight_Hash[ID_1] <= H2H_TreeHeight_Hash[ID_2]) {
			LCASize = H2H_TreeHeight_Hash[ID_1];
		}
		int resu = INT_MAX;


		for (int i = 0; i < LCASize; i++) {

			int templength = H2H_dis[H2H_dis_hash[x] + (int64_t)i] + H2H_dis[H2H_dis_hash[y] + (int64_t)i];
			if (templength < resu) {
				resu = templength;
			}
		}
		return resu;
	}

	long long int Graph_D_H::Graph::H2Hquery_bunch_noHub(int size)
	{
		Graph_D_H::time_Mine time;
		cout << "H2H_sequence start query,querySize=" << size << endl;
		time.updateStart();

		for (int i = 0; i < size; i++) {
			result[i] = H2HDistancQuery_UsingLCA_noHub(x[i], y[i]);
		}

		time.updateEnd();
		cout << "using time: " << time.get_microsecond_duration() << "us" << endl;

		result.assign(queryMaxSize, INT_MAX);

		return time.get_microsecond_duration();
	}


	long long int Graph_D_H::Graph::H2Hquery_bunch_MultiThread_noHub(int size)
	{
		const unsigned int numThreads = threadNumber;
		const size_t chunkSize = size / numThreads + 1;
		std::vector<std::thread> threads;

		Graph_D_H::time_Mine time;

		cout << "H2H_multithread start query, thread size : " << numThreads << " ,querySize=" << size << endl;
		time.updateStart();

		for (int i = 0; i < numThreads; ++i) {
			size_t start = i * chunkSize;
			size_t end = (i == numThreads - 1) ? size : (i + 1) * chunkSize;
			end = min((size_t)size, (i + 1) * chunkSize);
			threads.emplace_back([&, start, end] {
				for (size_t j = start; j < end; ++j) {
					result[j] = H2HDistancQuery_UsingLCA_noHub(x[j], y[j]);
				}
				});
		}
		for (auto& thread : threads) {
			thread.join();
		}

		time.updateEnd();
		cout << "using time: " << time.get_microsecond_duration() << "us" << endl;

		result.assign(queryMaxSize, INT_MAX);
		return time.get_microsecond_duration();
	}

	void Graph_D_H::Graph::answerRandomQuery()
	{
		generateQuery();
		translateQuery();
		int endTime = 500000;
		bool seqContinue = true;
		bool MTContinue_4 = true;
		bool MTContinue_6 = true;
		bool MTContinue_8 = true;
		bool MTContinue_10 = true;
		bool MTContinue_12 = true;
		bool MTContinue_14 = true;
		bool MTContinue_16 = true;
		for (int i = 10000; i <= queryMaxSize; i += queryStep) {

			if (seqContinue) {
				long long int seqtime = H2Hquery_bunch(i);
				QueryTime_seq.push_back(seqtime);
				seqContinue = (seqtime < endTime);
			}
			else {
				QueryTime_seq.push_back(endTime);
			}

			if (MTContinue_4) {
				threadNumber = 4;
				long long int seqtime = H2Hquery_bunch_MultiThread(i);
				QueryTime_MT_4.push_back(seqtime);
				MTContinue_4 = (seqtime < endTime);
			}
			else {
				QueryTime_MT_4.push_back(endTime);
			}

			if (MTContinue_6) {
				threadNumber = 6;
				long long int seqtime = H2Hquery_bunch_MultiThread(i);
				QueryTime_MT_6.push_back(seqtime);
				MTContinue_6 = (seqtime < endTime);
			}
			else {
				QueryTime_MT_6.push_back(endTime);
			}

			if (MTContinue_8) {
				threadNumber = 8;
				long long int seqtime = H2Hquery_bunch_MultiThread(i);
				QueryTime_MT_8.push_back(seqtime);
				MTContinue_8 = (seqtime < endTime);
			}
			else {
				QueryTime_MT_8.push_back(endTime);
			}

			if (MTContinue_10) {
				threadNumber = 10;
				long long int seqtime = H2Hquery_bunch_MultiThread(i);
				QueryTime_MT_10.push_back(seqtime);
				MTContinue_10 = (seqtime < endTime);
			}
			else {
				QueryTime_MT_10.push_back(endTime);
			}

			if (MTContinue_12) {
				threadNumber = 12;
				long long int seqtime = H2Hquery_bunch_MultiThread(i);
				QueryTime_MT_12.push_back(seqtime);
				MTContinue_12 = (seqtime < endTime);
			}
			else {
				QueryTime_MT_12.push_back(endTime);
			}

			if (MTContinue_14) {
				threadNumber = 14;
				long long int seqtime = H2Hquery_bunch_MultiThread(i);
				QueryTime_MT_14.push_back(seqtime);
				MTContinue_14 = (seqtime < endTime);
			}
			else {
				QueryTime_MT_14.push_back(endTime);
			}

			if (MTContinue_16) {
				threadNumber = 16;
				long long int seqtime = H2Hquery_bunch_MultiThread(i);
				QueryTime_MT_16.push_back(seqtime);
				MTContinue_16 = (seqtime < endTime);
			}
			else {
				QueryTime_MT_16.push_back(endTime);
			}
			//QueryTime_seq.push_back(H2Hquery_bunch(i));
			//QueryTime_MT.push_back(H2Hquery_bunch_MultiThread(i));
			cudaThreadNum = 64;
			QueryTime_noLCA_64.push_back(H2Hquery_bunch_noLCA_D(i));
			QueryTime_LCA_64.push_back(H2Hquery_bunch_UsingLCA_D(i));

			cudaThreadNum = 128;
			QueryTime_noLCA_128.push_back(H2Hquery_bunch_noLCA_D(i));
			QueryTime_LCA_128.push_back(H2Hquery_bunch_UsingLCA_D(i));

			cudaThreadNum = 256;
			QueryTime_noLCA_256.push_back(H2Hquery_bunch_noLCA_D(i));
			QueryTime_LCA_256.push_back(H2Hquery_bunch_UsingLCA_D(i));

			cudaThreadNum = 512;
			QueryTime_noLCA_512.push_back(H2Hquery_bunch_noLCA_D(i));
			QueryTime_LCA_512.push_back(H2Hquery_bunch_UsingLCA_D(i));

			cudaThreadNum = 1024;
			QueryTime_noLCA_1024.push_back(H2Hquery_bunch_noLCA_D(i));
			QueryTime_LCA_1024.push_back(H2Hquery_bunch_UsingLCA_D(i));
		}
		string resultFile = "./QueryResult/" + graphName + "/PTHeight-" + to_string(TreeHeight) + "-ChangeHeight-" + to_string(changeHeight) + "-random.csv";
		fstream queryresult(resultFile, ios::app | ios::in | ios::out);
		if (queryresult.is_open()) {
			//queryresult << "CPUThreadNum,CUDAThreadNum\n";
			//queryresult <<threadNumber<<","<< cudaThreadNum << "\n";
			//if(queryresult.peek() == std::ifstream::traits_type::eof())
			queryresult << "queryNumber,sequence,multiThread-4,multiThread-6,multiThread-8,multiThread-10,multiThread-12,multiThread-14,multiThread-16,GPUnoLCA-64,GPUwithLCA-64,GPUnoLCA-128,GPUwithLCA-128,GPUnoLCA-256,GPUwithLCA-256,GPUnoLCA-512,GPUwithLCA-512,GPUnoLCA-1024,GPUwithLCA-1024\n";
			for (int i = 0; i < QueryTime_seq.size(); i++) {
				queryresult << 10000 + i * queryStep << "," << QueryTime_seq[i] << "," << QueryTime_MT_4[i] << ","
					<< QueryTime_MT_6[i] << "," << QueryTime_MT_8[i] << "," << QueryTime_MT_10[i] << "," << QueryTime_MT_12[i] << ","
					<< QueryTime_MT_14[i] << "," << QueryTime_MT_16[i] << ","
					<< QueryTime_noLCA_64[i] << "," << QueryTime_LCA_64[i] << ","
					<< QueryTime_noLCA_128[i] << "," << QueryTime_LCA_128[i] << ","
					<< QueryTime_noLCA_256[i] << "," << QueryTime_LCA_256[i] << ","
					<< QueryTime_noLCA_512[i] << "," << QueryTime_LCA_512[i] << ","
					<< QueryTime_noLCA_1024[i] << "," << QueryTime_LCA_1024[i] << "\n";
			}
			queryresult.close();
		}

	}

	void Graph_D_H::Graph::answerRealNYQuery()
	{
		uploadRealQuery();
		translateQuery();
		int endTime = 500000;
		bool seqContinue = true;
		bool MTContinue_4 = true;
		bool MTContinue_6 = true;
		bool MTContinue_8 = true;
		bool MTContinue_10 = true;
		bool MTContinue_12 = true;
		bool MTContinue_14 = true;
		bool MTContinue_16 = true;
		for (int i = 10000; i <= queryMaxSize; i += queryStep) {

			if (seqContinue) {
				long long int seqtime = H2Hquery_bunch(i);
				QueryTime_seq.push_back(seqtime);
				seqContinue = (seqtime < endTime);
			}
			else {
				QueryTime_seq.push_back(endTime);
			}

			if (MTContinue_4) {
				threadNumber = 4;
				long long int seqtime = H2Hquery_bunch_MultiThread(i);
				QueryTime_MT_4.push_back(seqtime);
				MTContinue_4 = (seqtime < endTime);
			}
			else {
				QueryTime_MT_4.push_back(endTime);
			}

			if (MTContinue_6) {
				threadNumber = 6;
				long long int seqtime = H2Hquery_bunch_MultiThread(i);
				QueryTime_MT_6.push_back(seqtime);
				MTContinue_6 = (seqtime < endTime);
			}
			else {
				QueryTime_MT_6.push_back(endTime);
			}

			if (MTContinue_8) {
				threadNumber = 8;
				long long int seqtime = H2Hquery_bunch_MultiThread(i);
				QueryTime_MT_8.push_back(seqtime);
				MTContinue_8 = (seqtime < endTime);
			}
			else {
				QueryTime_MT_8.push_back(endTime);
			}

			if (MTContinue_10) {
				threadNumber = 10;
				long long int seqtime = H2Hquery_bunch_MultiThread(i);
				QueryTime_MT_10.push_back(seqtime);
				MTContinue_10 = (seqtime < endTime);
			}
			else {
				QueryTime_MT_10.push_back(endTime);
			}

			if (MTContinue_12) {
				threadNumber = 12;
				long long int seqtime = H2Hquery_bunch_MultiThread(i);
				QueryTime_MT_12.push_back(seqtime);
				MTContinue_12 = (seqtime < endTime);
			}
			else {
				QueryTime_MT_12.push_back(endTime);
			}

			if (MTContinue_14) {
				threadNumber = 14;
				long long int seqtime = H2Hquery_bunch_MultiThread(i);
				QueryTime_MT_14.push_back(seqtime);
				MTContinue_14 = (seqtime < endTime);
			}
			else {
				QueryTime_MT_14.push_back(endTime);
			}

			if (MTContinue_16) {
				threadNumber = 16;
				long long int seqtime = H2Hquery_bunch_MultiThread(i);
				QueryTime_MT_16.push_back(seqtime);
				MTContinue_16 = (seqtime < endTime);
			}
			else {
				QueryTime_MT_16.push_back(endTime);
			}
			//QueryTime_seq.push_back(H2Hquery_bunch(i));
			//QueryTime_MT.push_back(H2Hquery_bunch_MultiThread(i));
			cudaThreadNum = 64;
			QueryTime_noLCA_64.push_back(H2Hquery_bunch_noLCA_D(i));
			QueryTime_LCA_64.push_back(H2Hquery_bunch_UsingLCA_D(i));

			cudaThreadNum = 128;
			QueryTime_noLCA_128.push_back(H2Hquery_bunch_noLCA_D(i));
			QueryTime_LCA_128.push_back(H2Hquery_bunch_UsingLCA_D(i));

			cudaThreadNum = 256;
			QueryTime_noLCA_256.push_back(H2Hquery_bunch_noLCA_D(i));
			QueryTime_LCA_256.push_back(H2Hquery_bunch_UsingLCA_D(i));

			cudaThreadNum = 512;
			QueryTime_noLCA_512.push_back(H2Hquery_bunch_noLCA_D(i));
			QueryTime_LCA_512.push_back(H2Hquery_bunch_UsingLCA_D(i));

			cudaThreadNum = 1024;
			QueryTime_noLCA_1024.push_back(H2Hquery_bunch_noLCA_D(i));
			QueryTime_LCA_1024.push_back(H2Hquery_bunch_UsingLCA_D(i));
		}
		string resultFile = "./QueryResult/" + graphName + "/PTHeight-" + to_string(TreeHeight) + "-ChangeHeight-" + to_string(changeHeight) + "-real.csv";
		fstream queryresult(resultFile, ios::app | ios::in | ios::out);
		if (queryresult.is_open()) {
			//queryresult << "CPUThreadNum,CUDAThreadNum\n";
			//queryresult <<threadNumber<<","<< cudaThreadNum << "\n";
			//if(queryresult.peek() == std::ifstream::traits_type::eof())
			queryresult << "queryNumber,sequence,multiThread-4,multiThread-6,multiThread-8,multiThread-10,multiThread-12,multiThread-14,multiThread-16,GPUnoLCA-64,GPUwithLCA-64,GPUnoLCA-128,GPUwithLCA-128,GPUnoLCA-256,GPUwithLCA-256,GPUnoLCA-512,GPUwithLCA-512,GPUnoLCA-1024,GPUwithLCA-1024\n";
			for (int i = 0; i < QueryTime_seq.size(); i++) {
				queryresult << 10000 + i * queryStep << "," << QueryTime_seq[i] << "," << QueryTime_MT_4[i] << ","
					<< QueryTime_MT_6[i] << "," << QueryTime_MT_8[i] << "," << QueryTime_MT_10[i] << "," << QueryTime_MT_12[i] << ","
					<< QueryTime_MT_14[i] << "," << QueryTime_MT_16[i] << ","
					<< QueryTime_noLCA_64[i] << "," << QueryTime_LCA_64[i] << ","
					<< QueryTime_noLCA_128[i] << "," << QueryTime_LCA_128[i] << ","
					<< QueryTime_noLCA_256[i] << "," << QueryTime_LCA_256[i] << ","
					<< QueryTime_noLCA_512[i] << "," << QueryTime_LCA_512[i] << ","
					<< QueryTime_noLCA_1024[i] << "," << QueryTime_LCA_1024[i] << "\n";
			}
			queryresult.close();
		}
	}



	void Graph_D_H::Graph::answerRandomQuery_noHub()
	{
		generateQuery();
		translateQuery();

		

		int endTime = 500000;
		bool seqContinue = true;
		bool MTContinue_4 = true;
		bool MTContinue_6 = true;
		bool MTContinue_8 = true;
		bool MTContinue_10 = true;
		bool MTContinue_12 = true;
		bool MTContinue_14 = true;
		bool MTContinue_16 = true;
		for (int i = 10000; i <= queryMaxSize; i += queryStep) {

			if (seqContinue) {
				long long int seqtime = H2Hquery_bunch_noHub(i);
				QueryTime_seq.push_back(seqtime);
				seqContinue = (seqtime < endTime);
			}
			else {
				QueryTime_seq.push_back(endTime);
			}

			if (MTContinue_4) {
				threadNumber = 4;
				long long int seqtime = H2Hquery_bunch_MultiThread_noHub(i);
				QueryTime_MT_4.push_back(seqtime);
				MTContinue_4 = (seqtime < endTime);
			}
			else {
				QueryTime_MT_4.push_back(endTime);
			}

			if (MTContinue_6) {
				threadNumber = 6;
				long long int seqtime = H2Hquery_bunch_MultiThread_noHub(i);
				QueryTime_MT_6.push_back(seqtime);
				MTContinue_6 = (seqtime < endTime);
			}
			else {
				QueryTime_MT_6.push_back(endTime);
			}

			if (MTContinue_8) {
				threadNumber = 8;
				long long int seqtime = H2Hquery_bunch_MultiThread_noHub(i);
				QueryTime_MT_8.push_back(seqtime);
				MTContinue_8 = (seqtime < endTime);
			}
			else {
				QueryTime_MT_8.push_back(endTime);
			}

			if (MTContinue_10) {
				threadNumber = 10;
				long long int seqtime = H2Hquery_bunch_MultiThread_noHub(i);
				QueryTime_MT_10.push_back(seqtime);
				MTContinue_10 = (seqtime < endTime);
			}
			else {
				QueryTime_MT_10.push_back(endTime);
			}

			if (MTContinue_12) {
				threadNumber = 12;
				long long int seqtime = H2Hquery_bunch_MultiThread_noHub(i);
				QueryTime_MT_12.push_back(seqtime);
				MTContinue_12 = (seqtime < endTime);
			}
			else {
				QueryTime_MT_12.push_back(endTime);
			}

			if (MTContinue_14) {
				threadNumber = 14;
				long long int seqtime = H2Hquery_bunch_MultiThread_noHub(i);
				QueryTime_MT_14.push_back(seqtime);
				MTContinue_14 = (seqtime < endTime);
			}
			else {
				QueryTime_MT_14.push_back(endTime);
			}

			if (MTContinue_16) {
				threadNumber = 16;
				long long int seqtime = H2Hquery_bunch_MultiThread_noHub(i);
				QueryTime_MT_16.push_back(seqtime);
				MTContinue_16 = (seqtime < endTime);
			}
			else {
				QueryTime_MT_16.push_back(endTime);
			}
			//QueryTime_seq.push_back(H2Hquery_bunch(i));
			//QueryTime_MT.push_back(H2Hquery_bunch_MultiThread(i));
			cudaThreadNum = 64;
			QueryTime_noLCA_64.push_back(H2Hquery_bunch_noLCA_D_noHub(i));
			QueryTime_LCA_64.push_back(H2Hquery_bunch_UsingLCA_D_noHub(i));

			cudaThreadNum = 128;
			QueryTime_noLCA_128.push_back(H2Hquery_bunch_noLCA_D_noHub(i));
			QueryTime_LCA_128.push_back(H2Hquery_bunch_UsingLCA_D_noHub(i));

			cudaThreadNum = 256;
			QueryTime_noLCA_256.push_back(H2Hquery_bunch_noLCA_D_noHub(i));
			QueryTime_LCA_256.push_back(H2Hquery_bunch_UsingLCA_D_noHub(i));

			cudaThreadNum = 512;
			QueryTime_noLCA_512.push_back(H2Hquery_bunch_noLCA_D_noHub(i));
			QueryTime_LCA_512.push_back(H2Hquery_bunch_UsingLCA_D_noHub(i));

			cudaThreadNum = 1024;
			QueryTime_noLCA_1024.push_back(H2Hquery_bunch_noLCA_D_noHub(i));
			QueryTime_LCA_1024.push_back(H2Hquery_bunch_UsingLCA_D_noHub(i));
		}
		string resultFile = "./QueryResult/" + graphName + "/PTHeight-" + to_string(TreeHeight) + "-ChangeHeight-" + to_string(changeHeight) + "-random.csv";
		fstream queryresult(resultFile, ios::app | ios::in | ios::out);
		if (queryresult.is_open()) {
			//queryresult << "CPUThreadNum,CUDAThreadNum\n";
			//queryresult <<threadNumber<<","<< cudaThreadNum << "\n";
			//if(queryresult.peek() == std::ifstream::traits_type::eof())
			queryresult << "queryNumber,sequence,multiThread-4,multiThread-6,multiThread-8,multiThread-10,multiThread-12,multiThread-14,multiThread-16,GPUnoLCA-64,GPUwithLCA-64,GPUnoLCA-128,GPUwithLCA-128,GPUnoLCA-256,GPUwithLCA-256,GPUnoLCA-512,GPUwithLCA-512,GPUnoLCA-1024,GPUwithLCA-1024\n";
			for (int i = 0; i < QueryTime_seq.size(); i++) {
				queryresult << 10000 + i * queryStep << "," << QueryTime_seq[i] << "," << QueryTime_MT_4[i] << ","
					<< QueryTime_MT_6[i] << "," << QueryTime_MT_8[i] << "," << QueryTime_MT_10[i] << "," << QueryTime_MT_12[i] << ","
					<< QueryTime_MT_14[i] << "," << QueryTime_MT_16[i] << ","
					<< QueryTime_noLCA_64[i] << "," << QueryTime_LCA_64[i] << ","
					<< QueryTime_noLCA_128[i] << "," << QueryTime_LCA_128[i] << ","
					<< QueryTime_noLCA_256[i] << "," << QueryTime_LCA_256[i] << ","
					<< QueryTime_noLCA_512[i] << "," << QueryTime_LCA_512[i] << ","
					<< QueryTime_noLCA_1024[i] << "," << QueryTime_LCA_1024[i] << "\n";
			}
			queryresult.close();
		}

	}

	void Graph_D_H::Graph::answerRealNYQuery_noHub()
	{
		uploadRealQuery();
		translateQuery();
		int endTime = 500000;
		bool seqContinue = true;
		bool MTContinue_4 = true;
		bool MTContinue_6 = true;
		bool MTContinue_8 = true;
		bool MTContinue_10 = true;
		bool MTContinue_12 = true;
		bool MTContinue_14 = true;
		bool MTContinue_16 = true;
		for (int i = 10000; i <= queryMaxSize; i += queryStep) {

			if (seqContinue) {
				long long int seqtime = H2Hquery_bunch_noHub(i);
				QueryTime_seq.push_back(seqtime);
				seqContinue = (seqtime < endTime);
			}
			else {
				QueryTime_seq.push_back(endTime);
			}

			if (MTContinue_4) {
				threadNumber = 4;
				long long int seqtime = H2Hquery_bunch_MultiThread_noHub(i);
				QueryTime_MT_4.push_back(seqtime);
				MTContinue_4 = (seqtime < endTime);
			}
			else {
				QueryTime_MT_4.push_back(endTime);
			}

			if (MTContinue_6) {
				threadNumber = 6;
				long long int seqtime = H2Hquery_bunch_MultiThread_noHub(i);
				QueryTime_MT_6.push_back(seqtime);
				MTContinue_6 = (seqtime < endTime);
			}
			else {
				QueryTime_MT_6.push_back(endTime);
			}

			if (MTContinue_8) {
				threadNumber = 8;
				long long int seqtime = H2Hquery_bunch_MultiThread_noHub(i);
				QueryTime_MT_8.push_back(seqtime);
				MTContinue_8 = (seqtime < endTime);
			}
			else {
				QueryTime_MT_8.push_back(endTime);
			}

			if (MTContinue_10) {
				threadNumber = 10;
				long long int seqtime = H2Hquery_bunch_MultiThread_noHub(i);
				QueryTime_MT_10.push_back(seqtime);
				MTContinue_10 = (seqtime < endTime);
			}
			else {
				QueryTime_MT_10.push_back(endTime);
			}

			if (MTContinue_12) {
				threadNumber = 12;
				long long int seqtime = H2Hquery_bunch_MultiThread_noHub(i);
				QueryTime_MT_12.push_back(seqtime);
				MTContinue_12 = (seqtime < endTime);
			}
			else {
				QueryTime_MT_12.push_back(endTime);
			}

			if (MTContinue_14) {
				threadNumber = 14;
				long long int seqtime = H2Hquery_bunch_MultiThread_noHub(i);
				QueryTime_MT_14.push_back(seqtime);
				MTContinue_14 = (seqtime < endTime);
			}
			else {
				QueryTime_MT_14.push_back(endTime);
			}

			if (MTContinue_16) {
				threadNumber = 16;
				long long int seqtime = H2Hquery_bunch_MultiThread_noHub(i);
				QueryTime_MT_16.push_back(seqtime);
				MTContinue_16 = (seqtime < endTime);
			}
			else {
				QueryTime_MT_16.push_back(endTime);
			}
			//QueryTime_seq.push_back(H2Hquery_bunch(i));
			//QueryTime_MT.push_back(H2Hquery_bunch_MultiThread(i));
			cudaThreadNum = 64;
			QueryTime_noLCA_64.push_back(H2Hquery_bunch_noLCA_D_noHub(i));
			QueryTime_LCA_64.push_back(H2Hquery_bunch_UsingLCA_D_noHub(i));

			cudaThreadNum = 128;
			QueryTime_noLCA_128.push_back(H2Hquery_bunch_noLCA_D_noHub(i));
			QueryTime_LCA_128.push_back(H2Hquery_bunch_UsingLCA_D_noHub(i));

			cudaThreadNum = 256;
			QueryTime_noLCA_256.push_back(H2Hquery_bunch_noLCA_D_noHub(i));
			QueryTime_LCA_256.push_back(H2Hquery_bunch_UsingLCA_D_noHub(i));

			cudaThreadNum = 512;
			QueryTime_noLCA_512.push_back(H2Hquery_bunch_noLCA_D_noHub(i));
			QueryTime_LCA_512.push_back(H2Hquery_bunch_UsingLCA_D_noHub(i));

			cudaThreadNum = 1024;
			QueryTime_noLCA_1024.push_back(H2Hquery_bunch_noLCA_D_noHub(i));
			QueryTime_LCA_1024.push_back(H2Hquery_bunch_UsingLCA_D_noHub(i));
		}
		string resultFile = "./QueryResult/" + graphName + "/PTHeight-" + to_string(TreeHeight) + "-ChangeHeight-" + to_string(changeHeight) + "-real.csv";
		fstream queryresult(resultFile, ios::app | ios::in | ios::out);
		if (queryresult.is_open()) {
			//queryresult << "CPUThreadNum,CUDAThreadNum\n";
			//queryresult <<threadNumber<<","<< cudaThreadNum << "\n";
			//if(queryresult.peek() == std::ifstream::traits_type::eof())
			queryresult << "queryNumber,sequence,multiThread-4,multiThread-6,multiThread-8,multiThread-10,multiThread-12,multiThread-14,multiThread-16,GPUnoLCA-64,GPUwithLCA-64,GPUnoLCA-128,GPUwithLCA-128,GPUnoLCA-256,GPUwithLCA-256,GPUnoLCA-512,GPUwithLCA-512,GPUnoLCA-1024,GPUwithLCA-1024\n";
			for (int i = 0; i < QueryTime_seq.size(); i++) {
				queryresult << 10000 + i * queryStep << "," << QueryTime_seq[i] << "," << QueryTime_MT_4[i] << ","
					<< QueryTime_MT_6[i] << "," << QueryTime_MT_8[i] << "," << QueryTime_MT_10[i] << "," << QueryTime_MT_12[i] << ","
					<< QueryTime_MT_14[i] << "," << QueryTime_MT_16[i] << ","
					<< QueryTime_noLCA_64[i] << "," << QueryTime_LCA_64[i] << ","
					<< QueryTime_noLCA_128[i] << "," << QueryTime_LCA_128[i] << ","
					<< QueryTime_noLCA_256[i] << "," << QueryTime_LCA_256[i] << ","
					<< QueryTime_noLCA_512[i] << "," << QueryTime_LCA_512[i] << ","
					<< QueryTime_noLCA_1024[i] << "," << QueryTime_LCA_1024[i] << "\n";
			}
			queryresult.close();
		}
	}
}