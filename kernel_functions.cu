#include "hip/hip_runtime.h"
#pragma once
#include"kernel_functions.cuh"
#include<iostream>
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include<thrust/device_vector.h>
#include<thrust/host_vector.h>

#define CUDA_CHECK_ERROR() { \
    hipError_t err = hipGetLastError(); \
    if (err != hipSuccess) { \
        std::cerr << "CUDA error: " << hipGetErrorString(err) << " at line " << __LINE__ << std::endl; \
        exit(EXIT_FAILURE); \
    } \
}

using namespace std;

namespace tools {
    __device__ 
    void bobbleSortPairs(Graph_D_H::TDrank* array, int left, int right)
    {
        for (int i = left; i <= right; i++) {
            //bool swapped = false;

            for (int j = left; j <= right - 1; j++) {
                if (array[j + 1] < array[j]) {
                    Graph_D_H::TDrank temp = array[j];
                    array[j] = array[j + 1];
                    array[j + 1] = temp;

                    //swapped = true;
                }
            }

            //if (!swapped) {
            //    break;
            //}
        }
    }

    __device__
        void bobbleSortPairs_2(Graph_D_H::TDrank* array, int left, int right)
    {
        for (int i = left; i <= right; i++) {
            //bool swapped = false;

            for (int j = left; j <= right - 1; j++) {
                if (array[j + 1] < array[j]) {
                    Graph_D_H::TDrank temp = array[j];
                    array[j] = array[j + 1];
                    array[j + 1] = temp;

                    //swapped = true;
                }
            }

            //if (!swapped) {
               // break;
            //}
        }
    }

    __device__
        void bobbleSortmyPair(Graph_D_H::myPair<int>* array, int left, int right)
    {
        for (int i = left; i <= right; i++) {
            bool swapped = false;

            for (int j = left; j <= right - 1; j++) {
                if (array[j + 1] < array[j]) {
                    Graph_D_H::myPair<int> temp = array[j];
                    array[j] = array[j + 1];
                    array[j + 1] = temp;

                    swapped = true;
                }
            }

            if (!swapped) {
                break;
            }
        }
    }

    __device__ int partition(Graph_D_H::TDrank* array, int low, int high) {
        Graph_D_H::TDrank pivot = array[high];
        int i = (low - 1);

        for (int j = low; j <= high - 1; j++) {
            if (array[j] <  pivot) {
                i++;
                Graph_D_H::TDrank temp = array[i];
                array[i] = array[j];
                array[j] = temp;
            }
        }

        Graph_D_H::TDrank temp = array[i + 1];
        array[i + 1] = array[high];
        array[high] = temp;
        return (i + 1);
    }

    __device__ void quickSort(Graph_D_H::TDrank* array, int low, int high) {
        if (low < high) {
            int pi = partition(array, low, high);
            quickSort(array, low, pi - 1);
            quickSort(array, pi + 1, high);
        }
    }

}

__global__
void sets(int s)
{
    //int index = threadIdx.x;
    printf("cuda kernel in\n");
}

void tests()
{
    sets<<<1,1>>>(1);
    hipDeviceSynchronize();
}

void checkCudaError(hipError_t err) {
    if (err != hipSuccess) {
        std::cerr << "CUDA Error: " << hipGetErrorString(err) << std::endl;
        exit(err);
    }
}


__global__
void makeCHPerHeight_D(int lowestIndexStart, int lowestIndexEnd,
    Graph_D_H::TDrank* nonAdjcentNode,
    int* ID_hash, 
    Graph_D_H::myPair<int>* CHTreeHash, 
    Graph_D_H::myPair<int>* NANHash,
    Graph_D_H::myPair<int>* CHTree, 
    bool* visited)
{
    unsigned int tid = threadIdx.x + (blockDim.x * blockIdx.x);
    if (tid > lowestIndexEnd - lowestIndexStart)
        return;
    int nonAdjStartIndex = NANHash[tid + lowestIndexStart].first;
    const int TDNodeSize = NANHash[tid + lowestIndexStart].NodeID;

    for (int j = nonAdjStartIndex; j < nonAdjStartIndex + TDNodeSize; j++)
    {
        //tools::quickSort(nonAdjcentNode, j, nonAdjStartIndex + TDNodeSize -1);
        tools::bobbleSortPairs(nonAdjcentNode, j, nonAdjStartIndex + TDNodeSize - 1);
        //for (int it = j; it < nonAdjStartIndex + TDNodeSize; it++) {

        //}
        int TDNodeNow = nonAdjcentNode[j].second;
        visited[TDNodeNow] = true;
        int CHTreeHash_index = ID_hash[TDNodeNow];
        int CHTree_Index = CHTreeHash[CHTreeHash_index].NodeID;
        int CHTree_ActualSize = CHTreeHash[CHTreeHash_index].first;
        //int act_empty = firstempty[CHTreeHash_index];
        int act = 1;
        tools::bobbleSortmyPair(CHTree,CHTree_Index,CHTree_Index + CHTree_ActualSize - 1);
        for (int k = CHTree_Index + 1; k < CHTree_Index + CHTree_ActualSize; k++)
        {
            act++;
            if (CHTree[k].first == CHTree[k - 1].first && CHTree[k].first != INT_MAX)
            {
                if (CHTree[k - 1].second < CHTree[k].second)
                {
                    CHTree[k].second = CHTree[k - 1].second;
                    CHTree[k].NodeID = CHTree[k - 1].NodeID;
                }
                CHTree[k - 1].first = INT_MAX;
                act--;
            }
           // CHTree[k].first == INT_MAX;
        }
        tools::bobbleSortmyPair(CHTree, CHTree_Index, CHTree_Index + CHTree_ActualSize - 1);
        for (int k = CHTree_Index; k < CHTree_Index + CHTree_ActualSize; k++)
        {
            if (CHTree[k].first == INT_MAX)
            {
                break;
            }
            CHTreeHash[CHTreeHash_index].first = k - CHTree_Index + 1;
        }
        CHTree_ActualSize = CHTreeHash[CHTreeHash_index].first;

        for (int k = CHTree_Index; k < CHTree_Index + CHTree_ActualSize; k++)
        {
            int outpoint = CHTree[k].first;
            int outpointCHTreeHash_index = ID_hash[outpoint];
            int outPoint_CHTree_index = CHTreeHash[outpointCHTreeHash_index].NodeID;
            int outPoint_CHTree_actualSize = CHTreeHash[outpointCHTreeHash_index].first;
            for (int q = CHTreeHash[outpointCHTreeHash_index].NodeID; q < CHTreeHash[outpointCHTreeHash_index].NodeID + CHTreeHash[outpointCHTreeHash_index].first; q++)
            {
                if (CHTree[q].first == TDNodeNow)
                {
                    CHTree[q].first = INT_MAX;
                    //tools::bobbleSortmyPair(CHTree, outPoint_CHTree_index, outPoint_CHTree_index + outPoint_CHTree_actualSize - 1);
                    //CHTreeHash[outpointCHTreeHash_index].first--;
                    //break;
                }
            }
            tools::bobbleSortmyPair(CHTree, outPoint_CHTree_index, outPoint_CHTree_index + outPoint_CHTree_actualSize - 1);
            for (int k = CHTreeHash[outpointCHTreeHash_index].NodeID; k < outPoint_CHTree_index + outPoint_CHTree_actualSize; k++)
            {
                if (CHTree[k].first == INT_MAX)
                {
                    break;
                }
                CHTreeHash[outpointCHTreeHash_index].first = k - CHTreeHash[outpointCHTreeHash_index].NodeID + 1;
            }
        }
        //add left->right and right -> left
        for (int k = CHTree_Index; k < CHTree_Index + CHTree_ActualSize; k++)
        {
            //if (CHTree[k].first == INT_MAX)
               // continue;
            int leftID = CHTree[k].first;
            for (int q = k + 1; q < CHTree_Index + CHTree_ActualSize; q++)
            {
               // if (CHTree[q].first == INT_MAX)
                   // continue;
                int rightID = CHTree[q].first;
                if (leftID == rightID)
                    continue;
                int length = CHTree[k].second + CHTree[q].second;
                bool leftHaveRight = false;
               
                for (int p = CHTreeHash[ID_hash[leftID]].NodeID; p < CHTreeHash[ID_hash[leftID]].NodeID + CHTreeHash[ID_hash[leftID]].first; p++)
                {
                    if (CHTree[p].first == rightID)
                    {
                        leftHaveRight = true;
                        if (CHTree[p].second > length)
                        {
                            CHTree[p].second = length;
                            CHTree[p].NodeID = TDNodeNow;
                        }
                        break;
                    }
                }
                for (int p = CHTreeHash[ID_hash[rightID]].NodeID; p < CHTreeHash[ID_hash[rightID]].NodeID + CHTreeHash[ID_hash[rightID]].first; p++)
                {
                    if (CHTree[p].first == leftID)
                    {
                        if (CHTree[p].second > length)
                        {
                            CHTree[p].second = length;
                            CHTree[p].NodeID = TDNodeNow;
                        }
                        break;
                    }
                }
                if (leftHaveRight)
                    continue;
                else
                {
                    //left->right

                    //no such link
                    //left->right
                    //int leftStartIndex = CHTreeHash[ID_hash[leftID]].NodeID;
                    int emptyPlace = CHTreeHash[ID_hash[leftID]].NodeID + CHTreeHash[ID_hash[leftID]].first;
                    CHTree[emptyPlace].NodeID = TDNodeNow;
                    CHTree[emptyPlace].first = rightID;
                    CHTree[emptyPlace].second = length;
                    CHTreeHash[ID_hash[leftID]].first++;

                    //right->left
                    //leftStartIndex = CHTreeHash[ID_hash[rightID]].NodeID;
                    emptyPlace = CHTreeHash[ID_hash[rightID]].NodeID + CHTreeHash[ID_hash[rightID]].first;
                    CHTree[emptyPlace].NodeID = TDNodeNow;
                    CHTree[emptyPlace].first = leftID;
                    CHTree[emptyPlace].second = length;
                    CHTreeHash[ID_hash[rightID]].first++;
                }
                
            }
        }


        //change vertex degree
        for (int k = j + 1; k < nonAdjStartIndex + TDNodeSize; k++)
        {

            nonAdjcentNode[k].first = CHTreeHash[ID_hash[nonAdjcentNode[k].second]].first;
            for (int q = CHTree_Index; q < CHTree_Index + CHTree_ActualSize; q++)
            {
                if (CHTree[q].first == INT_MAX)
                    continue;
                //int outpoint = CHTree[q].first;
                if (nonAdjcentNode[k].second == CHTree[q].first)
                {
                    nonAdjcentNode[k].third = (nonAdjcentNode[j].third + 1 > nonAdjcentNode[k].third) ? nonAdjcentNode[j].third + 1 : nonAdjcentNode[k].third;
                    break;
                }
            }
        }
    }
}


__global__
void makeCHPerHeight_D_2(int lowestIndexStart, int lowestIndexEnd, 
    Graph_D_H::TDrank* nonAdjcentNode,
    int* ID_hash, 
    Graph_D_H::myPair<int>* CHTreeHash, 
    Graph_D_H::myPair<int>* NANHash,
    Graph_D_H::myPair<int>* CHTree, 
    bool* visited, 
    Graph_D_H::pairs* CHInfo)
{
    unsigned int tid = threadIdx.x + (blockDim.x * blockIdx.x);
    if (tid > lowestIndexEnd - lowestIndexStart)
        return;
    int nonAdjStartIndex = NANHash[tid + lowestIndexStart].first;
    const int TDNodeSize = NANHash[tid + lowestIndexStart].NodeID;

    for (int j = nonAdjStartIndex; j < nonAdjStartIndex + TDNodeSize; j++)
    {
        //tools::quickSort(nonAdjcentNode, j, nonAdjStartIndex + TDNodeSize -1);
        tools::bobbleSortPairs(nonAdjcentNode, j, nonAdjStartIndex + TDNodeSize - 1);
        int TDNodeNow = nonAdjcentNode[j].second;

        visited[TDNodeNow] = true;
        int CHTreeHash_index = ID_hash[TDNodeNow];
        int CHTree_Index = CHTreeHash[CHTreeHash_index].NodeID;
        int CHTree_ActualSize = CHTreeHash[CHTreeHash_index].first;
        //int act_empty = firstempty[CHTreeHash_index];

        //reorder
        tools::bobbleSortmyPair(CHTree, CHTree_Index, CHTree_Index + CHTree_ActualSize - 1);
        for (int k = CHTree_Index + 1; k < CHTree_Index + CHTree_ActualSize; k++)
        {

            if (CHTree[k].first == CHTree[k - 1].first && CHTree[k].first != INT_MAX)
            {
                if (CHTree[k - 1].second < CHTree[k].second)
                {
                    CHTree[k].second = CHTree[k - 1].second;
                    CHTree[k].NodeID = CHTree[k - 1].NodeID;
                }
                CHTree[k - 1].first = INT_MAX;
            }
        }
        tools::bobbleSortmyPair(CHTree, CHTree_Index, CHTree_Index + CHTree_ActualSize - 1);
        for (int k = CHTree_Index; k < CHTree_Index + CHTree_ActualSize; k++)
        {
            if (CHTree[k].first == INT_MAX)
            {
                break;
            }
            CHTreeHash[CHTreeHash_index].first = k - CHTree_Index + 1;
        }
        CHTree_ActualSize = CHTreeHash[CHTreeHash_index].first;

        //delete
        for (int k = CHTree_Index; k < CHTree_Index + CHTree_ActualSize; k++)
        {
            int outpoint = CHTree[k].first;
            int outpointCHTreeHash_index = ID_hash[outpoint];
            int outPoint_CHTree_index = CHTreeHash[outpointCHTreeHash_index].NodeID;
            int outPoint_CHTree_actualSize = CHTreeHash[outpointCHTreeHash_index].first;
            for (int q = CHTreeHash[outpointCHTreeHash_index].NodeID; q < CHTreeHash[outpointCHTreeHash_index].NodeID + CHTreeHash[outpointCHTreeHash_index].first; q++)
            {
                if (CHTree[q].first == TDNodeNow)
                {
                    CHTree[q].first = INT_MAX;
                    //tools::bobbleSortmyPair(CHTree, outPoint_CHTree_index, outPoint_CHTree_index + outPoint_CHTree_actualSize - 1);
                    //CHTreeHash[outpointCHTreeHash_index].first--;
                    //break;
                }
            }
            tools::bobbleSortmyPair(CHTree, outPoint_CHTree_index, outPoint_CHTree_index + outPoint_CHTree_actualSize - 1);
            for (int k = CHTreeHash[outpointCHTreeHash_index].NodeID; k < outPoint_CHTree_index + outPoint_CHTree_actualSize; k++)
            {
                if (CHTree[k].first == INT_MAX)
                {
                    break;
                }
                CHTreeHash[outpointCHTreeHash_index].first = k - CHTreeHash[outpointCHTreeHash_index].NodeID + 1;
            }
            
        }
        //add left->right and right -> left
        for (int k = CHTree_Index; k < CHTree_Index + CHTree_ActualSize; k++)
        {
            //if (CHTree[k].first == INT_MAX)
               // continue;
            int leftID = CHTree[k].first;
            for (int q = k + 1; q < CHTree_Index + CHTree_ActualSize; q++)
            {
                // if (CHTree[q].first == INT_MAX)
                    // continue;
                int rightID = CHTree[q].first;
                if (leftID == rightID)
                    continue;
                int length = CHTree[k].second + CHTree[q].second;
                bool leftHaveRight = false;

                for (int p = CHTreeHash[ID_hash[leftID]].NodeID; p < CHTreeHash[ID_hash[leftID]].NodeID + CHTreeHash[ID_hash[leftID]].first; p++)
                {
                    if (CHTree[p].first == rightID)
                    {
                        leftHaveRight = true;
                        if (CHTree[p].second > length)
                        {
                            CHTree[p].second = length;
                            CHTree[p].NodeID = TDNodeNow;
                        }
                        break;
                    }
                }
                for (int p = CHTreeHash[ID_hash[rightID]].NodeID; p < CHTreeHash[ID_hash[rightID]].NodeID + CHTreeHash[ID_hash[rightID]].first; p++)
                {
                    if (CHTree[p].first == leftID)
                    {
                        if (CHTree[p].second > length)
                        {
                            CHTree[p].second = length;
                            CHTree[p].NodeID = TDNodeNow;
                        }
                        break;
                    }
                }
                if (leftHaveRight)
                    continue;
                else
                {
                    //left->right

                    //no such link
                    //left->right
                    //int leftStartIndex = CHTreeHash[ID_hash[leftID]].NodeID;
                    int emptyPlace = CHTreeHash[ID_hash[leftID]].NodeID + CHTreeHash[ID_hash[leftID]].first;
                    CHTree[emptyPlace].NodeID = TDNodeNow;
                    CHTree[emptyPlace].first = rightID;
                    CHTree[emptyPlace].second = length;
                    CHTreeHash[ID_hash[leftID]].first++;

                    //right->left
                    //leftStartIndex = CHTreeHash[ID_hash[rightID]].NodeID;
                    emptyPlace = CHTreeHash[ID_hash[rightID]].NodeID + CHTreeHash[ID_hash[rightID]].first;
                    CHTree[emptyPlace].NodeID = TDNodeNow;
                    CHTree[emptyPlace].first = leftID;
                    CHTree[emptyPlace].second = length;
                    CHTreeHash[ID_hash[rightID]].first++;
                }

            }
        }


        //change vertex degree

        for (int q = CHTree_Index; q < CHTree_Index + CHTree_ActualSize; q++)
        {
            if (CHTree[q].first == INT_MAX)
                continue;
            int outpoint = CHTree[q].first;
            int HeightNow = nonAdjcentNode[j].third;
            CHInfo[outpoint].first = CHTreeHash[ID_hash[outpoint]].first;// global degree
            CHInfo[outpoint].second = (HeightNow + 1 > CHInfo[outpoint].second) ? HeightNow + 1 : CHInfo[outpoint].second;//global height
            
        }
        for (int k = j + 1; k < nonAdjStartIndex + TDNodeSize; k++)
        {
            nonAdjcentNode[k].first = CHTreeHash[ID_hash[nonAdjcentNode[k].second]].first;//degree
            nonAdjcentNode[k].third = CHInfo[nonAdjcentNode[k].second].second; //height
        }

    }
}

void makeCHPerHeight(int TreeHeight,int cudaThreadNum,int TreeHeightTarget,
    thrust::device_vector<Graph_D_H::TDrank>& nonAdjcentNode_D,
    thrust::device_vector<int>& ID_hash_D, 
    thrust::device_vector<Graph_D_H::myPair<int>>& CHTreeHash_D,
    thrust::device_vector<Graph_D_H::myPair<int>>& CHTree_D,
    thrust::device_vector<Graph_D_H::myPair<int>>& NANHash_D,
    thrust::device_vector<bool>& visited_D)
{
    int tempHeight = TreeHeight;
    Graph_D_H::TDrank* nonAdjcentNode = thrust::raw_pointer_cast(nonAdjcentNode_D.data());
    int* ID_hash = thrust::raw_pointer_cast(ID_hash_D.data());
    Graph_D_H::myPair<int>* CHTreeHash = thrust::raw_pointer_cast(CHTreeHash_D.data());
    Graph_D_H::myPair<int>* CHTree = thrust::raw_pointer_cast(CHTree_D.data());
    Graph_D_H::myPair<int>* NANHash = thrust::raw_pointer_cast(NANHash_D.data());
    bool* visited = thrust::raw_pointer_cast(visited_D.data());
   // Graph_D_H::time_Mine time;
    //cout << "TD start " << endl;
    //time.updateStart();
    while (tempHeight > TreeHeightTarget)
    {
        int lowestIndexStart = (int)(std::pow(2, tempHeight - 1) - 1);
        int lowestIndexEnd = (int)std::pow(2, tempHeight) - 2;
       // cout << "height now : " << tempHeight << endl;
       // Graph_D_H::time_Mine time1;

       // time1.updateStart();
        tempHeight--;
        int blockSize = ((lowestIndexStart + 1) / cudaThreadNum) + 1;

        makeCHPerHeight_D<<<blockSize,cudaThreadNum>>>(lowestIndexStart, lowestIndexEnd, nonAdjcentNode, ID_hash, CHTreeHash, NANHash, CHTree,visited);
        hipDeviceSynchronize();
       //time1.updateEnd();
       // cout << "time using: " << time1.get_microsecond_duration() << endl;
    }
   // time.updateEnd();
   // cout << " using time: " << time.get_microsecond_duration() << endl;
}


void makeCHPerHeight_2(int TreeHeight, int cudaThreadNum, int TreeHeightTarget,
    thrust::device_vector<Graph_D_H::TDrank>& nonAdjcentNode_D,
    thrust::device_vector<int>& ID_hash_D,
    thrust::device_vector<Graph_D_H::myPair<int>>& CHTreeHash_D,
    thrust::device_vector<Graph_D_H::myPair<int>>& CHTree_D,
    thrust::device_vector<Graph_D_H::myPair<int>>& NANHash_D,
    thrust::device_vector<bool>& visited_D,
    thrust::device_vector<Graph_D_H::pairs>& CHInfo_D)
{
    int tempHeight = TreeHeight;
    Graph_D_H::TDrank* nonAdjcentNode = thrust::raw_pointer_cast(nonAdjcentNode_D.data());
    int* ID_hash = thrust::raw_pointer_cast(ID_hash_D.data());
    Graph_D_H::myPair<int>* CHTreeHash = thrust::raw_pointer_cast(CHTreeHash_D.data());
    Graph_D_H::myPair<int>* CHTree = thrust::raw_pointer_cast(CHTree_D.data());
    Graph_D_H::myPair<int>* NANHash = thrust::raw_pointer_cast(NANHash_D.data());
    bool* visited = thrust::raw_pointer_cast(visited_D.data());
    Graph_D_H::pairs* CHInfo = thrust::raw_pointer_cast(CHInfo_D.data());
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    while (tempHeight > TreeHeightTarget)
    {
        int lowestIndexStart = (int)std::pow(2, tempHeight - 1) - 1;
        int lowestIndexEnd = (int)std::pow(2, tempHeight) - 2;
        tempHeight--;
        int blockSize = ((lowestIndexStart + 1) / cudaThreadNum) + 1;
        hipEventRecord(start);
        makeCHPerHeight_D_2 <<<blockSize, cudaThreadNum >>> (lowestIndexStart, lowestIndexEnd, nonAdjcentNode, ID_hash, CHTreeHash, NANHash, CHTree, visited, CHInfo);
        hipEventRecord(stop);
        hipEventSynchronize(stop);
    }
    hipEventDestroy(start);
    hipEventDestroy(stop);
}


__global__
void makeH2HLabel_D(Graph_D_H::pairs* H2H_startIndex,
    Graph_D_H::H2HLabel* H2H_label,
    int* frontier, int maxSize
)
{
    unsigned int tid = threadIdx.x + (blockDim.x * blockIdx.x);
    if (tid >= maxSize)
        return;
    int ID = frontier[tid];

    int H2HSize = H2H_startIndex[ID].second - 1;
    int IDNowH2Hindex = H2H_startIndex[ID].first;

    for (int k = IDNowH2Hindex + H2HSize - 1; k >= IDNowH2Hindex; k--)
    {
        if (!H2H_label[k].isTreeNode) {
            continue;
        }
        
        for (int q = k - 1; q >= IDNowH2Hindex; q--)
        {
            if (H2H_label[q].isTreeNode) {
                continue;
            }
            int templength = H2H_label[k].dis + H2H_label[q - IDNowH2Hindex + H2H_startIndex[H2H_label[k].Node].first].dis;
            if (H2H_label[q].dis > templength) {
                H2H_label[q].dis = templength;
                H2H_label[q].Hub = H2H_label[k].Node;
            }
           
        }
    }
}

void makeH2HLabel(int tempHeight,int cudaThreadNum,
    thrust::device_vector<Graph_D_H::pairs>& H2H_startIndex_D, 
    thrust::device_vector<Graph_D_H::H2HLabel>& H2H_label_D, 
    thrust::host_vector<int>& Childs,
    thrust::host_vector<Graph_D_H::pairs>& ChildHash,
    thrust::host_vector<int>& frontier
    ) {
    Graph_D_H::H2HLabel* H2H_label = thrust::raw_pointer_cast(H2H_label_D.data());
    Graph_D_H::pairs* H2H_startIndex = thrust::raw_pointer_cast(H2H_startIndex_D.data());
    thrust::host_vector<int> frontier_now = frontier;
   
   //frontier_now.clear();
    int height = tempHeight;

    //cout << "height now: " << height << endl;


    while (!frontier_now.empty()) {
        thrust::host_vector<int> frontier_next;
        int size = (int)frontier_now.size();
        thrust::device_vector<int> frontier_D = frontier_now;
        int* front = thrust::raw_pointer_cast(frontier_D.data());
        int blockSize = (int)size / cudaThreadNum + 1;

        makeH2HLabel_D<<<blockSize,cudaThreadNum>>>(H2H_startIndex, H2H_label, front, size);
        hipDeviceSynchronize();

        //cout << " node construct now: ";
        for (auto& ID : frontier_now) {
            //cout << ID << " ";
            for (int j = ChildHash[ID].second; j < ChildHash[ID].second + ChildHash[ID].first; j++)
            {
                frontier_next.push_back(Childs[j]);
            }
        }
        //cout << endl;
        frontier_now.clear();
        frontier_now = frontier_next;
        frontier_next.clear();
        height++;
    }

}


__global__
void makeH2HLabel_noCommunication_D(Graph_D_H::pairs* H2H_startIndex,
    Graph_D_H::H2HLabel* H2H_label,
    int* TreeBFS,int startIndex,int endIndex
)
{
    unsigned int tid = threadIdx.x + (blockDim.x * blockIdx.x);
    if (tid >= endIndex - startIndex)
        return;
    int ID = TreeBFS[tid + startIndex];

    int H2HSize = H2H_startIndex[ID].second - 1;
    int IDNowH2Hindex = H2H_startIndex[ID].first;

    for (int k = IDNowH2Hindex + H2HSize - 1; k >= IDNowH2Hindex; k--)
    {
        if (!H2H_label[k].isTreeNode) {
            continue; 
        }

        for (int q = k - 1; q >= IDNowH2Hindex; q--)
        {
            //if (H2H_label[q].isTreeNode) {
            //	continue;
            //}
            int templength = H2H_label[k].dis + H2H_label[q - IDNowH2Hindex + H2H_startIndex[H2H_label[k].Node].first].dis;
            if (H2H_label[q].dis > templength) {
                H2H_label[q].dis = templength;
                H2H_label[q].Hub = H2H_label[k].Node;
            }
        }
        for (int q = k + 1; q <= IDNowH2Hindex + H2HSize - 1; q++)
        {
            int templength = H2H_label[k].dis + H2H_label[k - IDNowH2Hindex + H2H_startIndex[H2H_label[q].Node].first].dis;
            if (H2H_label[q].dis > templength) {
                H2H_label[q].dis = templength;
                H2H_label[q].Hub = H2H_label[k].Node;
            }
        }
    }
}

void makeH2HLabel_noCommunication(int tempHeight, int cudaThreadNum,
    thrust::device_vector<Graph_D_H::pairs>& H2H_startIndex_D,
    thrust::device_vector<Graph_D_H::H2HLabel>& H2H_label_D,
    thrust::device_vector<int>& TreeBFS_D,
    thrust::host_vector<int>& TreeHash
) {
    Graph_D_H::H2HLabel* H2H_label = thrust::raw_pointer_cast(H2H_label_D.data());
    Graph_D_H::pairs* H2H_startIndex = thrust::raw_pointer_cast(H2H_startIndex_D.data());
    int* TreeBFS = thrust::raw_pointer_cast(TreeBFS_D.data());
    //int* TreeHash = thrust::raw_pointer_cast(TreeHash_D.data());

    size_t maxHeight = TreeHash.size() -1;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    for (int i = tempHeight; i < maxHeight; i++)
    {
        int startIndex = TreeHash[i];
        int endIndex = TreeHash[i + 1];
        int size = endIndex - startIndex;

        int blockSize = size / cudaThreadNum + 1;
        hipEventRecord(start);
        makeH2HLabel_noCommunication_D <<<blockSize, cudaThreadNum >>>(H2H_startIndex, H2H_label, TreeBFS, startIndex, endIndex);
        hipEventRecord(stop);
        hipEventSynchronize(stop);
    }
    hipEventDestroy(start);
    hipEventDestroy(stop);

}








__global__ void makeH2HLabel_noCommunication_D_noHub_3(
    int64_t* H2H_dis_hash, int* H2H_dis,
    int* TreeBFS_ID, int* TreeBFS_adj, int* TreeBFS_pos, //int* TreeBFS_changeTime,
    int* father,
    int64_t startIndex, int64_t endIndex
) {
    int64_t tid = threadIdx.x + (blockDim.x * blockIdx.x);
    if (tid >= endIndex - startIndex)
        return;

    int nodeID = TreeBFS_ID[tid + startIndex];
    //int adjID = TreeBFS_adj[tid + startIndex];
    int pos = TreeBFS_pos[tid + startIndex];

    int64_t indexNode = H2H_dis_hash[nodeID];
    int64_t index1_adj = H2H_dis_hash[TreeBFS_adj[tid + startIndex]];
    int tempLength_ = H2H_dis[indexNode + pos];
    for (int64_t j = pos - 1; j >= 0; j--) {
        atomicMin(&H2H_dis[indexNode + j], (tempLength_ + H2H_dis[index1_adj + j]));
    }

    //L4 label
    //int fatherID = nodeID;
    //for (int64_t j = (int)(H2H_dis_hash[nodeID + 1] - H2H_dis_hash[nodeID]) - 2; j > pos; j--) {
    //    fatherID = father[fatherID];
    //    atomicMin(&H2H_dis[indexNode + j], (tempLength_ + H2H_dis[H2H_dis_hash[fatherID] + pos]));
    //}
}



double makeH2HLabel_noCommunication_noHub_3(int tempHeight, int cudaThreadNum,
    thrust::device_vector<int64_t>& H2H_dis_hash_D,
    thrust::device_vector<int>& H2H_dis_D,
    thrust::device_vector<int>& TreeBFS_ID_D,
    thrust::device_vector<int>& TreeBFS_adj_D,
    thrust::device_vector<int>& TreeBFS_pos_D,
    //thrust::device_vector<int>& TreeBFS_changeTime_D,
    thrust::device_vector<int>& father_D,

    thrust::host_vector<int64_t>& TreeBFS_Hash
) {
    //cout << "H2H label size: \n";
    //cout << "\t pos hash: " << H2H_pos_hash_D.size() << " pos id: " << H2H_pos_ID_D.size() << " pos: " << H2H_pos_POS_D.size() << endl;
    //cout << "\t dis hash: " << H2H_dis_hash_D.size() << " dis: " << H2H_dis_D.size()<<endl;
    //cout << "TreeBFS size: " << TreeBFS_D.size()<<endl ;
    int64_t* H2H_dis_hash = thrust::raw_pointer_cast(H2H_dis_hash_D.data());
    int* H2H_dis = thrust::raw_pointer_cast(H2H_dis_D.data());
    int* TreeBFS_ID = thrust::raw_pointer_cast(TreeBFS_ID_D.data());
    int* TreeBFS_adj = thrust::raw_pointer_cast(TreeBFS_adj_D.data());
    int* TreeBFS_pos = thrust::raw_pointer_cast(TreeBFS_pos_D.data());
    //int* TreeBFS_changeTime = thrust::raw_pointer_cast(TreeBFS_changeTime_D.data());
    int* father = thrust::raw_pointer_cast(father_D.data());
    hipEvent_t start, stop;
    double Usingtime = 0;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    size_t maxHeight = TreeBFS_Hash.size() - 1;

    for (int i = tempHeight; i < maxHeight; i++)
    {
        int64_t startIndex = TreeBFS_Hash[i];
        int64_t endIndex = TreeBFS_Hash[i + 1];
        int size = endIndex - startIndex;
        int blockSize = size / cudaThreadNum + 1;
        hipEventRecord(start);

        makeH2HLabel_noCommunication_D_noHub_3 << <blockSize, cudaThreadNum >> > (H2H_dis_hash, H2H_dis, TreeBFS_ID, TreeBFS_adj, TreeBFS_pos, father, startIndex, endIndex);

        hipEventRecord(stop);
        hipEventSynchronize(stop);
        float milliseconds = 0;
        hipEventElapsedTime(&milliseconds, start, stop);
        Usingtime += milliseconds * 1000;
        //std::cout << "At height: " << i << " Kernel execution time: " << milliseconds << " ms" << std::endl;
    }
    hipEventDestroy(start);
    hipEventDestroy(stop);
    return Usingtime;
}






__global__ void makeH2HLabel_noCommunication_D_noHub(
    int64_t* H2H_pos_hash, int64_t* H2H_dis_hash, int* H2H_pos_ID, int* H2H_pos_POS, int* H2H_dis, int* father, 
    int* TreeBFS, int startIndex, int endIndex
)
{
    unsigned int tid = threadIdx.x + (blockDim.x * blockIdx.x);
    if (tid >= endIndex - startIndex)
        return;
    //printf(" nodeID: %d\n" , tid);
    int nodeID = TreeBFS[tid + startIndex];
    //if (nodeID == 2) {
       // printf("\t %d size is: %d\n", nodeID, H2H_pos_hash[nodeID + 1] - H2H_pos_hash[nodeID]);
    //}
    for (int64_t temp = H2H_pos_hash[nodeID + 1] - 2; temp >= H2H_pos_hash[nodeID]; temp--) {
        int adjID = H2H_pos_ID[temp];
        int pos = H2H_pos_POS[temp];
        int64_t indexNode = H2H_dis_hash[nodeID];
        int64_t index1_adj = H2H_dis_hash[adjID];
        int tempLength_ = H2H_dis[indexNode + pos];
        for (int64_t j = pos - 1; j >= 0; j--) {
            bool check = H2H_dis[indexNode + j] > tempLength_ + H2H_dis[index1_adj + j];
            H2H_dis[indexNode + j] = check * (tempLength_ + H2H_dis[index1_adj + j]) + (!check) * (H2H_dis[indexNode + j]);
            //if (H2H_dis[indexNode + j] > tempLength_ + H2H_dis[index1_adj + j]) {
            //    H2H_dis[indexNode + j] = tempLength_ + H2H_dis[index1_adj + j];
            //}
        }

        //L4 label
        //int fatherID = nodeID;
        //for (int64_t j = (int)(H2H_dis_hash[nodeID + 1] - H2H_dis_hash[nodeID]) - 2; j > pos; j--) {
        //    fatherID = father[fatherID];
        //    bool check = H2H_dis[indexNode + j] > tempLength_ + H2H_dis[H2H_dis_hash[fatherID] + pos];
        //    H2H_dis[indexNode + j] = check * (tempLength_ + H2H_dis[H2H_dis_hash[fatherID] + pos]) + (!check) * (H2H_dis[indexNode + j]);

        //    //if (H2H_dis[indexNode + j] > tempLength_ + H2H_dis[H2H_dis_hash[fatherID] + pos]) {
        //    //    H2H_dis[indexNode + j] = tempLength_ + H2H_dis[H2H_dis_hash[fatherID] + pos];
        //    //}
        //}
    }
}



double makeH2HLabel_noCommunication_noHub(int tempHeight, int cudaThreadNum,
    thrust::device_vector<int64_t>& H2H_pos_hash_D,
    thrust::device_vector<int64_t>& H2H_dis_hash_D,
    thrust::device_vector<int>& H2H_pos_ID_D,
    thrust::device_vector<int>& H2H_pos_POS_D,
    thrust::device_vector<int>& H2H_dis_D ,
    thrust::device_vector<int>& father_D,
    thrust::device_vector<int>& TreeBFS_D,
    thrust::host_vector<int>& TreeHash
) {
    //cout << "H2H label size: \n";
    //cout << "\t pos hash: " << H2H_pos_hash_D.size() << " pos id: " << H2H_pos_ID_D.size() << " pos: " << H2H_pos_POS_D.size() << endl;
    //cout << "\t dis hash: " << H2H_dis_hash_D.size() << " dis: " << H2H_dis_D.size()<<endl;
    //cout << "TreeBFS size: " << TreeBFS_D.size()<<endl ;

    int64_t* H2H_pos_hash = thrust::raw_pointer_cast(H2H_pos_hash_D.data());
    int64_t* H2H_dis_hash = thrust::raw_pointer_cast(H2H_dis_hash_D.data());
    int* H2H_pos_ID = thrust::raw_pointer_cast(H2H_pos_ID_D.data());
    int* H2H_pos_POS = thrust::raw_pointer_cast(H2H_pos_POS_D.data());
    int* H2H_dis = thrust::raw_pointer_cast(H2H_dis_D.data());
    int* father = thrust::raw_pointer_cast(father_D.data());
    int* TreeBFS = thrust::raw_pointer_cast(TreeBFS_D.data());
    hipEvent_t start, stop;
    double Usingtime = 0;

    hipEventCreate(&start);
    hipEventCreate(&stop);
    size_t maxHeight = TreeHash.size() - 1;
    for (int i = tempHeight; i < maxHeight; i++)
    {
        int startIndex = TreeHash[i];
        int endIndex = TreeHash[i + 1];
        int size = endIndex - startIndex;
        //cout << "at Height: " << i << " and size: " << size << endl;
        int blockSize = size / cudaThreadNum + 1;
        hipEventRecord(start);
        makeH2HLabel_noCommunication_D_noHub << <blockSize, cudaThreadNum >> > (H2H_pos_hash, H2H_dis_hash, H2H_pos_ID, H2H_pos_POS, H2H_dis, father, TreeBFS, startIndex, endIndex);
        //hipDeviceSynchronize();
        hipEventRecord(stop);
        hipEventSynchronize(stop);
        float milliseconds = 0;
        hipEventElapsedTime(&milliseconds, start, stop);
        Usingtime += milliseconds*1000;
        //std::cout <<"At height: "<< i << " Kernel execution time: " << milliseconds << " ms" << std::endl;
    }
    hipEventDestroy(start);
    hipEventDestroy(stop);
    return Usingtime;
}


__global__ void makeH2HLabel_noCommunication_D_noHub_2(
    int64_t* H2H_pos_hash, int64_t* H2H_dis_hash, int* H2H_pos_ID, int* H2H_pos_POS, int* H2H_dis, int* father,
    int* frontier_Hash, int* frontier_ID, int size
)
{
    unsigned int tid = threadIdx.x + (blockDim.x * blockIdx.x);
    if (tid >= size)
        return;
    int start = frontier_Hash[tid];
    int end = frontier_Hash[tid + 1];
    for (int i = start; i < end; i++) {

        int nodeID = frontier_ID[i];
        for (int64_t temp = H2H_pos_hash[nodeID + 1] - 2; temp >= H2H_pos_hash[nodeID]; temp--) {
            int adjID = H2H_pos_ID[temp];
            int pos = H2H_pos_POS[temp];
            int64_t indexNode = H2H_dis_hash[nodeID];
            int64_t index1_adj = H2H_dis_hash[adjID];
            int tempLength_ = H2H_dis[indexNode + pos];
            for (int j = pos - 1; j >= 0; j--) {
                if (H2H_dis[indexNode + j] > tempLength_ + H2H_dis[index1_adj + j]) {
                    H2H_dis[indexNode + j] = tempLength_ + H2H_dis[index1_adj + j];
                }
            }

            //L4 label
            //int fatherID = nodeID;
            //for (int j = (int)(H2H_dis_hash[nodeID + 1] - H2H_dis_hash[nodeID]) - 2; j > pos; j--) {
            //    fatherID = father[fatherID];
            //    if (H2H_dis[indexNode + j] > tempLength_ + H2H_dis[H2H_dis_hash[fatherID] + pos]) {
            //        H2H_dis[indexNode + j] = tempLength_ + H2H_dis[H2H_dis_hash[fatherID] + pos];
            //    }
            //}
        }

    }
}

void makeH2HLabel_noCommunication_noHub_2(int cudaThreadNum,
    thrust::device_vector<int64_t>& H2H_pos_hash_D,
    thrust::device_vector<int64_t>& H2H_dis_hash_D,
    thrust::device_vector<int>& H2H_pos_ID_D,
    thrust::device_vector<int>& H2H_pos_POS_D,
    thrust::device_vector<int>& H2H_dis_D,
    thrust::device_vector<int>& father_D,

    thrust::device_vector<int> frontier_Hash_D,
    thrust::device_vector<int> frontier_ID_D,
    int64_t size
) {
    //cout << "H2H label size: \n";
    //cout << "\t pos hash: " << H2H_pos_hash_D.size() << " pos id: " << H2H_pos_ID_D.size() << " pos: " << H2H_pos_POS_D.size() << endl;
    //cout << "\t dis hash: " << H2H_dis_hash_D.size() << " dis: " << H2H_dis_D.size()<<endl;
    //cout << "TreeBFS size: " << TreeBFS_D.size()<<endl ;
    int64_t* H2H_pos_hash = thrust::raw_pointer_cast(H2H_pos_hash_D.data());
    int64_t* H2H_dis_hash = thrust::raw_pointer_cast(H2H_dis_hash_D.data());
    int* H2H_pos_ID = thrust::raw_pointer_cast(H2H_pos_ID_D.data());
    int* H2H_pos_POS = thrust::raw_pointer_cast(H2H_pos_POS_D.data());
    int* H2H_dis = thrust::raw_pointer_cast(H2H_dis_D.data());
    int* father = thrust::raw_pointer_cast(father_D.data());

    int* frontier_Hash = thrust::raw_pointer_cast(frontier_Hash_D.data());
    int* frontier_ID = thrust::raw_pointer_cast(frontier_ID_D.data());


    int blockSize = size / cudaThreadNum + 1;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    makeH2HLabel_noCommunication_D_noHub_2 << <blockSize, cudaThreadNum >> > (H2H_pos_hash, H2H_dis_hash, H2H_pos_ID, H2H_pos_POS, H2H_dis, father, frontier_Hash, frontier_ID, size);
    //hipDeviceSynchronize();
    hipEventRecord(stop);
    hipEventSynchronize(stop);

        // \BC\C6\CB\E3ʱ\BC\E4
        float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    std::cout << "Kernel execution time: " << milliseconds << " ms" << std::endl;
    hipEventDestroy(start);
    hipEventDestroy(stop);
}







__global__
void H2Hquery_UsingLCA_D(int* x, int* y, int* result, const int querySize, Graph_D_H::pairs* H2H_startIndex, 
    Graph_D_H::H2HLabel* H2H_label,
    int* firstAppare, Graph_D_H::pairs* RMQ_OneLine, int RMQ_Size, int RMQ_Line_Size) {
    unsigned int tid = threadIdx.x + (blockDim.x * blockIdx.x);
    if (querySize <= tid)
        return;
    if (x[tid] == y[tid])
    {
        result[tid] = 0;
        return;
    }
    int FirstA = firstAppare[x[tid]];//position  in  OULA_DFS
    int FirstB = firstAppare[y[tid]];
    int firstA = (FirstA < FirstB) * FirstA + (FirstB <= FirstA) * FirstB;
    int firstB = (FirstA < FirstB) * FirstB + (FirstB <= FirstA) * FirstA;
    int k = (int)(log2((double)(firstB - firstA + 1)));//k means log2(gap);

    bool check = (RMQ_OneLine[firstA * RMQ_Line_Size + k].first <= RMQ_OneLine[(firstB - (1 << k) + 1) * RMQ_Line_Size + k].first);
    int LCA = check * RMQ_OneLine[firstA * RMQ_Line_Size + k].second +
        (!check) * RMQ_OneLine[(firstB - (1 << k) + 1) * RMQ_Line_Size + k].second;

    int checkPointX = H2H_startIndex[x[tid]].first;
    int checkPointY = H2H_startIndex[y[tid]].first;
    int LCApoint = H2H_startIndex[LCA].first;
    int frontier = H2H_startIndex[LCA].second;
    int resu = INT_MAX;
    for (int i = 0; i < frontier; i++)
    {
        if (!H2H_label[LCApoint + i].isTreeNode)
            continue;
        resu = ((H2H_label[checkPointX + i].dis + H2H_label[checkPointY + i].dis) < resu) ?
            (H2H_label[checkPointX + i].dis + H2H_label[checkPointY + i].dis) : resu;
    }
    result[tid] = resu;

}

long long int H2HQuery_UsingLCA(
    thrust::device_vector<int>& x,
    thrust::device_vector<int>& y,
    thrust::device_vector<int>& result,
    thrust::device_vector<int>& firstAppeare_D,
    thrust::device_vector<Graph_D_H::pairs>& RMQ_OneLine_D,
    thrust::device_vector<Graph_D_H::pairs>& H2H_startIndex_D,
    thrust::device_vector<Graph_D_H::H2HLabel>& H2H_label_D, 
    int RMQ_Size,int  RMQ_Line_Size, int cudaThreadNum,int querysize) {
    int querySize = querysize;
    int* X = thrust::raw_pointer_cast(x.data());
    int* Y = thrust::raw_pointer_cast(y.data());
    int* RESULT = thrust::raw_pointer_cast(result.data());
    int* firstAppare = thrust::raw_pointer_cast(firstAppeare_D.data());
    Graph_D_H::pairs* RMQ_Oneline = thrust::raw_pointer_cast(RMQ_OneLine_D.data());
    Graph_D_H::pairs* H2H_startIndex = thrust::raw_pointer_cast(H2H_startIndex_D.data());
    Graph_D_H::H2HLabel* H2H_label = thrust::raw_pointer_cast(H2H_label_D.data());

    int blockSize = querySize / cudaThreadNum + 1;

    Graph_D_H::time_Mine time;
    cout << "H2H_D_UsingLCA start query,querySize=" <<querysize << endl;
    time.updateStart();

    H2Hquery_UsingLCA_D<<<blockSize, cudaThreadNum >>>(X, Y, RESULT, querySize, H2H_startIndex, H2H_label, firstAppare,
        RMQ_Oneline, RMQ_Size, RMQ_Line_Size);
    hipDeviceSynchronize();

    time.updateEnd();
    cout << "using time: " << time.get_microsecond_duration() << "us" << endl;
    return time.get_microsecond_duration();
}




__global__
void H2Hquery_NoLCA_D(int* x, int* y, int* result, const int querySize, Graph_D_H::pairs* H2H_startIndex, 
    Graph_D_H::H2HLabel* H2H_label) {
    unsigned int tid = threadIdx.x + (blockDim.x * blockIdx.x);
    if (querySize <= tid)
        return;
    if (x[tid] == y[tid])
    {
        result[tid] = 0;
        return;
    }
    int checkPointX = H2H_startIndex[x[tid]].first;
    int checkPointY = H2H_startIndex[y[tid]].first;
    int frontier = -1;
    if (H2H_startIndex[x[tid]].second > H2H_startIndex[y[tid]].second) {
        frontier = H2H_startIndex[y[tid]].second;
    }
    else {
        frontier = H2H_startIndex[x[tid]].second;
    }
    int resu = INT_MAX;
    for (int i = 0; i < frontier; i++)
    {
        if (H2H_label[checkPointX + i].Node != H2H_label[checkPointY + i].Node)
            break;
        if ((H2H_label[checkPointX + i].dis + H2H_label[checkPointY + i].dis) < resu) {
            resu = (H2H_label[checkPointX + i].dis + H2H_label[checkPointY + i].dis);
        }
        //resu = ((H2H_label[checkPointX + i].dis + H2H_label[checkPointY + i].dis) < resu) ?
        //    (H2H_label[checkPointX + i].dis + H2H_label[checkPointY + i].dis) : resu;
    }
    result[tid] = resu;
}

long long int H2HQuery_NoLCA(
    thrust::device_vector<int>& x,
    thrust::device_vector<int>& y,
    thrust::device_vector<int>& result,
    thrust::device_vector<Graph_D_H::pairs>& H2H_startIndex_D,
    thrust::device_vector<Graph_D_H::H2HLabel>& H2H_label_D, int cudaThreadNum,int querysize) {
    //int querySize = querysize;
    int* X = thrust::raw_pointer_cast(x.data());
    int* Y = thrust::raw_pointer_cast(y.data());
    int* RESULT = thrust::raw_pointer_cast(result.data());
    Graph_D_H::pairs* H2H_startIndex = thrust::raw_pointer_cast(H2H_startIndex_D.data());
    Graph_D_H::H2HLabel* H2H_label = thrust::raw_pointer_cast(H2H_label_D.data());

    int blockSize = querysize / cudaThreadNum + 1;

    Graph_D_H::time_Mine time;
    cout << "H2H_D_noLCA start query,querySize=" << querysize << endl;
    time.updateStart();

    H2Hquery_NoLCA_D << <blockSize, cudaThreadNum >> > (X, Y, RESULT, querysize, H2H_startIndex, H2H_label);
    hipDeviceSynchronize();

    time.updateEnd();
    cout << "using time: " << time.get_microsecond_duration() << "us" << endl;
    return time.get_microsecond_duration();
}






















__global__
void H2HQuery_UsingLCA_D_noHub(int* x, int* y, int* result, const int querySize, 

    int64_t* H2H_pos_hash,
    int* H2H_pos_POS,
    int64_t* H2H_dis_hash,
    int* H2H_dis,

    int* firstAppeare,
    int64_t* RMQHash,
    int* RMQ_ID,
    int* H2H_TreeHeight_Hash
) 
{
    unsigned int tid = threadIdx.x + (blockDim.x * blockIdx.x);
    if (querySize <= tid)
        return;
    if (x[tid] == y[tid])
    {
        result[tid] = 0;
        return;
    }
    int FirstA = firstAppeare[x[tid]];//position  in  OULA_DFS
    int FirstB = firstAppeare[y[tid]];
    int firstA = (FirstA < FirstB) * FirstA + (FirstB <= FirstA) * FirstB;
    int firstB = (FirstA < FirstB) * FirstB + (FirstB <= FirstA) * FirstA;
    int k = (int)(log2((double)(firstB - firstA + 1)));//k means log2(gap);

    int ID_1 = RMQ_ID[RMQHash[firstA] + (int64_t)k];
    int ID_2 = RMQ_ID[RMQHash[firstB - (1 << k) + 1] + (int64_t)k];
    int LCA = ID_2;
    if (H2H_TreeHeight_Hash[ID_1] < H2H_TreeHeight_Hash[ID_2]) {
      LCA = ID_1;
    }

    int resu = INT_MAX;
    for (int64_t i = H2H_pos_hash[LCA]; i < H2H_pos_hash[LCA + 1]; i++) {
       
        int templength = H2H_dis[H2H_dis_hash[x[tid]] + (int64_t)H2H_pos_POS[i]] + 
            H2H_dis[H2H_dis_hash[y[tid]] + (int64_t)H2H_pos_POS[i]];
        if (templength < resu) {
            resu = templength;
        }
        //atomicMin(&resu, templength);
        //atomicMin(&result[tid], templength);
    }
    result[tid] = resu;
}

long long int H2HQuery_UsingLCA_noHub(
    thrust::device_vector<int>& x,
    thrust::device_vector<int>& y,
    thrust::device_vector<int>& result,

    thrust::device_vector<int64_t>& H2H_pos_hash_D,
    thrust::device_vector<int64_t>& H2H_dis_hash_D,
    thrust::device_vector<int>& H2H_pos_POS_D,
    thrust::device_vector<int>& H2H_dis_D,

    thrust::device_vector<int>& firstAppeare_D,
    thrust::device_vector<int64_t>& RMQHash_D,
    thrust::device_vector<int>& RMQ_ID_D, 
    thrust::device_vector<int>& H2H_TreeHeight_Hash_D,

    int cudaThreadNum, int querysize
) {
    //int querySize = querysize;
    thrust::fill(result.begin(), result.end(), INT_MAX);
    int* X = thrust::raw_pointer_cast(x.data());
    int* Y = thrust::raw_pointer_cast(y.data());
    int* RESULT = thrust::raw_pointer_cast(result.data());


    int64_t* H2H_pos_hash = thrust::raw_pointer_cast(H2H_pos_hash_D.data());
    int64_t* H2H_dis_hash = thrust::raw_pointer_cast(H2H_dis_hash_D.data());
    int* H2H_pos_POS = thrust::raw_pointer_cast(H2H_pos_POS_D.data());
    int* H2H_dis = thrust::raw_pointer_cast(H2H_dis_D.data());

    int* firstAppare = thrust::raw_pointer_cast(firstAppeare_D.data());
    int64_t* RMQHash = thrust::raw_pointer_cast(RMQHash_D.data());
    int* RMQ_ID = thrust::raw_pointer_cast(RMQ_ID_D.data());
    int* H2H_TreeHeight_Hash = thrust::raw_pointer_cast(H2H_TreeHeight_Hash_D.data());

    int blockSize = querysize / cudaThreadNum + 1;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    Graph_D_H::time_Mine time;
    cout << "H2H_D_UsingLCA start query,querySize=" << querysize << endl;
    time.updateStart();
    hipEventRecord(start);
    H2HQuery_UsingLCA_D_noHub << <blockSize, cudaThreadNum >> > (X, Y, RESULT, querysize, H2H_pos_hash, H2H_pos_POS, H2H_dis_hash, H2H_dis, firstAppare, RMQHash, RMQ_ID, H2H_TreeHeight_Hash);
    //hipDeviceSynchronize();
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float milliseconds = 0;
    long long int micro = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    micro = (long long int)(milliseconds * 1000);
    //time.updateEnd();

    hipEventDestroy(start);
    hipEventDestroy(stop);
    //CUDA_CHECK_ERROR();

    cout << "using time: " << micro << "us" << endl;
    return micro;
}


__global__
void H2HQuery_noLCA_D_noHub(int* x, int* y, int* result, const int querySize,
    int64_t* H2H_dis_hash,
    int* H2H_dis,
    int* firstAppeare,
    int64_t* RMQHash,
    int* RMQ_ID,
    int* H2H_TreeHeight_Hash)
{
    unsigned int tid = threadIdx.x + (blockDim.x * blockIdx.x);
    if (querySize <= tid)
        return;
    if (x[tid] == y[tid])
    {
        result[tid] = 0;
        return;
    }
    int FirstA = firstAppeare[x[tid]];//position  in  OULA_DFS
    int FirstB = firstAppeare[y[tid]];
    int firstA = (FirstA < FirstB) * FirstA + (FirstB <= FirstA) * FirstB;
    int firstB = (FirstA < FirstB) * FirstB + (FirstB <= FirstA) * FirstA;
    int k = (int)(log2((double)(firstB - firstA + 1)));//k means log2(gap);

    int ID_1 = RMQ_ID[RMQHash[firstA] + (int64_t)k];
    int ID_2 = RMQ_ID[RMQHash[firstB - (1 << k) + 1] + (int64_t)k];
    int LCASize = H2H_TreeHeight_Hash[ID_2];
    //atomicMin(&LCASize, H2H_TreeHeight_Hash[ID_1]);
    if (H2H_TreeHeight_Hash[ID_1] < H2H_TreeHeight_Hash[ID_2]) {
        LCASize = H2H_TreeHeight_Hash[ID_1];
    }

    int resu = INT_MAX;

    for (int64_t i = 0; i < LCASize; i++) {

        int templength = H2H_dis[H2H_dis_hash[x[tid]] + i] + H2H_dis[H2H_dis_hash[y[tid]] + i];
        if (templength < resu) {
            resu = templength;
        }
        //atomicMin(&resu, templength);
        //atomicMin(&result[tid], templength);
    }
    result[tid] = resu;
}

long long int H2HQuery_noLCA_noHub(
    thrust::device_vector<int>& x,
    thrust::device_vector<int>& y,
    thrust::device_vector<int>& result,

    thrust::device_vector<int64_t>& H2H_dis_hash_D,
    thrust::device_vector<int>& H2H_dis_D,

    thrust::device_vector<int>& firstAppeare_D,
    thrust::device_vector<int64_t>& RMQHash_D,
    thrust::device_vector<int>& RMQ_ID_D,
    thrust::device_vector<int>& H2H_TreeHeight_Hash_D,
    
    int cudaThreadNum, int querysize) {

    thrust::fill(result.begin(), result.end(), INT_MAX);
    //int querySize = querysize;
    int* X = thrust::raw_pointer_cast(x.data());
    int* Y = thrust::raw_pointer_cast(y.data());
    int* RESULT = thrust::raw_pointer_cast(result.data());


    int64_t* H2H_dis_hash = thrust::raw_pointer_cast(H2H_dis_hash_D.data());
    int* H2H_dis = thrust::raw_pointer_cast(H2H_dis_D.data());

    int* firstAppare = thrust::raw_pointer_cast(firstAppeare_D.data());
    int64_t* RMQHash = thrust::raw_pointer_cast(RMQHash_D.data());
    int* RMQ_ID = thrust::raw_pointer_cast(RMQ_ID_D.data());
    int* H2H_TreeHeight_Hash = thrust::raw_pointer_cast(H2H_TreeHeight_Hash_D.data());


    int blockSize = querysize / cudaThreadNum + 1;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    //Graph_D_H::time_Mine time;
    cout << "H2H_D_noLCA start query,querySize = " << querysize << endl;
    //time.updateStart();
    hipEventRecord(start);
    H2HQuery_noLCA_D_noHub <<<blockSize, cudaThreadNum>>>(X, Y, RESULT, querysize, H2H_dis_hash, H2H_dis, firstAppare, RMQHash, RMQ_ID, H2H_TreeHeight_Hash);
    //hipDeviceSynchronize();
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float milliseconds = 0;
    long long int micro = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    micro = (long long int)(milliseconds * 1000);
    //time.updateEnd();

    hipEventDestroy(start);
    hipEventDestroy(stop);
    //CUDA_CHECK_ERROR();

     cout << "using time: " << micro << "us" << endl;
     return micro;
}
