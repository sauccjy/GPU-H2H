#include "hip/hip_runtime.h"
#pragma once
#include"Graph_D.cuh"

namespace Graph_D_H
{
	void Graph::makeAdjcentNode(int goalHeight)
	{
		Graph_D_H::time_Mine time;

		cout << "start construct Partition Rank tree " << endl;
		time.updateStart();

		int tempHeight = TreeHeight;
		thrust::host_vector<bool> isAdjcent(NodeNumber, true);
		//thrust::host_vector<pairs> range(NodeNumber, pairs(INT_MAX, INT_MIN));
		thrust::host_vector<int> placeOccupy(NodeNumber, 0);

		for (int i = 0; i < NodeNumber; i++) {
			placeOccupy[i] = adjList[i].size();
		}

		NANHash.assign(partition_Tree.size(), myPair<int>());
		nonAdjcentNode.clear();
		vector<double> afs(TreeHeight + 1, 0);
		while (tempHeight > goalHeight)
		{
			int lowestIndexStart = std::pow(2, tempHeight - 1) - 1;
			int lowestIndexEnd = std::pow(2, tempHeight) - 2;
			thrust::host_vector<bool> isAdjcent_cp = isAdjcent;
			for (int i = lowestIndexEnd; i >= lowestIndexStart; i--)
			{
				NANHash[i].first = nonAdjcentNode.size();//index
				NANHash[i].second = 0; //maxSize
				NANHash[i].NodeID = 0;//nonAdjcent Size
				int left = partition_Tree[i].first;
				int right = partition_Tree[i].second;
				//vector<int> outPartitionSize(right - left, 0);
				map<int, int> subgraph;
				for (int j = left; j < right; j++)//construct sub graph
				{
					if (!isAdjcent[NE_P[j].NodeID])
						continue;
					int nodeID = NE_P[j].NodeID;
					subgraph.emplace(nodeID, 0);
				}
				NANHash[i].second = subgraph.size();//record max size
				for (auto& it : subgraph) {//define outsize
					it.second = subgraph.size();
					for (auto& adj : adjList[it.first]) {
						if (!isAdjcent[adj.first]) {
							continue;
						}
						if (subgraph.find(adj.first) == subgraph.end()) {
							it.second++;
							//isAdjcent_cp[it.first] = false;
						}
					}
				}

				for (auto& it : subgraph) {
					if (it.second == NANHash[i].second) {// mark non-adjNode
						isAdjcent_cp[it.first] = false;
						NANHash[i].NodeID++;
						nonAdjcentNode.push_back(TDrank(adjList[it.first].size(), it.first, 0));
					}
					placeOccupy[it.first] = max(placeOccupy[it.first], it.second);

				}
			}
			isAdjcent = isAdjcent_cp;
			afs[tempHeight] = nonAdjcentNode.size();
			tempHeight--;
		}



		time.updateEnd();
		makePartitionRankTreeTime = time.get_microsecond_duration();
		cout << "\t construct Partition Rank tree end,  using time: " << time.get_microsecond_duration() << endl;
		string latitudePartition = "LatitudePartitionRate.csv";
		if (PartitionMethod == 2) latitudePartition = "MinimumPartitionRate.csv";
		if (PartitionMethod == 3) latitudePartition = "metisPartitionRate.csv";
		if (PartitionMethod == 4) latitudePartition = "scotchPartitionRate.csv";
		std::fstream heightdata(latitudePartition, ios::in | ios::out | ios::app);

		heightdata << graphName << ",";
		////cout << "NodeNumber per layer: \n";
		for (int i = 1; i < afs.size(); i++) {
			heightdata << afs[i] / NodeNumber << ",";
			cout << "\t layer : " << i << " size: " << std::fixed << afs[i] << endl;
		}
		heightdata << "\n";
		heightdata.close();

		//allowcate CHTree
		cout << "start Allocate LUB " << endl;
		time.updateStart();
		CHTree.clear();
		CHTreeHash.assign(NodeNumber, myPair<int>());
		
		for (int i = 0; i < NodeNumber; i++)
		{
			int nodeID = NE_P[i].NodeID;
			CHTreeHash[i].first = 0;//actual size
			CHTreeHash[i].second = placeOccupy[nodeID];//maxSize
			//cout << placeOccupy[i] << endl;
			CHTreeHash[i].NodeID = CHTree.size();//CHTree index
			CHTree.insert(CHTree.end(), CHTreeHash[i].second, myPair<int>(INT_MAX,0,-1)); //insert maxSize label(outpoint,weight,hub)
			
			for (auto& it : adjList[nodeID])
			{
				CHTree[CHTreeHash[i].NodeID + CHTreeHash[i].first].setPairs(-1,it.first,it.second);
				CHTreeHash[i].first++;
				if (CHTreeHash[i].first > CHTreeHash[i].second) {
					cout << "LUB error~! :::" << CHTreeHash[i].first - CHTreeHash[i].second << endl;
					cout << "\t actual adjcent size: " << adjList[nodeID].size() << " lub size: " << CHTreeHash[i].second << endl;
					throw("LUB allocate Error!!");
				}
			}
		}
		
		chSize = CHTree.size();

		time.updateEnd();
		AllocateLUBTime = time.get_microsecond_duration();
		cout << "\t Malloc LUB end,  using time: " << time.get_microsecond_duration() << endl;
		//cout << "LUB Size: " << ((double)(chSize) * sizeof(Graph_D_H::myPair<int>)) / (1024 * 1024) << "MB" << endl;
		LUBSize = ((double)(chSize) * sizeof(Graph_D_H::myPair<int>)) / (1024 * 1024);
		cout << "LUB Size: " << ((double)(chSize) * sizeof(Graph_D_H::myPair<int>)) / (1024 * 1024) << "MB" << endl;
		cout << "LUB length: " << chSize << endl;

		NE_P.clear();
		NE_P.shrink_to_fit();
		partition_Tree.clear();
		partition_Tree.shrink_to_fit();
	}

}
