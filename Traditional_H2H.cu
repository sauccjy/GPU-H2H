#include "hip/hip_runtime.h"
#pragma once
#include"Graph_D.cuh"

namespace Graph_D_H
{
	void Graph_D_H::Graph::TDInCPU(int tempheight)
	{
		Graph_D_H::time_Mine time;
		std::cout << "translate and malloc back to CPU start " << endl;
		time.updateStart();
		uploadCHset(tempheight);

		time.updateEnd();
		translateBackAndMallocToCPUTime = time.get_microsecond_duration();
		std::cout << "\t translate and malloc back to CPU end, using time: " << time.get_microsecond_duration() << endl;

		std::cout << "\n" << "TD in CPU start " << endl;
		time.updateStart();
		CHUsingHeap_2();
		time.updateEnd();
		TDInCPUTime = time.get_microsecond_duration();
		std::cout << "\t TD in CPU end, using time: " << time.get_microsecond_duration() << endl;


	}


	void Graph::uploadCHset(int tempheight)
	{
		for (auto& it : adjList) {
			it.clear();
			it.shrink_to_fit();
		}
		adjList.clear();
		adjList.shrink_to_fit();
		int tempHeight = tempheight;
		CHInfo.assign(NodeNumber, pairs());
		CHChild.assign(NodeNumber, 0);
		DBC.assign(NodeNumber, 1);
		//bool haveless = false;
		//int wrongSize = 0;
		cout << "clear finished!" << endl;
		while (tempHeight > 0)
		{
			int lowestIndexStart = std::pow(2, tempHeight - 1) - 1;
			int lowestIndexEnd = std::pow(2, tempHeight) - 2;
			tempHeight--;

			for (int i = lowestIndexEnd; i >= lowestIndexStart; i--)
			{
				int startIndex = NANHash[i].first;
				int TDNodeSize = NANHash[i].NodeID;
				tempnonAdjcentNodeIndex = min(startIndex, tempnonAdjcentNodeIndex);
				for (int j = startIndex; j < startIndex + TDNodeSize; j++)
				{
					int TDNodeNow = nonAdjcentNode[j].second;
					int CHTreeHash_index = ID_hash[TDNodeNow];
					int CHTree_Index = CHTreeHash[CHTreeHash_index].NodeID;
					int CHTree_ActualSize = CHTreeHash[CHTreeHash_index].first;
					int size = 0;
					for (int k = CHTree_Index; k < CHTree_Index + CHTree_ActualSize; k++)
					{
						if (CHTree[k].first == INT_MAX)
							continue;
						size++;
						CHAdjlist[TDNodeNow].emplace(CHTree[k].first, pairs(CHTree[k].second, CHTree[k].NodeID));
						//CHTree[k].first = INT_MAX;
					}

					Graph_D_H::CHInfo[TDNodeNow].pairsReset(size, CHInfo_H[TDNodeNow].second);
					heap.insert(HostRank(TDNodeNow));

				}
			}
		}
		tempHeight = TreeHeight;
		while (tempHeight > tempheight)
		{
			int lowestIndexStart = std::pow(2, tempHeight - 1) - 1;
			int lowestIndexEnd = std::pow(2, tempHeight) - 2;
			tempHeight--;
			for (int i = lowestIndexEnd; i >= lowestIndexStart; i--)
			{
				int startIndex = NANHash[i].first;
				int TDNodeSize = NANHash[i].NodeID;
				for (int j = startIndex; j < startIndex + TDNodeSize; j++)
				{
					int TDNodeNow = nonAdjcentNode[j].second;
					int CHTreeHash_index = ID_hash[TDNodeNow];
					int CHTree_Index = CHTreeHash[CHTreeHash_index].NodeID;
					int CHTree_ActualSize = CHTreeHash[CHTreeHash_index].first;
					//int size = 0;
					for (int k = CHTree_Index; k < CHTree_Index + CHTree_ActualSize; k++)
					{
						if (CHTree[k].first == INT_MAX)
							continue;
						//size++;
						CHAdjlist[TDNodeNow].emplace(CHTree[k].first, pairs(CHTree[k].second, CHTree[k].NodeID));
						//CHTree[k].first = INT_MAX;
					}
					//Graph_D_H::CHInfo[TDNodeNow].pairsReset(size, 0);
				}
			}
		}
		CHTree.clear();
		CHTree.shrink_to_fit();
		CHTreeHash.clear();
		CHTreeHash.shrink_to_fit();
		CHTree_D.clear();
		CHTree_D.shrink_to_fit();
		CHTreeHash_D.clear();
		CHTreeHash_D.shrink_to_fit();
		CHInfo_H.clear();
		CHInfo_H.shrink_to_fit();
		CHInfo_D.clear();
		CHInfo_D.shrink_to_fit();
		std::cout << " \t node in heap now: " << heap.size() << endl;
	}


	void Graph::CHUsingHeap()
	{
		long long int tempNANIndex = tempnonAdjcentNodeIndex;
		vertexesConstructedInGPU = tempnonAdjcentNodeIndex;
		std::cout<<"\t vertexes Constructed in lower layer: " << tempNANIndex << endl;
		int total = 0;
		while (!heap.empty())
		{
			int TDNodeNow = heap.begin()->x;
			heap.erase(heap.begin());

			if (heap.size() < 10000) {
				std::cout << "file in " << endl;
			}

			total++;
			vector<pairs> info;
			vector<int> nei;
			nonAdjcentNode[tempNANIndex++].setDetal(CHAdjlist[TDNodeNow].size(), TDNodeNow, Graph_D_H::CHInfo[TDNodeNow].second);
			for (auto i = CHAdjlist[TDNodeNow].begin(); i != CHAdjlist[TDNodeNow].end(); i++)
			{
				nei.push_back(i->first);
				info.push_back(i->second);
			}
			for (int i = 0; i < nei.size(); i++)
			{
				int neiID = nei[i];
				auto it = CHAdjlist[neiID].find(TDNodeNow);
				if (it != CHAdjlist[neiID].end())
				{
					CHAdjlist[neiID].erase(it);
				}
			}
			for (int i = 0; i < nei.size(); i++)
			{
				int leftNode = nei[i];
				for (int j = i + 1; j < nei.size(); j++)
				{
					int rightNode = nei[j];
					int weight = info[i].first + info[j].first;
					if (CHAdjlist[leftNode].find(rightNode) == CHAdjlist[leftNode].end())
					{
						CHAdjlist[leftNode].insert(make_pair(rightNode, pairs(weight, TDNodeNow)));
					}
					else
					{
						if (CHAdjlist[leftNode][rightNode].first > weight)
						{
							CHAdjlist[leftNode][rightNode].first = weight;
							CHAdjlist[leftNode][rightNode].second = TDNodeNow;
						}
					}
					if (CHAdjlist[rightNode].find(leftNode) == CHAdjlist[rightNode].end())
					{
						CHAdjlist[rightNode].insert(make_pair(leftNode, pairs(weight, TDNodeNow)));
					}
					else
					{
						if (CHAdjlist[rightNode][leftNode].first > weight)
						{
							CHAdjlist[rightNode][leftNode].first = weight;
							CHAdjlist[rightNode][leftNode].second = TDNodeNow;
						}
					}
				}
			}
			
			for (int i = 0; i < nei.size(); i++)
			{
				int neiID = nei[i];
				int height = max(Graph_D_H::CHInfo[TDNodeNow].second + 1, Graph_D_H::CHInfo[neiID].second);
				if (height == Graph_D_H::CHInfo[neiID].second) {
					CHChild[neiID]++;
				}
				else {
					CHChild[neiID] = 1;
				}
				Graph_D_H::CHInfo[neiID].pairsReset(CHAdjlist[neiID].size(), height);
			}
			
		}

		int head = nonAdjcentNode[nonAdjcentNode.size() - 1].second;
		std::cout << "head: "<< head <<"and  CH adjlist size: " << CHAdjlist[head].size() << endl;
		long long int size1 = 0;
		for (auto& it : CHAdjlist) {
			size1 += it.size() + 1;
		}
		std::cout << "vertexes constructed in CPU: " << total << endl;
		DecompositionTreeSize = ((double)(size1 * sizeof(Graph_D_H::myPair<int>))) / (1024 * 1024);
		std::cout << "\t actual Decomposition Tree size = " << ((double)(size1* sizeof(Graph_D_H::myPair<int>)))/(1024*1024)<<"MB" << endl;
		std::cout << endl;
	}


	void Graph::CHUsingHeap_2()
	{
		long long int tempNANIndex = tempnonAdjcentNodeIndex;
		vertexesConstructedInGPU = tempnonAdjcentNodeIndex;
		std::cout << "\t vertexes Constructed in GPU: " << tempNANIndex << endl;
		int total = 0;
		while (!heap.empty())
		{

			int TDNodeNow = heap.begin()->x;
			heap.erase(heap.begin());
			if (visited[TDNodeNow])
			{
				std::cout << "constructNode: " << TDNodeNow << " ~" << endl;
			}
			total++;
			visited[TDNodeNow] = true;

			vector<pairs> info;
			vector<int> nei;
			nonAdjcentNode[tempNANIndex++].setDetal(CHAdjlist[TDNodeNow].size(), TDNodeNow, Graph_D_H::CHInfo[TDNodeNow].second);
			//std::cout <<"nodeID: "<<TDNodeNow << " adjlist size: " << CHAdjlist[TDNodeNow].size() ;
			//std::cout << "heap size: " << heap.size();
				//int CHTreeHash_index = ID_hash[TDNodeNow];
				//int CHTree_Index = CHTreeHash[CHTreeHash_index].NodeID;

			for (auto i = CHAdjlist[TDNodeNow].begin(); i != CHAdjlist[TDNodeNow].end(); i++)
			{
				//if (visited[i->first])
				//{
				//	cout << "stopedID: " << TDNodeNow<<"adjNode: "<<i->first << endl;
				//	cin.get();
				//	continue;
				//}
				nei.push_back(i->first);
				info.push_back(i->second);
				//CHTree[CHTree_Index++].setPairs((i->second).second, i->first, (i->second).first);
			}

			//firstempty[CHTreeHash_index] = CHAdjlist[TDNodeNow].size();
			//CHTreeHash[CHTreeHash_index].first = CHAdjlist[TDNodeNow].size();
			//std::cout << "erase id: ";
			for (int i = 0; i < nei.size(); i++)
			{
				int neiID = nei[i];
				auto it = CHAdjlist[neiID].find(TDNodeNow);
				if (it != CHAdjlist[neiID].end())
				{
					CHAdjlist[neiID].erase(it);
				}
				//CHAdjlist[neiID].erase(TDNodeNow);
				//std::cout << " " << neiID;
				heap.erase(HostRank(neiID));
			}
			//std::cout << "; erase adj , heapSize: " << heap.size() << "";
			for (int i = 0; i < nei.size(); i++)
			{
				int leftNode = nei[i];
				for (int j = i + 1; j < nei.size(); j++)
				{
					int rightNode = nei[j];
					int weight = info[i].first + info[j].first;
					if (CHAdjlist[leftNode].find(rightNode) == CHAdjlist[leftNode].end())
					{
						CHAdjlist[leftNode].insert(make_pair(rightNode, pairs(weight, TDNodeNow)));
					}
					else
					{
						if (CHAdjlist[leftNode][rightNode].first > weight)
						{
							CHAdjlist[leftNode][rightNode].first = weight;
							CHAdjlist[leftNode][rightNode].second = TDNodeNow;
						}
					}
					if (CHAdjlist[rightNode].find(leftNode) == CHAdjlist[rightNode].end())
					{
						CHAdjlist[rightNode].insert(make_pair(leftNode, pairs(weight, TDNodeNow)));
					}
					else
					{
						if (CHAdjlist[rightNode][leftNode].first > weight)
						{
							CHAdjlist[rightNode][leftNode].first = weight;
							CHAdjlist[rightNode][leftNode].second = TDNodeNow;
						}
					}
				}
			}

			for (int i = 0; i < nei.size(); i++)
			{
				int neiID = nei[i];
				int height = max(Graph_D_H::CHInfo[TDNodeNow].second + 1, Graph_D_H::CHInfo[neiID].second);
				if (height == Graph_D_H::CHInfo[neiID].second) {
					//CHChild[neiID]++;
					//DBC[neiID] = DBC[neiID] * (CHChild[TDNodeNow] + 1);
					CHChild[neiID] += CHChild[TDNodeNow] + 1;
				}
				else {
					CHChild[neiID] = 1 + CHChild[TDNodeNow];
					//DBC[neiID] =1 + CHChild[TDNodeNow];
				}
				Graph_D_H::CHInfo[neiID].pairsReset(CHAdjlist[neiID].size(), height);
				//if(!visited[neiID])
				heap.insert(HostRank(neiID));
			}
			//std::cout << "; add adj , heapSize: " << heap.size() << ""<<endl;
		}

		int head = nonAdjcentNode[nonAdjcentNode.size() - 1].second;
		std::cout << "head: " << head << "and  CH adjlist size: " << CHAdjlist[head].size() << endl;

		for (auto& it : CHAdjlist) {
			size1 += it.size() + 1;
		}


		std::cout << "vertexes constructed in CPU: " << total << endl;
		DecompositionTreeSize = ((double)(size1 * sizeof(Graph_D_H::myPair<int>))) / (1024 * 1024);
		std::cout << "\t actual Decomposition Tree size = " << ((double)(size1 * sizeof(Graph_D_H::myPair<int>))) / (1024 * 1024) << "MB" << endl;
		std::cout << endl;
		//set<int> mai;
		//for (auto& i : nonAdjcentNode)
		//{
		//	mai.insert(i.second);
		//}
		//std::cout << "Number:" << NodeNumber << "  Actual : " << mai.size() << endl;
		//int size = 0;
		//for (int i = 0; i < NodeNumber; i++)
		//{
		//	if (!visited[i]) {
		//		size++;
		//	}
		//}
		//cout <<"left"<<  size << endl;
	}


	void Graph_D_H::Graph::partition_TD_T(int height)
	{
		CHTree.clear();
		CHTree.shrink_to_fit();
		CHTreeHash.clear();
		CHTreeHash.shrink_to_fit();
		CHTree_D.clear();
		CHTree_D.shrink_to_fit();
		CHTreeHash_D.clear();
		CHTreeHash_D.shrink_to_fit();

		CHInfo.assign(NodeNumber, pairs());
		CHChild.assign(NodeNumber, 0);
		DBC.assign(NodeNumber, 1);
		//loading CHAdjList
		for (int i = 0; i < NodeNumber; i++) {
			for (auto& it : adjList[i]) {
				CHAdjlist[i].emplace(it.first, pairs(it.second, -1));
			}
			CHInfo[i].pairsReset(adjList[i].size(), 0);
			adjList[i].clear();
			adjList[i].shrink_to_fit();
		}
		adjList.clear();
		adjList.shrink_to_fit();
		int tempHeight = TreeHeight;
		cout << "start TD" << endl;
		while (tempHeight > height) {
			int lowestIndexStart = std::pow(2, tempHeight - 1) - 1;
			int lowestIndexEnd = std::pow(2, tempHeight) - 1;

			int totalSubGraph = lowestIndexEnd - lowestIndexStart;
			int maxSubGraphPerThread = totalSubGraph / threadNumber + 1;
			vector<vector<HostRank>> candidateHeap(threadNumber, vector<HostRank>());
			vector<int> range(threadNumber, INT_MAX);
			int it = 0;
			int temp = 0;
			//allocate contract vertexes on different thread
			for (int i = lowestIndexStart; i < lowestIndexEnd; i++) {
				int left = NANHash[i].first;
				int right = NANHash[i].NodeID;

				for (int j = left; j < right + left; j++) {
					candidateHeap[it].push_back(HostRank(nonAdjcentNode[j].second));
				}
				range[it] = min(range[it], left);
				temp++;
				if (temp == maxSubGraphPerThread) {
					it++;
					temp = 0;
				}
			}
			std::vector<std::thread> threads;
			for (int i = 0; i < threadNumber; i++) {
				threads.emplace_back(
					[this, &candidateHeap, &range, i]() {

						while (!candidateHeap[i].empty()) {
							thrust::stable_sort(candidateHeap[i].begin(), candidateHeap[i].end());
							int TDNodeNow = candidateHeap[i].begin()->x;
							vector<pairs> info;
							vector<int> nei;
							nonAdjcentNode[range[i]].setDetal(CHAdjlist[TDNodeNow].size(), TDNodeNow, Graph_D_H::CHInfo[TDNodeNow].second);
							range[i]++;
							for (auto it = CHAdjlist[TDNodeNow].begin(); it != CHAdjlist[TDNodeNow].end(); it++) {
								nei.push_back(it->first);
								info.push_back(it->second);
							}
							for (int its = 0; its < nei.size(); its++) {
								int neiID = nei[its];
								if (CHAdjlist[neiID].find(TDNodeNow) != CHAdjlist[neiID].end()) {
									CHAdjlist[neiID].erase(TDNodeNow);
								}
							}
							for (int it = 0; it < nei.size(); it++) {
								int leftNode = nei[it];
								for (int jt = it + 1; jt < nei.size(); jt++) {
									int rightNode = nei[jt];
									int weight = info[it].first + info[jt].first;
									if (CHAdjlist[leftNode].find(rightNode) == CHAdjlist[leftNode].end()) {
										CHAdjlist[leftNode].insert(make_pair(rightNode, pairs(weight, TDNodeNow)));
									}
									else {
										if (CHAdjlist[leftNode][rightNode].first > weight) {
											CHAdjlist[leftNode][rightNode].first = weight;
											CHAdjlist[leftNode][rightNode].second = TDNodeNow;
										}
									}
									if (CHAdjlist[rightNode].find(leftNode) == CHAdjlist[rightNode].end()) {
										CHAdjlist[rightNode].insert(make_pair(leftNode, pairs(weight, TDNodeNow)));
									}
									else {
										if (CHAdjlist[rightNode][leftNode].first > weight) {
											CHAdjlist[rightNode][leftNode].first = weight;
											CHAdjlist[rightNode][leftNode].second = TDNodeNow;
										}
									}
								}
							}
							for (int it = 0; it < nei.size(); it++) {
								int neiID = nei[it];
								int height = max(Graph_D_H::CHInfo[TDNodeNow].second + 1, Graph_D_H::CHInfo[neiID].second);
								if (height == Graph_D_H::CHInfo[neiID].second) {
									//CHChild[neiID]++;
									CHChild[neiID] += CHChild[TDNodeNow];
								}
								else {
									CHChild[neiID] = 1 + CHChild[TDNodeNow];
								}
								Graph_D_H::CHInfo[neiID].pairsReset(CHAdjlist[neiID].size(), height);

							}
							candidateHeap[i].erase(candidateHeap[i].begin());
						}
					}
				);
			}

			for (auto& its : threads) {
				its.join();
			}
			tempHeight--;
		}





		while (tempHeight > 0) {
			int lowestIndexStart = std::pow(2, tempHeight - 1) - 1;
			int lowestIndexEnd = std::pow(2, tempHeight) - 1;
			tempHeight--;
			for (int i = lowestIndexStart; i < lowestIndexEnd; i++) {
				int left = NANHash[i].first;
				int right = NANHash[i].NodeID;
				tempnonAdjcentNodeIndex = min(left, tempnonAdjcentNodeIndex);
				for (int j = left; j < right + left; j++) {
					heap.insert(HostRank(nonAdjcentNode[j].second));
				}
			}
		}

	}


	void Graph_D_H::Graph::TDInCPUMultiThread(int tempheight)
	{
		Graph_D_H::time_Mine time;
		std::cout << "CPU partition contract start: " << endl;
		time.updateStart();
		partition_TD_T(tempheight);

		time.updateEnd();
		translateBackAndMallocToCPUTime = time.get_microsecond_duration();
		std::cout << "\t CPU multithread  partition contract end, using time: " << time.get_microsecond_duration() << endl;

		std::cout << "\n" << "TD in CPU serial start " << endl;
		time.updateStart();
		CHUsingHeap_2();
		time.updateEnd();
		TDInCPUTime = time.get_microsecond_duration() + translateBackAndMallocToCPUTime;
		std::cout << "\t TD in CPU serial end, using time: " << time.get_microsecond_duration() << endl;
	}
}