#include "hip/hip_runtime.h"
#pragma once
#include"Graph_D.cuh"

namespace Graph_D_H
{
	void Graph_D_H::Graph::beforeH2H()
	{
		getrank();
		getTreeStructure();
		if (useRMQ)
			makeOULAAndRMQ();
		else
			onlyOULA();

		cout << "\t CHTreeHeight: " << CHTreeHeight << endl;
		TDTREEHeight = CHTreeHeight;
		cout << "\t RMQ size : " << ((double)(RMQ_Line_Size * RMQ_Size * sizeof(Graph_D_H::pairs))) / (1024 * 1024)
			<< "MB, H2H Label size: " << ((double)H2H_Size * sizeof(Graph_D_H::H2HLabel)) / (1024 * 1024) << "MB" << endl;
		RMQsize = ((double)(RMQ_Line_Size * RMQ_Size * sizeof(Graph_D_H::pairs))) / (1024 * 1024);
		H2HLabelSize = ((double)H2H_Size * sizeof(Graph_D_H::H2HLabel)) / (1024 * 1024);
		father_D = father;
		cleanBeforeConstruct();
	}

	void Graph_D_H::Graph::cleanBeforeConstruct()
	{
		CHInfo.clear();
		CHInfo.shrink_to_fit();
		//CHChild.clear();
		//CHChild.shrink_to_fit();
		ID_hash.clear();
		ID_hash.shrink_to_fit();
		ID_hash_D.clear();
		ID_hash_D.shrink_to_fit();
		visited.clear();
		visited.shrink_to_fit();
		visited_D.clear();
		visited_D.shrink_to_fit();
	}

	void Graph_D_H::Graph::beforeH2H_noHub()
	{
		getrank();
		getTreeStructure();
		if (useRMQ)
			makeOULAAndRMQ();
		else
			onlyOULA();

		father_D = father;
		cout << "H2H_Size: " << (double)(H2H_Size * sizeof(int)) / (1024 * 1024) << endl;
		cleanBeforeConstruct();
	}

	void Graph_D_H::Graph::getrank()
	{
		ranks.assign(NodeNumber,-1);
		for (int i = 0; i < nonAdjcentNode.size(); i++)
		{
			ranks[nonAdjcentNode[i].second] = i;
		}
		head = nonAdjcentNode[nonAdjcentNode.size() - 1].second;

		nonAdjcentNode.clear();
		nonAdjcentNode_D.clear();
		NANHash.clear();
		NANHash_D.clear();
		nonAdjcentNode.shrink_to_fit();
		nonAdjcentNode_D.shrink_to_fit();
		NANHash.shrink_to_fit();
		NANHash_D.shrink_to_fit();
	}

	void Graph_D_H::Graph::getTreeStructure() {
		father.assign(NodeNumber, -1);

		ChildHash.assign(father.size(), pairs(0, -1));
		thrust::host_vector<int> ChildHash_actual_pos(father.size(), 0);
		Childs.assign(father.size() - 1, -1);
		for (int i = 0; i < NodeNumber; i++) {
			int fatherID = -1;
			int tempRank = INT_MAX;
			for (auto& it : CHAdjlist[i]) {
				int adjID = it.first;
				if (tempRank > ranks[adjID])
				{
					tempRank = ranks[adjID];
					fatherID = adjID;
				}
			}

			father[i] = fatherID;
		}

		ranks.clear();
		ranks.shrink_to_fit();
		for (int i = 0; i < NodeNumber; i++) {
			if (father[i] == -1)
				continue;
			ChildHash[father[i]].first++;
		}
		int temp = 0;
		for (int i = 0; i < NodeNumber; i++)
		{
			ChildHash[i].second = temp;
			temp += ChildHash[i].first;
		}
		for (int i = 0; i < NodeNumber; i++)
		{
			if (father[i] == -1)
				continue;
			//int fatherID = father[i];
			Childs[ChildHash[father[i]].second + ChildHash_actual_pos[father[i]]] = i;
			ChildHash_actual_pos[father[i]]++;
		}
	}
	
	void Graph::onlyOULA()
	{
		oulaOnly = true;
		OULA_Only.assign(NodeNumber, 0);
		thrust::host_vector<bool> visit(NodeNumber, false);
		if (head == -1)
			return;
		onlyOULADFS(head, 1, visit);
	}

	void Graph::onlyOULADFS(const int headNow, const int depth, thrust::host_vector<bool>& visit)
	{
		OULA_Only[headNow] = depth;
		H2H_Size += depth;
		visit[headNow] = true;
		CHTreeHeight = max(CHTreeHeight, depth);
		for (int i = ChildHash[headNow].second; i < ChildHash[headNow].second + ChildHash[headNow].first; i++)
		{
			if (!visit[Childs[i]])
			{
				onlyOULADFS(Childs[i], depth + 1, visit);
			}
		}
	}

	void Graph::makeOULAAndRMQ()
	{
		thrust::host_vector<bool> visit(NodeNumber, false);
		firstAppeare.assign(NodeNumber, -1);
		H2H_TreeHeight_Hash.assign(NodeNumber, -1);
		if (head == -1)
			return;

		OULADFS(head, 1, visit);

		//construct RMQ by DP
		int jSize = (int)(log2((double)OULA_DFS_.size()));

		RMQ_Size = OULA_DFS_.size();
		RMQ_Line_Size = jSize + 1;

		//RMQ_OneLine = new pairs[RMQ_Line_Size * RMQ_Size];
		RMQ_OneLine.assign(RMQ_Line_Size * RMQ_Size, pairs());
		for (auto i = 0; i < RMQ_Size; i++)
		{
			RMQ_OneLine[i * RMQ_Line_Size].first = OULA_DFS_[i].second;
			RMQ_OneLine[i * RMQ_Line_Size].second = OULA_DFS_[i].first;
		}
		for (int j = 1; j <= jSize; j++)
		{
			auto davicate = (1 << (j - 1));
			for (auto i = 0; i + davicate < RMQ_Size; i++)
			{
				if (RMQ_OneLine[i * RMQ_Line_Size + j - 1].first <= RMQ_OneLine[(i + davicate) * RMQ_Line_Size + j - 1].first)
					RMQ_OneLine[i * RMQ_Line_Size + j].pairsCopy(RMQ_OneLine[i * RMQ_Line_Size + j - 1]);
				else
					RMQ_OneLine[i * RMQ_Line_Size + j].pairsCopy(RMQ_OneLine[(i + davicate) * RMQ_Line_Size + j - 1]);
			}
		}

		RMQHash.assign(OULA_DFS_.size(), -1);
		for (int64_t i = 0; i < RMQ_Size; i++) {
			RMQHash[i] = RMQ_ID.size();
			for (int j = 0; j < RMQ_Line_Size; j++) {
				if (RMQ_OneLine[i * RMQ_Line_Size + j].first == -1) {
					break;
				}
				//RMQ_Height.push_back(RMQ_OneLine[i * RMQ_Line_Size + j].first);
				RMQ_ID.push_back(RMQ_OneLine[i * RMQ_Line_Size + j].second);
			}
		}
		RMQ_OneLine.clear();
		RMQ_OneLine.shrink_to_fit();


	}

	void Graph::OULADFS(const int headNow, const int depth, thrust::host_vector<bool>& visit)
	{
		OULA_DFS_.push_back(pairs(headNow, depth));
		firstAppeare[headNow] = OULA_DFS_.size() - 1;
		H2H_TreeHeight_Hash[headNow] = depth;
		H2H_Size += depth;
		visit[headNow] = true;
		CHTreeHeight = max(CHTreeHeight, depth);
		//int child_hash_index = ID_hash[headNow];
		int CHildStart = ChildHash[headNow].second;
		int CHildEnd = CHildStart + ChildHash[headNow].first;
		for (int i = CHildStart; i < CHildEnd; i++)
		{
			int childID = Childs[i];
			if (!visit[childID])
			{
				OULADFS(childID, depth + 1, visit);
				OULA_DFS_.push_back(pairs(headNow, depth));
			}
		}
	}
}